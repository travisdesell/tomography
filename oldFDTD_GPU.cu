//#define GLEW_STATIC
//#pragma comment(lib,"glew32.lib")
//#include <windows.h>
//#include <gl/glew.h>
//#include <glut.h>
#include <complex>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <cstdlib>
#include <fstream>
#include <hip/hip_runtime.h>
//#include "stdafx.h"
#include <iomanip>
#include <time.h>
//#include <cuda_gl_interop.h>

//#include <cuComplex.h>
#include <vector>
#include <math_functions.h>
//#include "EasyBMP.h"
//#include "EasyBMP_DataStructures.h"
//#include "EasyBMP_VariousBMPutilities.h"

#define GL_GLEXT_PROTOTYPES
#define PI 3.141592653589793238
#define alpha_max 0.01
#define alpha_min 0.000
#define eps0 8.85418e-12
#define sigma_factor 1.0
#define ncells 10
#define mu0 (PI*4e-7)
#define center_freq (5e9)
#define eta0 (sqrt(mu0/eps0))
#define c0 (1.0/sqrt(mu0*eps0))
#define dt (dx/c0/2)// dx/c0/2
#define domain_size 0.18
#define dx (0.001)
#define NF2FFdistfromboundary ((int)floor((3.2*breast_radius/dx)))
#define source_position 0.5
#define dy (0.001)
#define number_of_time_steps 3000
#define f1x (nx/2 - 150)       
#define f2x (nx/2+150) 
#define f1y (ny/2)
#define f2y (ny/2)
//#define nx ((int)ceil(domain_size/dx))
//#define ny ((int)ceil(domain_size/dy))
#define nx ((int)ceil(12.7*breast_radius/dx))
#define ny ((int)ceil(12.7*breast_radius/dy))
#define d (10*dx)
#define npml 2
#define kmax 10
#define numberofexcitationangles 4
#define isPW 1
#define isscattering 1
#define HANDLE_ERROR( err ) err
#define sigma_max_pml (3/(200*PI*dx))
#define size_NF2FF_total (2*nx-8*NF2FFdistfromboundary+2*ny-4)
#define size_cjzy (nx-2*NF2FFdistfromboundary-2)
#define size_cjzx (ny-2*NF2FFdistfromboundary)
#define numberofobservationangles  60
#define t0 (sqrt(20.0)*tau) // t0 = sqrt(20)*tau
#define l0 (nx*dx/2-breast_radius) 
#define pwidth 10
#define nc 20 // 20 cells per wavelength
#define  fmax  (c0/(nc*dx))// change if dy is bigger though now they're the same  fmax is the highest frequency this program can handle
#define tau (3.3445267e-11) // float ta bu = sqrt(2.3)*nc*dx/(PI*c0*1/sqrt(eps_r_MAX));  from a calculation of fmax.
//#define tau (5.288161e-11)
#define target_x (nx/2+15)//105 is breast_radius / dx
#define target_y (ny/2-15)
#define source_x (nx/2)      //(target_x-105-80)
#define source_y (ny/2)
#define breast_radius 0.0315 //87.535 mm  .  Sample size = 1.
#define tumor_size (0.01)

//#include <unistd.h>
//const cuComplex jcmpx (0.0, 1.0);
/*static void HandleError( cudaError_t err, const char *file,  int line ) {
  if (err != cudaSuccess) {
  printf( "%s in %s at line %d\n", cudaGetErrorString( err ),  file, line );
  exit( EXIT_FAILURE );
  }
  }*/


//__constant__ float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_Cezj,*dev_Jz,*dev_Chyh,*dev_Chxh,*dev_Chyez,*dev_Chxez,*dev_bex,*dev_bey,*dev_aex,*dev_aey,*dev_bmy,*dev_bmx,*dev_amy,*dev_amx,*dev_C_Psi_ezy,
//*dev_C_Psi_ezx,*dev_C_Psi_hxy,*dev_C_Psi_hyx;
struct hipComplex {
    float   r;
    float   i;
    __host__  __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __host__ __device__ hipComplex(float a): r(a), i(0) {}
    float magnitude2( void ) { return r * r + i * i; }
    __host__  __device__  hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __host__ __device__ hipComplex operator*(const float& a){
        return hipComplex(r*a,i*a);
    }

    __host__  __device__  hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
    __host__ __device__ hipComplex operator+(const float& a){
        return hipComplex(r+a,i);
    }
    __host__ __device__ void operator+=(const float& f){
        r += f;
    }
    __host__ __device__ void operator+=(const hipComplex& C);
    hipComplex();
};

__host__ __device__ hipComplex operator*(const float &f, const hipComplex &C)
{

    return hipComplex(C.r*f,C.i*f);
}

__host__ __device__ void hipComplex::operator+=(const hipComplex& C)
{
    r +=C.r;
    i += C.i;
}

__host__ __device__ float cuabs(hipComplex x)
{
    return sqrt(x.i*x.i + x.r*x.r);
}

__host__ __device__ hipComplex cuexp(hipComplex arg)
{
    hipComplex res(0,0);
    float s, c;
    float e = expf(arg.r);
    sincosf(arg.i,&s,&c);
    res.r = c * e;
    res.i = s * e;
    return res;

}

__device__ int isOnNF2FFBound(int x, int y)
{
    if(x==NF2FFdistfromboundary||x==nx-NF2FFdistfromboundary||y==NF2FFdistfromboundary||y==ny-NF2FFdistfromboundary)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ int getxfromthreadIdNF2FF(int index)
{
    int x=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))//yn
    {
        x = index+NF2FFdistfromboundary+1;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))//xp
    {
        x = nx-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))//yp
    {
        x = nx-NF2FFdistfromboundary  - (index-(nx-4*NF2FFdistfromboundary+ny-2))-2;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))//xn notice 2*nx-8*NF2FFdistfromboundary+2*ny-4 is the max index term.
    {
        x = NF2FFdistfromboundary;
    }
    return x;
}

__device__ int getyfromthreadIdNF2FF(int index)
{
    int y=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))
    {
        y = NF2FFdistfromboundary;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))
    {
        y = (index-(nx-2*NF2FFdistfromboundary-2))+NF2FFdistfromboundary;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))
    {
        y = ny-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))
    {
        y = ny-NF2FFdistfromboundary-(index-(2*nx-6*NF2FFdistfromboundary+ny-4))-1;
    }
    return y;
}

int CPUgetxfromthreadIdNF2FF(int index)
{
    int x=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))//yn
    {
        x = index+NF2FFdistfromboundary+1;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))//xp
    {
        x = nx-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))//yp
    {
        x = nx-NF2FFdistfromboundary  - (index-(nx-4*NF2FFdistfromboundary+ny-2))-2;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))//xn notice 2*nx-8*NF2FFdistfromboundary+2*ny-4 is the max index term.
    {
        x = NF2FFdistfromboundary;
    }
    return x;
}

int CPUgetyfromthreadIdNF2FF(int index)
{
    int y=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))
    {
        y = NF2FFdistfromboundary;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))
    {
        y = (index-(nx-2*NF2FFdistfromboundary-2))+NF2FFdistfromboundary;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))
    {
        y = ny-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))
    {
        y = ny-NF2FFdistfromboundary-(index-(2*nx-6*NF2FFdistfromboundary+ny-4))-1;
    }
    return y;
}

__device__ __host__ int isOnxn(int x)
{
    if(x==(NF2FFdistfromboundary))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnxp(int x)
{
    if(x==(nx-NF2FFdistfromboundary-1))
    { 
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnyp(int x,int y)
{
    if(y==(ny-NF2FFdistfromboundary-1)&&!isOnxn(x)&&!isOnxp(x))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnyn(int x, int y)
{
    if((y==(NF2FFdistfromboundary))&&!isOnxn(x)&&!(isOnxp(x)))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ int dgetCell(int x, int y, int size)
{
    return x +y*size;
}

__global__ void calculate_JandM(float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex*cjzxn,hipComplex*cjzyn,hipComplex*cmxyp,hipComplex*cmyxp,hipComplex*cmxyn,hipComplex*cmyxn)
{
    float freq = *f;
    int index = threadIdx.x+blockIdx.x*blockDim.x;// should launch 2*nx-8*NF2FFdistfromboundary+2*ny-4 threads. 
    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(isOnyp(x,y))
        {
            Ez = (dev_Ez[dgetCell(x,y+1,nx+1)]+dev_Ez[dgetCell(x,y,nx+1)])/2;
            float Hx = dev_Hx[dgetCell(x,y,nx)];
            cjzyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Hx*deltatime*cuexp((float)(-1)*j*(float)2*pi*freq*(float)(*timestep)*deltatime);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements

            cmxyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Ez*deltatime*cuexp((float)-1.0*j*(float)2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
        else if(isOnxp(x))//X faces override y faces at their intersections
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            float Hy = dev_Hy[dgetCell(x,y,nx)];

            cjzxp[index-(nx-2*NF2FFdistfromboundary-2)] += Hy*deltatime*cuexp(-1*j*2*pi*freq*(float)(*timestep)*(float)dt);//cjzxp and cmyxp have ny-2*NF2FFBound elements

            cmyxp[index-(nx-2*NF2FFdistfromboundary-2)] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*pi*freq*((float)(*timestep)+0.5)*(float)dt);// this is the discrete fourier transform, by the way.
        }
        else if(isOnyn(x,y))
        {  
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x,y+1,nx+1)])/2;
            float Hx=dev_Hx[dgetCell(x,y,nx)];

            cjzyn[index] += Hx*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt);  //cjzyn and cmxyn need to have nx-2*NF2FFbound-2 elements
            cmxyn[index] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
        else if(isOnxn(x))
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            cjzxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*dev_Hy[dgetCell(x,y,nx)]*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt); // cjzxn and cmyxn must have ny-2*NFdistfromboundary elements
            cmyxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Ez*(float)dt*cuexp(-1.0*j*2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
    }

}


__host__ __device__ float fwf(float timestep,float x, float y,float Phi_inc,float l)
{

    float ar;
    float ky, kx;//k hat
    sincosf(Phi_inc,&ky,&kx);

    ar = (float)timestep*dt-(float)t0-(1/(float)c0)*(ky*y*dx+kx*x*dy-l);
    //ar = timestep*dt-t0;

    //return exp(-1*(ar*ar)/(tau*tau));// gaussian pulse  argument is k dot r, 
    return exp(-1*ar*ar/(tau*tau));
    //return sin(2*PI*1e9*timestep*dt);
}

__global__ void H_field_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex)
{
    float buffer_Hy;
    float buffer_Hx;
    float Chez = (dt/dx)/(mu0);
    int x = threadIdx.x +blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if(x<nx&&y<nx)
    {
        buffer_Hy = dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        buffer_Hx = dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        if(x<ncells)
        {
            buffer_Hy= dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-x];
            dev_Psi_hyx[dgetCell(x,y,20)]=dev_bmx[ncells-1-x]*dev_Psi_hyx[dgetCell(x,y,20)]+dev_amx[ncells-1-x]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x,y,20)] ;
        } 
        if(x>=(nx-ncells))
        {
            buffer_Hy=dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[x-nx+ncells];
            dev_Psi_hyx[dgetCell(x-nx+20,y,2*ncells)]=dev_bmx[x-nx+ncells]*dev_Psi_hyx[dgetCell(x-nx+20,y,20)]+dev_amx[x-nx+ncells]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x-nx+20,y,20)];
        }
        if(y<ncells)
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-y];
            dev_Psi_hxy[dgetCell(x,y,nx)]=dev_bmy[ncells-1-y]*dev_Psi_hxy[dgetCell(x,y,nx)]+dev_amy[ncells-1-y]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y,nx)];  
        }
        if(y>=(ny-ncells))
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[y-ny+ncells];
            dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]=dev_bmy[y-ny+ncells]*dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]+dev_amy[y-ny+ncells]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y-nx+20,nx)];
        }
        //__syncthreads();
        if(isnan(buffer_Hx)) 
        {
            dev_Hx[dgetCell(x,y,nx)] = 0.0;
        }
        else 
        {
            dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        }

        if(isnan(buffer_Hy)) {
            dev_Hy[dgetCell(x,y,nx)] = 0.0;
        }
        else
        {
            dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
        }

        //dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        //dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
    }
}

__global__ void H_inc_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex)
{
    float buffer_Hy;
    float buffer_Hx;
    float Chez = (dt/dx)/(mu0);
    int x = threadIdx.x +blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if(x<nx&&y<nx)
    {
        buffer_Hy = dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        buffer_Hx = dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        if(x<ncells)
        {
            buffer_Hy= dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-x];
            dev_Psi_hyx[dgetCell(x,y,20)]=dev_bmx[ncells-1-x]*dev_Psi_hyx[dgetCell(x,y,20)]+dev_amx[ncells-1-x]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x,y,20)] ;
        } 
        if(x>=(nx-ncells))
        {
            buffer_Hy=dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[x-nx+ncells];
            dev_Psi_hyx[dgetCell(x-nx+20,y,2*ncells)]=dev_bmx[x-nx+ncells]*dev_Psi_hyx[dgetCell(x-nx+20,y,20)]+dev_amx[x-nx+ncells]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x-nx+20,y,20)];
        }
        if(y<ncells)
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-y];
            dev_Psi_hxy[dgetCell(x,y,nx)]=dev_bmy[ncells-1-y]*dev_Psi_hxy[dgetCell(x,y,nx)]+dev_amy[ncells-1-y]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y,nx)];  
        }
        if(y>=(ny-ncells))
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[y-ny+ncells];
            dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]=dev_bmy[y-ny+ncells]*dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]+dev_amy[y-ny+ncells]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y-nx+20,nx)];
        }
        //__syncthreads();
        if(isnan(buffer_Hx)) 
        {
            dev_Hx[dgetCell(x,y,nx)] = 0.0;
        }
        else 
        {
            dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        }

        if(isnan(buffer_Hy)) {
            dev_Hy[dgetCell(x,y,nx)] = 0.0;
        }
        else
        {
            dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
        }

        //dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        //dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
    }
}

__global__ void E_field_update(int *i,float*dev_Ez,float*dev_Hy,float*dev_Hx,float*dev_Psi_ezx,float*dev_aex,float*dev_aey,float*dev_bex,float*dev_bey,float*dev_Psi_ezy,float*kex,float*Cezhy,float*Cezhx,float*Ceze,float*Cezeip,float*Cezeic,float*Phi)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    //  int offset = x+y*blockDim.x*gridDim.x;
    float buffer_Ez;
    //float Ceh = (dt/dx)/(eps0);
    float Cezj = -dt/eps0;
    float length_offset;

    if(x<=nx&&y<=ny)
    {

        //if(x==0||x==nx||y==0||y==ny)
        if(x==nx||y==ny||x==0||y==0)
        {
            buffer_Ez=0.0;
        }
        else
        {
            if(isscattering)
            {

                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])
                    +Cezeic[dgetCell(x,y,nx+1)]*fwf((float)(*i)+0.5,x-nx/2,y-ny/2,*Phi,-breast_radius)
                    +Cezeip[dgetCell(x,y,nx+1)]*fwf((float)(*i)-0.5,x-nx/2,y-ny/2,*Phi,-breast_radius);

            }
            else
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)]);
                if(x==(int)(source_x)&&y==(int)(source_y))
                {
                    buffer_Ez=buffer_Ez + 100*Cezj*fwf((float)(*i),0,0,0,0);
                }
            }

            //if(x==((int)nx/2)&&y==((int)nx/2))
            //{
            //	//buffer_Ez=buffer_Ez + Cezj*dev_Jz[*i];
            //	buffer_Ez=buffer_Ez + Cezj*fwf((float)(*i),0,0,0,0);
            //}
            if(x<=ncells&&x!=0)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[ncells-x]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[ncells-x];
                dev_Psi_ezx[dgetCell(x-1,y-1,20)] = dev_bex[ncells-x]*dev_Psi_ezx[dgetCell(x-1,y-1,20)]+dev_aex[ncells-x]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)]);
                buffer_Ez += Cezhy[dgetCell(x,y,nx+1)]*dx*dev_Psi_ezx[dgetCell(x-1,y-1,2*ncells)];
            }
            if(x>=(nx-ncells)&&x!=nx)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[x-nx+ncells]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[x-nx+ncells];
                dev_Psi_ezx[dgetCell(x-nx+20,y-1,20)]=dev_bex[x-nx+ncells]*dev_Psi_ezx[dgetCell(x-nx+20,y-1,20)]+dev_aex[x-nx+ncells]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)]);
                buffer_Ez+=Cezhy[dgetCell(x,y,nx+1)]*dx*dev_Psi_ezx[dgetCell(x-nx+20,y-1,2*ncells)];
            }
            if(y<=ncells&&y!=0)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[ncells-y]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[ncells-y];
                dev_Psi_ezy[dgetCell(x-1,y-1,nx)]=dev_bey[(ncells-y)]*dev_Psi_ezy[dgetCell(x-1,y-1,nx)]+dev_aey[(ncells-y)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhx[dgetCell(x,y,nx+1)]*dy*dev_Psi_ezy[dgetCell(x-1,y-1,nx)];
            }
            if(y>=(ny-ncells)&&y!=ny)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[y-ny+ncells]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[y-ny+ncells];
                dev_Psi_ezy[dgetCell(x-1,y-ny+20,nx)]=dev_bey[y-ny+ncells]*dev_Psi_ezy[dgetCell(x-1,y-ny+20,nx)]+dev_aey[y-ny+ncells]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhx[dgetCell(x,y,nx+1)]*dy*dev_Psi_ezy[dgetCell(x-1,y-ny+20,nx)];
            }
        }
        //		unsigned char green = 128+127*buffer_Ez/0.4;
        /*ptr[offset].x = 0;
          ptr[offset].y = green;
          ptr[offset].z = 0;
          ptr[offset].w = 255;*///OpenGL stuff

        //__syncthreads();
        if(isnan(buffer_Ez)) {
            dev_Ez[dgetCell(x,y,nx+1)] = 0.0;
        }
        else {
            dev_Ez[dgetCell(x,y,nx+1)] = buffer_Ez;
        }
        //dev_Ez[dgetCell(x,y,nx+1)] = buffer_Ez;
    }

}

__global__ void Field_reset(float* Ez, float* Hy, float* Hx, float* Psi_ezy,float* Psi_ezx,float* Psi_hyx,float* Psi_hxy,hipComplex*cjzyn,hipComplex*cjzxp,hipComplex*cjzyp,hipComplex*cjzxn,hipComplex*cmxyn,hipComplex*cmyxp,hipComplex*cmxyp,hipComplex*cmyxn)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockDim.y*blockIdx.y;
    int index = x + y*blockDim.x*gridDim.x;
    if(x<=ncells&&x!=0)
    {
        Psi_ezx[dgetCell(x-1,y-1,20)] =0;
    }
    if(x>=(nx-ncells)&&x!=nx)
    {
        Psi_ezx[dgetCell(x-nx+20,y-1,20)]=0;
    }
    if(y<=ncells&&y!=0)
    {
        Psi_ezy[dgetCell(x-1,y-1,nx)]=0;
    }
    if(y>=(ny-ncells)&&y!=ny)
    {
        Psi_ezy[dgetCell(x-1,y-ny+20,nx)]=0;
    }
    if(x<ncells)
    {

        Psi_hyx[dgetCell(x,y,20)]=0;
    } 
    if(x>=(nx-ncells))
    {
        Psi_hyx[dgetCell(x-nx+20,y,2*ncells)]=0.0;
    }
    if(y<ncells)
    {
        Psi_hxy[dgetCell(x,y,nx)]=0.0;
    }
    if(y>=(ny-ncells))
    {
        Psi_hxy[dgetCell(x,y-ny+20,nx)]=0.0;
    }
    if(x<=nx&&y<=ny)
    {
        Ez[dgetCell(x,y,nx+1)] = 0.0;
    }
    if(x<nx&&y<ny)
    {
        Hy[dgetCell(x,y,nx)] = 0.0;
        Hx[dgetCell(x,y,nx)] = 0.0;
    }

    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(index<size_cjzy)
        {
            cjzyp[index] = hipComplex(0,0);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements
            cjzyn[index] = hipComplex(0,0);
            cmxyp[index] = hipComplex(0,0);
            cmxyn[index] = hipComplex(0,0);
        }
        if(index<size_cjzx)
        {
            cjzxp[index] = hipComplex(0,0);
            cjzxn[index] = hipComplex(0,0);
            cmyxp[index] = hipComplex(0,0);
            cmyxn[index] = hipComplex(0,0);
        }

    }


}

__global__ void E_inc_update(int *i,float*dev_Hy_inc,float*dev_Hx_inc,float*dev_Psi_ezx_inc,float*dev_aex,float*dev_aey,float*dev_bex,float*dev_bey,float*dev_Psi_ezy_inc,float*kex,float*dev_Ezip,float*dev_Ezic,float*Phi)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    //	int offset = x+y*blockDim.x*gridDim.x;
    float buffer_Ez;
    //float Ceh = (dt/dx)/(eps0);
    float Cezj = -dt/eps0;
    float Ceze = 1;
    float Cezhy = (dt/(dx*eps0));

    if(x<=nx&&y<=ny)
    {

        //if(x==0||x==nx||y==0||y==ny)
        if(x==nx||y==ny||x==0||y==0)
        {
            buffer_Ez=0.0;
        }
        else
        {
            buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])
                -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)]);

            if(x==((int)source_x)&&y==(int)(source_y))
            {
                //buffer_Ez=buffer_Ez + Cezj*dev_Jz[*i];
                buffer_Ez=buffer_Ez + 100*Cezj*fwf((float)(*i),0,0,0,0);
            }
            if(x<=ncells&&x!=0)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[ncells-x]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[ncells-x];
                dev_Psi_ezx_inc[dgetCell(x-1,y-1,20)] = dev_bex[ncells-x]*dev_Psi_ezx_inc[dgetCell(x-1,y-1,20)]+dev_aex[ncells-x]*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)]);
                buffer_Ez += Cezhy*dx*dev_Psi_ezx_inc[dgetCell(x-1,y-1,2*ncells)];
            }
            if(x>=(nx-ncells)&&x!=nx)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[x-nx+ncells]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[x-nx+ncells];
                dev_Psi_ezx_inc[dgetCell(x-nx+20,y-1,20)]=dev_bex[x-nx+ncells]*dev_Psi_ezx_inc[dgetCell(x-nx+20,y-1,20)]+dev_aex[x-nx+ncells]*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)]);
                buffer_Ez+=Cezhy*dx*dev_Psi_ezx_inc[dgetCell(x-nx+20,y-1,2*ncells)];
            }
            if(y<=ncells&&y!=0)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[ncells-y]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[ncells-y];
                dev_Psi_ezy_inc[dgetCell(x-1,y-1,nx)]=dev_bey[(ncells-y)]*dev_Psi_ezy_inc[dgetCell(x-1,y-1,nx)]+dev_aey[(ncells-y)]*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhy*dy*dev_Psi_ezy_inc[dgetCell(x-1,y-1,nx)];
            }
            if(y>=(ny-ncells)&&y!=ny)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[y-ny+ncells]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[y-ny+ncells];
                dev_Psi_ezy_inc[dgetCell(x-1,y-ny+20,nx)]=dev_bey[y-ny+ncells]*dev_Psi_ezy_inc[dgetCell(x-1,y-ny+20,nx)]+dev_aey[y-ny+ncells]*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhy*dy*dev_Psi_ezy_inc[dgetCell(x-1,y-ny+20,nx)];
            }
        }
        dev_Ezip[dgetCell(x,y,nx+1)] = dev_Ezic[dgetCell(x,y,nx+1)];
        dev_Ezic[dgetCell(x,y,nx+1)] = buffer_Ez;
    }

}

float calc_radiated_power(hipComplex *cjzxp,hipComplex *cjzyp,hipComplex *cjzxn,hipComplex *cjzyn,hipComplex *cmxyp,hipComplex *cmyxp,hipComplex *cmxyn,hipComplex *cmyxn)
{
    int indexofleg1 = nx-2*NF2FFdistfromboundary-2;
    int indexofleg2 = nx+ny-4*NF2FFdistfromboundary-2;
    int indexofleg3 = 2*nx+ny-6*NF2FFdistfromboundary-4;
    int maxindex = 2*nx-8*NF2FFdistfromboundary+2*ny-4;
    int index;
    hipComplex cjz(0,0);
    hipComplex power = 0;

    for(index = 0; index<indexofleg1;index++)
    {   cjz = hipComplex(cjzyn[index].r,-1.0*cjzyn[index].i);//conjugation
        //z x x = y dot -y = -1
        power+=-1.0*cjz*cmxyn[index]*dx;// the negative one comes from the dot product between JxM and the n hat vector
    }
    for(index = indexofleg1; index<indexofleg2;index++)
    {
        cjz = hipComplex(cjzxp[index-indexofleg1].r,-1.0*cjzxp[index-indexofleg1].i);//making the conjugate
        // z cross y = -x dot x = -1
        power+= -1.0*cjz*cmyxp[index-indexofleg1]*dy;//positive x unit normal vector
    }
    for(index = indexofleg2;index<indexofleg3;index++)
    {
        // z cross x = y dot y = 1
        cjz = hipComplex(cjzyp[index-indexofleg2].r,-1.0*cjzyp[index-indexofleg2].i);
        power+= cjz*cmxyp[index-indexofleg2]*dx;//postive y unit normal vector
    }
    for(index = indexofleg3;index<maxindex;index++)
    {
        // z cross y = -x dot -x = 1 
        cjz = hipComplex(cjzxn[index-indexofleg3].r,-1.0*cjzxn[index-indexofleg3].i);
        power += cjz*cmyxn[index-indexofleg3]*dy;// negative x hat n vector
    }
    float realpower = power.r;
    realpower *= 0.5;
    return realpower;
}

float calc_incident_power(float freq)
{
    return (0.5/eta0)*pow(tau*sqrt(PI)*exp(-tau*tau*2*PI*freq*2*PI*freq/4),2);// just gonna assume gaussian pulse.  This is the fourier transform of the gaussian pulse.
}

__global__ void calculate_JandM_total(float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex*cjzxn,hipComplex*cjzyn,hipComplex*cmxyp,hipComplex*cmyxp,hipComplex*cmxyn,hipComplex*cmyxn,float*dev_Ezic,float*dev_Ezip,float*dev_Hx_inc,float*dev_Hy_inc)
{
    float freq = *f;
    int index = threadIdx.x+blockIdx.x*blockDim.x;// should launch 2*nx-8*NF2FFdistfromboundary+2*ny-4 threads. 
    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(isOnyp(x,y))
        {
            Ez = (dev_Ez[dgetCell(x,y+1,nx+1)]+dev_Ez[dgetCell(x,y,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x,y+1,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x,y+1,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hx = dev_Hx[dgetCell(x,y,nx)] + dev_Hx_inc[dgetCell(x,y,nx)];
            cjzyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Hx*deltatime*cuexp((float)(-1)*j*(float)2*pi*freq*(float)(*timestep)*deltatime);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements
            cmxyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Ez*deltatime*cuexp((float)-1.0*j*(float)2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
        else if(isOnxp(x))//X faces override y faces at their intersections
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x+1,y,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x+1,y,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hy = dev_Hy[dgetCell(x,y,nx)] + dev_Hy_inc[dgetCell(x,y,nx)];

            cjzxp[index-(nx-2*NF2FFdistfromboundary-2)] += Hy*deltatime*cuexp(-1*j*2*pi*freq*(float)(*timestep)*(float)dt);//cjzxp and cmyxp have ny-2*NF2FFBound elements

            cmyxp[index-(nx-2*NF2FFdistfromboundary-2)] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*pi*freq*((float)(*timestep)-0.5)*(float)dt);// this is the discrete fourier transform, by the way.
        }
        else if(isOnyn(x,y))
        {  
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x,y+1,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x,y+1,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x,y+1,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hx=dev_Hx[dgetCell(x,y,nx)]+dev_Hx_inc[dgetCell(x,y,nx)];

            cjzyn[index] += Hx*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt);	//cjzyn and cmxyn need to have nx-2*NF2FFbound-2 elements
            cmxyn[index] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
        else if(isOnxn(x))
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x+1,y,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x+1,y,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hy = dev_Hy[dgetCell(x,y,nx)] + dev_Hy_inc[dgetCell(x,y,nx)];
            cjzxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Hy*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt); // cjzxn and cmyxn must have ny-2*NFdistfromboundary elements
            cmyxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Ez*(float)dt*cuexp(-1.0*j*2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
    }

}

__host__ __device__ int getOptimizationCell(int x, int y)
{
    int x_coord,y_coord;
    x_coord = (x-(nx/2-(int)(breast_radius/dx)))/(2*breast_radius/(9*dx));
    y_coord = (y-(ny/2-breast_radius/dy))/(2*breast_radius/(9*dy));//the optimization space is 216 FDTD cells wide and high. //The optimization space is split into 25 by 25 optimization cells. 
    //each optimization cell has 24 by 24 FDTD cells within it. That's what the 108, 24 and 25 are about.  
    return x_coord+9*y_coord;//The max return should be, 9*9-1, hopefully.
}

void N2FPostProcess (float* D,float f, hipComplex *N,hipComplex *L,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex *cjzxn,hipComplex *cjzyn,hipComplex *cmxyp,hipComplex *cmyxp,hipComplex *cmxyn,hipComplex *cmyxn)
{
    int indexofleg1 = nx-2*NF2FFdistfromboundary-2;
    int indexofleg2 = nx+ny-4*NF2FFdistfromboundary-2;
    int indexofleg3 = 2*nx+ny-6*NF2FFdistfromboundary-4;
    int maxindex = 2*nx-8*NF2FFdistfromboundary+2*ny-4;
    int x,y;

    float rhoprime;
    float Psi;
    int Phi_index;
    hipComplex  Mphi(0,0);
    float Phi;


    float k = 2*PI*f/c0;
    hipComplex  negativeone(-1.0,0.0);
    int index = 0;
    hipComplex jcmpx(0,1);
    //float Prad = calc_radiated_power(cjzxp,cjzyp,cjzxn,cjzyn,cmxyp,cmyxp,cmxyn,cmyxn);
    float Prad = calc_incident_power(f);
    //std::cout<<"Prad = "<<Prad<<std::endl;
    float flx, fly;
    for(Phi_index = 0; Phi_index<numberofobservationangles;Phi_index++)
    {
        Phi = 2*PI/numberofobservationangles*(float)Phi_index;
        for(index = 0;index<indexofleg1;index++)
        {

            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x;//float x
            fly = (float)y + 0.5;
            rhoprime = sqrt(pow((dx*((-1.0*(float)nx/2)+1+flx)),2)+pow((dy*(-1.0*(float)ny/2+1+fly)),2));
            Psi = atan2(-1*((float)ny/2)+1+fly,-1*((float)nx/2)+1+flx)-Phi;
            N[Phi_index]+=-1.0*cjzyn[index]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;
            L[Phi_index]+=-1.0*sin(Phi)*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*cmxyn[index]*dx;//Lphi = 

        }
        for(index = indexofleg1;index<indexofleg2;index++)
        {

            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x+0.5;
            fly = (float)y;
            rhoprime = sqrt(pow((dx*(((float)nx/2)-1-flx)),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2(-1*((float)ny/2)+1+fly,(-1*((float)nx/2)+1+flx))-Phi;
            N[Phi_index]+=-1.0*cjzxp[index-indexofleg1]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
            L[Phi_index]+=cos(Phi)*cmyxp[index-indexofleg1]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;//L_phi = -Lxsin(phi)+Lycos(Phi) here we only have Ly
        }
        for(index=indexofleg2;index<indexofleg3;index++)
        {
            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x;
            fly = (float)y + 0.5;
            rhoprime = sqrt(pow((dx*(((float)nx/2)-1-flx)),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2((-1*(float)ny/2+1+fly),(-1*((float)nx/2)+1+flx))-Phi;
            N[Phi_index]+=-1.0*cjzyp[index-indexofleg2]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;
            L[Phi_index]+=-1.0*sin(Phi)*cmxyp[index-indexofleg2]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;//
        }
        for(index = indexofleg3;index<maxindex;index++)
        {
            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x+0.5;
            fly = (float)y;
            rhoprime = sqrt(pow(dx*(((float)nx/2)-1-flx),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2(-1*((float)ny/2)+1+fly,-1*(float)nx/2+1+flx)-Phi;
            N[Phi_index]+=-1.0*cjzxn[index-indexofleg3]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
            L[Phi_index]+= cos(Phi)*cmyxn[index-indexofleg3]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
        }
        D[Phi_index] = (k*k*cuabs(L[Phi_index]+(float)eta0*N[Phi_index])*cuabs(L[Phi_index]+(float)eta0*N[Phi_index])/((float)8*(float)PI*(float)eta0*Prad*33.329));//why 33.329?  I dunno, something is probably wrong with Prad.
    }

}

float fitness(float* D,int max_index, float* measurement)
{
    float fit = 0;
    for(int i =0;i<max_index;i++)
    {
        fit -= pow((measurement[i]-D[i]),2)/(numberofexcitationangles*pow(measurement[i],2));
    }

    return fit;
}

//static void draw_func(void){
//	glDrawPixels(nx,ny,GL_RGBA,GL_UNSIGNED_BYTE,0);
//	glutSwapBuffers;
//}

using namespace std;

void Ceze_init(float * eps_r_z, float* sig_e_z, float* Ceze);
void Cezhy_init(float* eps_r_z, float* sigma_e_z,float*Cezhy,float*kex);
void Cezhx_init(float* eps_r_z,float*sigma_e_z,float*Cezhx,float*kex);
void eps_r_z_init(float * eps_r_z,const vector<float> &argument);
void sigma_e_z_init(float *sigma_e_z,float*sigma_e_pml,const vector<float> &argument);
void Cezj_init(float*eps_r_z,float*sigma_e_z,float*Cezj);
void Ez_init(float*Ez);
void Ey_init(float*Ey);
//void Jz_init(float*Jz);
void Chxh_init(float*mu_r_x,float*sigma_m_x,float*Chxh);
void Chxez_init(float*mu_r_x,float*sigma_m_x,float*Chxez);
//void Chxm_init(float*mu_r_x,float*sigma_m_x,float*Chxm);
void Chyh_init(float*mu_r_y,float*sigma_m_y,float*Chyh);
void Chyez_init(float*mu_r_y,float*sigma_m_y,float*Chyez);
//void Chym_init(float*mu_r_y,float*sigma_m_y,float*Chym);
void Hy_init(float*Hy);
void Hx_init(float*Hx);
void My_init(float*My);
void Mx_init(float*Mx);
void mu_r_y_init(float*mu_r_y);
void mu_r_x_init(float*mu_r_x);
void sigma_m_y_init(float*sigma_m_y_init);
void sigma_m_x_init(float*sigma_m_x_init);
int getCell(int x,int y,int size);
void Jz_waveform(float * time,float*Jz_impressed);
void waveform_time_init(float*time1);
float* Make2DfloatArray(int arraySizeX, int arraySizeY);
void C_Psi_ezy_init(float *C_Psi_ezy,float*Cezhx);
void C_Psi_ezx_init(float* C_Psi_ezx,float*Cezhy);
void C_Psi_hyx_init(float*C_Psi_hyx,float*Chyez);
void C_psi_hxy_init(float *C_Psi_hxy,float*Chxez);

void aex_init(float*aex,float*sigma_e_pml,float*kex,float*alpha_e_x,float*bex);
void bex_init(float*bex ,float*sigma_e_pml,float*kex,float*alpha_e_x);   
void bey_init(float*bey,float*sigma_e_pml,float*key,float*alpha_e_y);
void amy_init(float*amy,float*sigma_m_pml,float*kmy,float*alpha_m_y,float*bmy);
void bmy_init(float*bmy,float*sigma_m_pml,float*kmy,float*alpha_m_y);
void amx_init(float*amx,float*sigma_m_pml,float*kmx,float*alpha_m_x,float*bmx);
void bmx_init(float*bmx,float*sigma_m_pml,float*kmx,float*alpha_m_x);
void alpha_e_init(float*alpha_e);
void alpha_m_init(float*alpha_e,float*alpha_m);
void k_e_init(float*k);
void k_m_init(float*k);
void sigma_e_pml_init(float* sigma_e_pml);
void sigma_m_pml_init(float*sigma_m_pml,float*sigma_e_pml);
void Psi_ezy_init(float*Psi_ezy);
void Psi_ezx_init(float*Psi_ezx);
void Psi_hyx_init(float*Psi_hyx);
void Psi_hxy_init(float*Psi_hxy);
void CJ_Init(hipComplex * cjzyn,int size);
__global__ void scattered_parameter_init(float*eps_r_z,float*sigma_e_z,float*Cezeic,float*Cezeip);

double FDTD_GPU(const vector<double> &arguments)
{
    // BMP Output_Image;
    //BMP Scattered_Field_snapshot;
    // Output_Image.SetSize((nx+1),(ny+1));
    // Output_Image.SetBitDepth(16);
    //Scattered_Field_snapshot.SetSize((nx+1),(ny+1));
    //Scattered_Field_snapshot.SetBitDepth(16);
    //RGBApixel Temp;
    // string outputfilename;

    cout << "calculating FDTD GPU" << endl;

    hipSetDevice(0);

    vector<float> image;
    for (int lerp = 0; lerp < 81; lerp++) {     //This is setting the material parameters of the optimization cells.
        image.push_back((float)arguments.at(lerp));
        //image.push_back(10);
    }

    for (int lerp = 81; lerp < 81 * 2; lerp++) {
        image.push_back((float)arguments.at(lerp));
        // image.push_back(0);
    }

    //GLuint bufferObj;
    //cudaGraphicsResource *resource;
    hipError_t error;
    //int dev;
    //cudaDeviceProp prop;
    //memset(&prop,sizeof(cudaDeviceProp),sizeof(cudaDeviceProp));
    //prop.major = 1;
    //prop.minor = 1;
    //cudaChooseDevice(&dev,&prop);
    //	cudaGLSetGLDevice(dev);
    /*glutInit(&argc,argv);
      glewInit();
      glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);  
      glutInitWindowSize(nx,ny);
      glutCreateWindow("bitmap");
      glGenBuffers(1,&bufferObj);
      glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
      glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, nx*ny*4,NULL,GL_DYNAMIC_DRAW_ARB);
      cudaGraphicsGLRegisterBuffer(&resource,bufferObj,cudaGraphicsMapFlagsNone);*/
    //uchar4* devPtr;
    //size_t size;
    //cudaGraphicsMapResources(1,&resource,NULL);
    //cudaGraphicsResourceGetMappedPointer((void**)&devPtr,&size,resource);

    float   *Ceze, *Cezhy, *Cezhx, *dev_Cezeic, *dev_Cezeip, *Ez, *eps_r_z, *sigma_e_z, *Hy, *Hx,
            *kex, *aex, *bex, *amx, *bmx, *alpha_e, *alpha_m, *sigma_e_pml, *sigma_m_pml,
            *Psi_ezy, *Psi_ezx, *Psi_hyx, *Psi_hxy, *kmx; //*Cezj later if using loop current source
    float* dev_sigma_e_z,*dev_eps_r_z;
    float freq = center_freq;
    float *dev_freq,*D,*dev_Phi;


    hipComplex *cjzxp,*cjzyp,*cjzxn,*cjzyn,*cmxyp,*cmyxp,*cmxyn,*cmyxn;
    hipComplex *hcjzxp,*hcjzyp,*hcjzxn,*hcjzyn,*hcmxyp,*hcmyxp,*hcmxyn,*hcmyxn;
    hipComplex *L,*N;
    int grid_x = int(ceil((float)nx/22));
    int grid_y = int(ceil((float)ny/22));
    dim3 grid(grid_x,grid_y);
    dim3 block(22,22);



    Ceze = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    Cezhy = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    Cezhx = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    //Cezj = (float*)malloc(sizeof(float)*(1+nx)*(1+ny)); // if using loop current source
    Ez = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    eps_r_z =  (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    sigma_e_z = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    D = (float*)malloc(sizeof(float)*numberofexcitationangles*numberofobservationangles);//D = (float*)malloc(numberofobservationangles*sizeof(float));
    Hy=(float*)malloc(sizeof(float)*nx*ny);
    Hx=(float*)malloc(sizeof(float)*nx*ny);
    kex = (float*)malloc(sizeof(float)*10);
    kmx = (float*)malloc(sizeof(float)*10);
    aex=(float*)malloc(sizeof(float)*10);
    bex=(float*)malloc(sizeof(float)*10);
    amx=(float*)malloc(sizeof(float)*10);
    bmx=(float*)malloc(sizeof(float)*10);
    alpha_e=(float*)malloc(sizeof(float)*10);
    alpha_m=(float*)malloc(sizeof(float)*10);
    sigma_e_pml=(float*)malloc(sizeof(float)*10);
    sigma_m_pml=(float*)malloc(sizeof(float)*10);
    Psi_ezy=(float*)malloc(sizeof(float)*ny*20);
    Psi_ezx=(float*)malloc(sizeof(float)*nx*20);
    Psi_hyx=(float*)malloc(sizeof(float)*ny*20);
    Psi_hxy=(float*)malloc(sizeof(float)*nx*20);
    hcjzyp = (hipComplex*)malloc(sizeof(hipComplex )*size_cjzy);
    hcjzyn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcjzxp = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcjzxn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcmxyn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcmxyp = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcmyxp = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcmyxn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    L  = (hipComplex*)malloc(sizeof(hipComplex)*size_NF2FF_total);
    N  = (hipComplex*)malloc(sizeof(hipComplex)*size_NF2FF_total);

    eps_r_z_init(eps_r_z,image);
    sigma_e_z_init(sigma_e_z,sigma_e_pml,image);
    //float*time1;
    //time1 = (float*)malloc(sizeof(float)*number_of_time_steps);
    Ceze_init(eps_r_z,sigma_e_z,Ceze);	
    k_e_init(kex);
    k_m_init(kmx);
    Cezhy_init(eps_r_z,sigma_e_z,Cezhy,kex);
    Cezhx_init(eps_r_z,sigma_e_z,Cezhx,kex);
    sigma_e_pml_init(sigma_e_pml);
    sigma_m_pml_init(sigma_m_pml,sigma_e_pml);
    alpha_e_init(alpha_e);
    alpha_m_init(alpha_e,alpha_m);
    bex_init(bex ,sigma_e_pml,kex,alpha_e);
    aex_init(aex,sigma_e_pml,kex,alpha_e,bex);
    bmx_init(bmx,sigma_m_pml,kmx,alpha_m);
    amx_init(amx,sigma_m_pml,kmx,alpha_m,bmx);

    for (int i = 0; i < 10; i++) {
        cout<<"kex["<<i<<"]= "<<kex[i]<<endl;
        cout<<"kmx["<<i<<"]= "<<kmx[i]<<endl;
        cout<<"aex["<<i<<"]= "<<aex[i]<<endl;
        cout<<"amx["<<i<<"]= "<<amx[i]<<endl;
        cout<<"bex["<<i<<"]= "<<bex[i]<<endl;
        cout<<"bmx["<<i<<"]= "<<bmx[i]<<endl;
        cout<<"alpha_e = "<<alpha_e[i]<<endl;
        cout<<"alpha_m = "<<alpha_m[i]<<endl;
        cout << endl;
    }

    //Jz_init(Jz);
    //system("pause");   
    //FILE* file = fopen("results.txt", "w");

    //float*Jz_impressed = (float*)malloc(sizeof(float)*number_of_time_steps);
    //waveform_time_init(time1);
    //Jz_waveform(time1,Jz_impressed);

    //int source_position_index_x = int(nx*source_position/domain_size)+1;

    //	int source_position_index_y = int(ny*source_position/domain_size)+1;
    float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_bex,*dev_aex,*dev_bmx,*dev_amx,*dev_kex,*dev_kmx;//dev_Cezj if using loop current source
    float *dev_Ez,*dev_Hy,*dev_Hx;

    float*dev_Psi_ezy,*dev_Psi_ezx,*dev_Psi_hyx,*dev_Psi_hxy;

    hipMalloc(&dev_eps_r_z,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_sigma_e_z,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezeic,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezeip,sizeof(float)*(nx+1)*(ny+1));
    hipMemcpy(dev_eps_r_z,eps_r_z,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_sigma_e_z,sigma_e_z,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    scattered_parameter_init<<<grid,block>>>(dev_eps_r_z,dev_sigma_e_z,dev_Cezeic,dev_Cezeip);
    //float *Cezeic = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    // float *Cezeip = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    //cudaMemcpy(Cezeic,dev_Cezeic,sizeof(float)*(nx+1)*(ny+1),cudaMemcpyDeviceToHost);
    //cudaMemcpy(Cezeip,dev_Cezeip,sizeof(float)*(nx+1)*(ny+1),cudaMemcpyDeviceToHost);
    float radius;


    hipMalloc(&dev_Phi,sizeof(float));
    hipMalloc(&dev_kex,sizeof(float)*10);
    hipMalloc(&dev_kmx,sizeof(float)*10);
    hipMalloc(&dev_Ez,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Hy,sizeof(float)*nx*ny);
    hipMalloc(&dev_freq ,sizeof(float));
    hipMalloc(&dev_Hx,sizeof(float)*nx*ny);
    hipMalloc(&dev_Psi_ezy,sizeof(float)*20*(nx+1));
    hipMalloc(&dev_Psi_ezx,sizeof(float)*20*(ny+1));
    hipMalloc(&dev_Psi_hyx,sizeof(float)*20*(ny));
    hipMalloc(&dev_Psi_hxy,sizeof(float)*20*(nx));

    hipMalloc(&cjzxp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzxn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxn,sizeof(hipComplex)*size_NF2FF_total);

    hipMemcpy(dev_freq,&freq,sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&dev_bex,sizeof(float)*10);
    hipMalloc(&dev_bmx,sizeof(float)*10);
    hipMalloc(&dev_amx,sizeof(float)*10);
    hipMalloc(&dev_aex,sizeof(float)*10);
    hipMalloc(&dev_Ceze,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezhy,sizeof(float)*(nx+1)*(ny+1));

    //cudaMalloc(&dev_Cezj,sizeof(float)*(nx+1)*(ny+1)); if using current source

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error after cuda Mallocs: %s\n",hipGetErrorString(error));
    }

    Field_reset<<<grid,block>>>(dev_Ez, dev_Hy, dev_Hx, dev_Psi_ezy, dev_Psi_ezx, dev_Psi_hyx, dev_Psi_hxy,cjzyn,cjzxp,cjzyp,cjzxn,cmxyn,cmyxp,cmxyp,cmyxn);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error after field reset: %s\n",hipGetErrorString(error));
    }
    //Field_reset is also good for making all these values zero.


    hipMemcpy(dev_kex,kex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_kmx,kmx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_aex,aex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_bex,bex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_bmx,bmx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_amx,amx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_Ceze,Ceze,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_Cezhy,Cezhy,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("Error after cuda Memcpy: %s\n",hipGetErrorString(error));
    }

    int*dev_i;
    hipMalloc(&dev_i,sizeof(int));
    float test_Ez;

    dim3 gridNF2FF((int)ceil(size_NF2FF_total/512.0));
    dim3 blockNF2FF(512);

    float test_Ez_2;
    float Phi;
    //Output_Image.SetBitDepth(16);

    //for(int x = 0;x<nx+1;x++)// This double loop makes an image of the target.  Delete when using this as forward solver.
    //	for(int y = 0; y<ny+1;y++)
    //	{
    //		{
    //			Temp.Green = 0;
    //			if(eps_r_z[getCell(x,y,nx+1)] >15)
    //			{
    //			Temp.Red = 255;
    //			Temp.Blue = 0;
    //			}
    //			else
    //			{
    //			Temp.Blue = 150;
    //			Temp.Red = 0;
    //			}
    //		}
    //		Output_Image.SetPixel(x,y,Temp);
    //	}
    //Output_Image.WriteToFile("Permittivity_map_measurement.bmp");
    /* The calculation part! */

    //ofstream measurement_data;
    //measurement_data.open("measurement_data.txt");

    for(int Phi_index = 0; Phi_index<numberofexcitationangles; Phi_index++)
    {

        Phi = Phi_index*2*PI/numberofexcitationangles;
        hipMemcpy(dev_Phi,&Phi,sizeof(float),hipMemcpyHostToDevice);

        for(int i=0;i<number_of_time_steps;i++)
        {

            hipMemcpy(dev_i,&i,sizeof(int),hipMemcpyHostToDevice);
            H_field_update<<<grid,block>>>(dev_Hy,dev_Hx,dev_Ez,dev_bmx,dev_Psi_hyx,dev_amx,dev_bmx,dev_amx,dev_Psi_hxy,dev_kmx);
            E_field_update<<<grid,block>>>(dev_i,dev_Ez,dev_Hy,dev_Hx,dev_Psi_ezx,dev_aex,dev_aex,dev_bex,dev_bex,dev_Psi_ezy,dev_kex,dev_Cezhy,dev_Cezhy,dev_Ceze,dev_Cezeip,dev_Cezeic,dev_Phi);
            calculate_JandM<<<gridNF2FF,blockNF2FF>>>(dev_freq, dev_i,dev_Ez,dev_Hy,dev_Hx,cjzxp,cjzyp,cjzxn,cjzyn,cmxyp,cmyxp,cmxyn,cmyxn);

        }

        hipMemcpy(hcjzyn,cjzyn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcjzxp,cjzxp,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
        hipMemcpy(hcjzyp,cjzyp,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcjzxn,cjzxn,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
        hipMemcpy(hcmxyn,cmxyn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcmyxp,cmyxp,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
        hipMemcpy(hcmxyp,cmxyp,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcmyxn,cmyxn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);

        CJ_Init(L,size_NF2FF_total);
        CJ_Init(N,size_NF2FF_total);

        N2FPostProcess(D + Phi_index*numberofobservationangles, freq,N,L,hcjzxp,hcjzyp,hcjzxn,hcjzyn,hcmxyp,hcmyxp,hcmxyn,hcmyxn);
        //notice the D + Phi_index*numberofobservationangles. D is in total 4*numberofobservaion angles, so that's how we fill them in sequentially.

        //for(int i = 0;i<numberofobservationangles;i++)  // This is for recording simulated measured data
        //{
        //measurement_data<<*(D+Phi_index*numberofobservationangles+i)<<" , ";
        //cout<<*(D+Phi_index*numberofobservationangles+i)<<endl;
        //}

        //measurement_data<<endl;
        Field_reset<<<grid,block>>>(dev_Ez, dev_Hy, dev_Hx, dev_Psi_ezy, dev_Psi_ezx, dev_Psi_hyx, dev_Psi_hxy,cjzyn,cjzxp,cjzyp,cjzxn,cmxyn,cmyxp,cmxyp,cmyxn);

    }






    float measurement[numberofobservationangles*numberofexcitationangles] = {0.544912 , 0.518606 , 0.439233 , 0.330533 , 0.219116 , 0.135115 , 0.0923969 , 0.0774134 , 0.0740459 , 0.0739238 , 0.0660047 , 0.0465372 , 0.0248307 , 0.00913681 , 0.00186162 , 0.0038402 , 0.0130785 , 0.0238094 , 0.0312918 , 0.035705 , 0.0388307 , 0.039513 , 0.0368443 , 0.0338221 , 0.0324815 , 0.0305907 , 0.0270149 , 0.0239178 , 0.0224438 , 0.021849 , 0.0217346 , 0.0222152 , 0.023146 , 0.0245181 , 0.0267161 , 0.0286964 , 0.0276803 , 0.0235098 , 0.0197177 , 0.0183168 , 0.0196998 , 0.0261493 , 0.0375584 , 0.0479223 , 0.0511598 , 0.0461443 , 0.035713 , 0.0249863 , 0.0203708 , 0.0260456 , 0.0395441 , 0.054163 , 0.0660136 , 0.0763823 , 0.0935922 , 0.132053 , 0.201299 , 0.299247 , 0.410792 , 0.504467 , 
        0.0490085 , 0.0278468 , 0.0123693 , 0.00899709 , 0.0196632 , 0.0401112 , 0.0623734 , 0.0809561 , 0.096057 , 0.113814 , 0.145125 , 0.200388 , 0.283438 , 0.386362 , 0.486139 , 0.549594 , 0.547993 , 0.475775 , 0.358033 , 0.230962 , 0.118935 , 0.039843 , 0.00700227 , 0.0112335 , 0.0300356 , 0.0494414 , 0.0605159 , 0.0585777 , 0.0503323 , 0.045704 , 0.0474064 , 0.0523123 , 0.0558987 , 0.0545722 , 0.0475098 , 0.0366045 , 0.0248037 , 0.0155752 , 0.0115322 , 0.0127167 , 0.0176523 , 0.0243556 , 0.0310764 , 0.037444 , 0.0432292 , 0.0469609 , 0.0471761 , 0.0435653 , 0.0369347 , 0.0293987 , 0.0235478 , 0.0206039 , 0.020754 , 0.0247748 , 0.0336772 , 0.047007 , 0.0618746 , 0.0734482 , 0.0763332 , 0.0674785 , 
        0.0463129 , 0.0448933 , 0.0398454 , 0.0319834 , 0.0239428 , 0.0174267 , 0.0129155 , 0.0116624 , 0.0154122 , 0.0247183 , 0.0376821 , 0.0494142 , 0.0552493 , 0.0544909 , 0.0501016 , 0.0466044 , 0.047395 , 0.0522298 , 0.0576919 , 0.0588555 , 0.0504011 , 0.0311956 , 0.0107719 , 0.00755493 , 0.0394798 , 0.116099 , 0.232324 , 0.36478 , 0.478314 , 0.541685 , 0.541186 , 0.484009 , 0.391878 , 0.291105 , 0.204554 , 0.145352 , 0.113254 , 0.0973423 , 0.0835717 , 0.0637299 , 0.0397899 , 0.0189781 , 0.00814281 , 0.0118845 , 0.0291142 , 0.0513172 , 0.0680543 , 0.0744519 , 0.0718442 , 0.0622228 , 0.0473734 , 0.0329352 , 0.0245156 , 0.0212818 , 0.0204027 , 0.0228792 , 0.0298908 , 0.0380399 , 0.0432513 , 0.0455291 , 
        0.0469428 , 0.049667 , 0.0453111 , 0.0370016 , 0.0278006 , 0.0201062 , 0.0173687 , 0.020228 , 0.0242543 , 0.0264199 , 0.0275476 , 0.027771 , 0.0262174 , 0.0237332 , 0.0219206 , 0.0212424 , 0.0214967 , 0.0226845 , 0.0248514 , 0.0275874 , 0.0300439 , 0.0318892 , 0.0340621 , 0.0369823 , 0.0388068 , 0.0379494 , 0.0350817 , 0.030462 , 0.0230471 , 0.0133404 , 0.00457234 , 0.00152755 , 0.00874873 , 0.0260448 , 0.0463293 , 0.0633742 , 0.0751071 , 0.0775575 , 0.0756597 , 0.0916989 , 0.141021 , 0.22185 , 0.328433 , 0.44207 , 0.524772 , 0.544711 , 0.498668 , 0.407614 , 0.29953 , 0.199594 , 0.128704 , 0.0929922 , 0.0772499 , 0.0654169 , 0.0536587 , 0.0399619 , 0.0255793 , 0.0193488 , 0.0253531 , 0.0373143 , 
    };//I've just hardcoded the measurement values.  Maybe later we'll read them from a text file.


    for (int i = 0; i < numberofexcitationangles*numberofobservationangles; i++) {
        cout << "D[" << i << " ]: " << D[i] << endl;
    }

    float fit;
    fit=fitness(D,numberofobservationangles*numberofexcitationangles, measurement);

    error = hipGetLastError();


    free(Ceze);
    free(Cezhy);
    free(Cezhx);
    free(Ez);
    free(eps_r_z);
    free(sigma_e_z);
    free(Hy);
    free(Hx);
    free(kex);
    free(aex);
    free(bex);
    free(amx);
    free(bmx);
    free(alpha_e);
    free(alpha_m);
    free(sigma_e_pml);
    free(sigma_m_pml);
    free(Psi_ezy);
    free(Psi_ezx);
    free(Psi_hyx);
    free(Psi_hxy);
    free(kmx);
    free(D);

    free(hcjzxp);
    free(hcjzyp);
    free(hcjzxn);
    free(hcjzyn);
    free(hcmxyp);
    free(hcmyxp);
    free(hcmxyn);
    free(hcmyxn);

    free(L);
    free(N);
    //free(measurement);
    //float *Cezeic = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    //float *Cezeip = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));

    //float*Ceze,*Cezhy,*Cezhx,*dev_Cezeic,*dev_Cezeip,*Ez,*eps_r_z,*sigma_e_z,*Hy,*Hx,
    //	*kex,*aex,*bex,*amx,*bmx,*alpha_e,*alpha_m,*sigma_e_pml,*sigma_m_pml
    //	,*Psi_ezy,*Psi_ezx,*Psi_hyx,*Psi_hxy,*kmx;//*Cezj later if using loop current source
    //float* dev_sigma_e_z,*dev_eps_r_z;
    //float freq = center_freq;
    //float *dev_freq,*D,*D_tot;
    //float* Ezip,*Ezic,*dev_Ezip,*dev_Ezic,*Hy_inc,*Hx_inc,*dev_Hy_inc,*dev_Hx_inc,*dev_Psi_ezy_inc,*dev_Psi_ezx_inc,*dev_Psi_hyx_inc,*dev_Psi_hxy_inc,
    //	*Psi_ezy_inc,*Psi_ezx_inc,*Psi_hyx_inc,*Psi_hxy_inc;
    //
    //cuComplex *cjzxp,*cjzyp,*cjzxn,*cjzyn,*cmxyp,*cmyxp,*cmxyn,*cmyxn,*cjzxp_tot,*cjzyp_tot,*cjzxn_tot,*cjzyn_tot,*cmxyp_tot,*cmyxp_tot,*cmxyn_tot,*cmyxn_tot;
    //cuComplex *hcjzxp,*hcjzyp,*hcjzxn,*hcjzyn,*hcmxyp,*hcmyxp,*hcmxyn,*hcmyxn,*hcjzxp_tot,*hcjzyp_tot,*hcjzxn_tot,*hcjzyn_tot,*hcmxyp_tot,*hcmyxp_tot,*hcmxyn_tot
    //	,*hcmyxn_tot;


    hipFree(dev_Cezeic);
    hipFree(dev_Cezeip);
    hipFree(dev_sigma_e_z);
    hipFree(dev_eps_r_z);
    hipFree(dev_freq);

    hipFree(cjzxp);
    hipFree(cjzyp);
    hipFree(cjzxn);
    hipFree(cjzyn);
    hipFree(cmxyp);
    hipFree(cmyxp);
    hipFree(cmxyn);
    hipFree(cmyxn);

    hipFree(dev_Ceze);
    hipFree(dev_Cezhy);
    hipFree(dev_Cezhx);

    hipFree(dev_bex);
    hipFree(dev_aex);
    hipFree(dev_bmx);
    hipFree(dev_amx);
    hipFree(dev_kex);
    hipFree(dev_kmx);
    hipFree(dev_Ez);
    hipFree(dev_Hy);
    hipFree(dev_Hx);
    hipFree(dev_Psi_ezy);
    hipFree(dev_Psi_ezx);
    hipFree(dev_Psi_hyx);
    hipFree(dev_Psi_hxy);
    //float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_Jz,*dev_bex,*dev_aex,*dev_bmx,*dev_amx,*dev_kex,*dev_kmx;//dev_Cezj if using loop current source
    //float *dev_Ez,*dev_Hy,*dev_Hx;

    //float*dev_Psi_ezy,*dev_Psi_ezx,*dev_Psi_hyx,*dev_Psi_hxy;

    cout << "fitness is: " << fit << endl;
    return (double)fit;
}

__global__ void scattered_parameter_init(float*eps_r_z,float*sigma_e_z,float*Cezeic,float*Cezeip)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    if(x<(nx+1)&&y<(ny+1))
    {
        Cezeic[dgetCell(x,y,nx+1)] = (2*(eps0-eps0*eps_r_z[dgetCell(x,y,nx+1)])-sigma_e_z[dgetCell(x,y,nx+1)]*dt)/(2*eps0*eps_r_z[dgetCell(x,y,nx+1)]+sigma_e_z[dgetCell(x,y,nx+1)]*dt);
        Cezeip[dgetCell(x,y,nx+1)] = -1*(2*(eps0-eps0*eps_r_z[dgetCell(x,y,nx+1)])+sigma_e_z[dgetCell(x,y,nx+1)]*dt)/(2*eps0*eps_r_z[dgetCell(x,y,nx+1)]+sigma_e_z[dgetCell(x,y,nx+1)]*dt);

    }
}

int getCell(int x, int y,int size)//size will just be the width in the x dimension of the array.
{
    return x+y*size;
}

float* Make2DfloatArray(int arraySizeX, int arraySizeY)
{
    float* theArray;
    theArray = (float*) malloc(arraySizeX*arraySizeY*sizeof(float*));

    return theArray;
} 

void waveform_time_init(float*time1)
{

    int size = number_of_time_steps;
    for(int i = 0;i<size;i++) 
    {
        time1[i]=(float)i*dt;
    }
}

void Jz_waveform(float * time,float*Jz_impressed)
{
    float w = 2*PI*center_freq;//center_freq is the frequency
    for(int i = 0;i<number_of_time_steps;i++)
    {
        Jz_impressed[i]= 10*sin(w*time[i]);
        //Jz_impressed[i]=exp(-1*((time[i]-2e-10)/5e-11)*(time[i]-2e-10)/(5e-11));

    }
}

void Ceze_init(float * eps_r_z, float* sig_e_z, float* Ceze)
{
    int size = nx+1;
    for(int j=0;j<ny+1;j++)
    {
        for(int i=0;i<size;i++)
        {
            Ceze[getCell(i,j,nx+1)] = (2*eps_r_z[getCell(i,j,nx+1)]*eps0-dt*sig_e_z[getCell(i,j,nx+1)])/(2*eps_r_z[getCell(i,j,nx+1)]*eps0+dt*sig_e_z[getCell(i,j,nx+1)]);
        }
    }
}

void Cezhy_init(float*eps_r_z, float* sigma_e_z,float* Cezhy,float*kex)
{
    int size = nx+1;
    for(int j =0;j<ny+1;j++)
    {
        for(int i=0;i<size;i++)
        {
            Cezhy[getCell(i,j,size)] = (2*dt/dx)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);

        }
    }
}

void Cezhx_init(float* eps_r_z,float*sigma_e_z,float*Cezhx,float*kex)
{
    int size=nx+1;
    for(int j=0;j<ny+1;j++)
    {
        for(int i =0;i<nx+1;i++)
        {
            Cezhx[getCell(i,j,size)]=(2*dt/dy)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);

        }
    }
}

void Cezj_init(float*eps_r_z,float*sigma_e_z,float*Cezj)
{
    int size =nx+1;
    for(int j=0;j<ny+1;j++)
    {
        for(int i=0;i<nx+1;i++)
        {
            Cezj[getCell(i,j,size)] = (-2*dt)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);

        }
    }
}

void Ez_init(float*Ez)
{
    int size=nx+1;
    for(int j = 0;j<ny+1;j++)
    {
        for(int i = 0;i<nx+1;i++)
        {
            Ez[getCell(i,j,size)] = (float)0;
        }
    }
}

/*void Jz_init(float*Jz)
  {
  for(int j =0;j<ny+1;j++)
  {
  for(int i = 0;i<nx+1;i++)
  {
  Jz[getCell(i,j,nx+1)] = 0;
  }
  }
  }*/

void Chyh_init(float*mu_r_y,float*sigma_m_y,float*Chyh)
{
    int size=nx;
    for(int i = 0;i<nx;i++)
        for(int j =0;j<ny;j++)
        {
            {
                Chyh[getCell(i,j,size)] = (2*mu_r_y[getCell(i,j,size)]*mu0-dt*sigma_m_y[getCell(i,j,size)])/(2*mu_r_y[getCell(i,j,size)]*mu0+dt*sigma_m_y[getCell(i,j,size)]);
            }
        }
}

void Chxh_init(float*mu_r_x,float*sigma_m_x,float*Chxh)
{
    int size=nx;
    for(int i = 0;i<nx;i++)
        for(int j =0;j<ny;j++)
        {
            {
                Chxh[getCell(i,j,size)] = (2*mu_r_x[getCell(i,j,size)]*mu0-dt*sigma_m_x[getCell(i,j,size)])/(2*mu_r_x[getCell(i,j,size)]*mu0+dt*sigma_m_x[getCell(i,j,size)]);
            }
        }
}

void Chyez_init(float*mu_r_y,float*sigma_m_y,float*Chyez)
{
    int size = nx;
    for(int j =0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Chyez[getCell(i,j,size)] = (2*dt/dx)/(2*mu_r_y[getCell(i,j,size)]*mu0+dt*sigma_m_y[getCell(i,j,size)]);
        }
    }
}

void Chxez_init(float*mu_r_x,float*sigma_m_x,float*Chxez)
{
    int size = nx;
    for(int j =0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Chxez[getCell(i,j,size)] = (2*dt/dy)/(2*mu_r_x[getCell(i,j,size)]*mu0+dt*sigma_m_x[getCell(i,j,size)]);
        }
    }
}

/*void Chym_init(float*mu_r_y,float*sigma_m_y,float*Chym)
  {
  int size = nx;
  for(int j =0;j<ny;j++)
  {
  for(int i = 0;i<size;i++)
  {
  Chym[getCell(i,j,size)] = (-2*dt)/(2*mu_r_y[getCell(i,j,size)]*mu0+dt*sigma_m_y[getCell(i,j,size)]);

  }
  }
  }
  void Chxm_init(float*mu_r_x,float*sigma_m_x,float*Chxm)
  {
  int size = nx;
  for(int j =0;j<ny;j++)
  {
  for(int i = 0;i<size;i++)
  {
  Chxm[getCell(i,j,size)] = (-2*dt)/(2*mu_r_x[getCell(i,j,size)]*mu0+dt*sigma_m_x[getCell(i,j,size)]);

  }
  }
  }*/

void eps_r_z_init(float * eps_r_z,const vector<float> &argument)
{
    int size = nx+1;
    float radius;//tumor_radius,tumor_radius_2,tumor_radius_3;
    for(int j =0;j<ny+1;j++)
    {
        for(int i = 0;i<nx+1;i++)
        {


            eps_r_z[getCell(i,j,size)] = 1;
            radius = sqrt(pow( ((float)i-nx/2)*dx,2) + pow( ((float)j-ny/2)*dy,2));
            //	tumor_radius = sqrt(pow( ((float)i - target_x)*dx,2) + pow( ((float)j-target_y)*dy,2));
            if(radius<=breast_radius)
            {
                eps_r_z[getCell(i,j,size)] = (float)argument.at(getOptimizationCell(i,j)); //This is the line that should be uncommented if using as forward solver
                //eps_r_z[getCell(i,j,size)] = 10;

                //if(tumor_radius <= tumor_size)//delete this if using as forward solver
                //{
                //	eps_r_z[getCell(i,j,size)] = 60;
                //}

            }



        }
    }
}

void sigma_e_z_init(float * sigma_e_z,float*sigma_e_pml, const vector<float> &argument)
{
    int size = nx+1;
    float radius;//,tumor_radius;


    for(int j =0;j<ny+1;j++)
    {
        for(int i = 0;i<nx+1;i++)
        {
            sigma_e_z[getCell(i,j,size)] = 0;
            radius = sqrt(pow( ((float)i-nx/2)*dx,2) + pow( ((float)j-ny/2)*dy,2));
            //tumor_radius = sqrt(pow( ((float)i - target_x)*dx,2) + pow( ((float)j-target_y)*dy,2));
            if(radius<=breast_radius)
            {
                sigma_e_z[getCell(i,j,size)] = (float)argument.at(getOptimizationCell(i,j)+9*9);
                //sigma_e_z[getCell(i,j,size)] = 0.15;
                //if(tumor_radius <= tumor_size)//delete this if using as forward solver
                //{
                //	sigma_e_z[getCell(i,j,size)] = 0.7;
                //}

            }

        }
    }
}

void Hy_init(float*Hy)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Hy[getCell(i,j,size)] = 0;
        }
    }
}

void Hx_init(float*Hx)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Hx[getCell(i,j,size)] = 0;
        }
    }
}

void My_init(float*My)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            My[getCell(i,j,size)] = 0;
        }
    }
}

void Mx_init(float*Mx)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Mx[getCell(i,j,size)] = 0;
        }
    }
}

void mu_r_y_init(float*mu_r_y)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            mu_r_y[getCell(i,j,size)] =1.000;
        }
    }

}

void mu_r_x_init(float*mu_r_x)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            mu_r_x[getCell(i,j,size)]=1.000;
        }
    }

}

void sigma_m_y_init(float*sigma_m_y)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            sigma_m_y[getCell(i,j,size)] = 0;
        }
    }
}

void sigma_m_x_init(float*sigma_m_x)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            sigma_m_x[getCell(i,j,size)] = 0;
        }
    }
}

void C_Psi_ezy_init(float *C_Psi_ezy,float*Cezhx)
{
    int size = 20;
    for(int j = 0;j<ny;j++)
        for( int i =0;i<size;i++)
        {
            if(i<10)
            {
                C_Psi_ezy[getCell(i,j,size)]=dy*Cezhx[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_ezy[getCell(i,j,size)]=dy*Cezhx[getCell(nx-20+i,j,nx)];
            }
        }
}

void C_Psi_ezx_init(float* C_Psi_ezx,float*Cezhy)
{
    int size_y=20;
    for(int j=0;j<size_y;j++)
    {
        for(int i=0;i<nx;i++)
        {
            if(j<10)
            {
                C_Psi_ezx[getCell(i,j,nx)] = dx*Cezhy[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_ezx[getCell(i,j,nx)] = dx*Cezhy[getCell(i,ny-20+j,nx)];
            }
        }
    }
}

void C_Psi_hyx_init(float*C_Psi_hyx,float*Chyez)
{
    int size_x=20;
    for(int j=0;j<ny;j++)
    {
        for(int i=0;i<size_x;i++)
        {
            if(i<10)
            {
                C_Psi_hyx[getCell(i,j,size_x)]=dx*Chyez[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_hyx[getCell(i,j,size_x)]=dx*Chyez[getCell(nx-20+i,j,nx)];
            }
        }
    }
}

void C_psi_hxy_init(float *C_Psi_hxy,float*Chxez)
{
    int size_y=20;
    for(int j=0;j<size_y;j++)
    {
        for(int i=0;i<nx;i++)
        {
            if(j<11)
            {
                C_Psi_hxy[getCell(i,j,nx)]=dy*Chxez[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_hxy[getCell(i,j,nx)]=dy*Chxez[getCell(i,ny-20+j,nx)];
            }
        }
    }
}

void aex_init(float*aex,float*sigma_e_pml,float*kex,float*alpha_e_x,float*bex)
{
    int size=ncells;
    //aex[0]=0.0;
    //cout<<"aex[0] = "<<aex[0]<<endl;
    for(int i=0;i<size;i++)
    {
        aex[i]=((bex[i]-1)*sigma_e_pml[i])/(dx*(sigma_e_pml[i]*kex[i]+alpha_e_x[i]*kex[i]*kex[i]));
        //cout<<"aex["<<i<<"] = "<<aex[i]<<endl;
    }
}

void bex_init(float*bex ,float*sigma_e_pml,float*kex,float*alpha_e_x)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        bex[i]=exp(-1*(dt/eps0)*(sigma_e_pml[i]/kex[i]+alpha_e_x[i]));
        //cout<<"bex["<<i<<"] = "<<bex[i]<<endl;
    }
}

void aey_init(float*aey,float*sigma_e_pml,float*key,float*alpha_e_y,float*bey)
{
    for(int i=0;i>ncells;i++)
    {
        aey[i]=(bey[i]-1)*sigma_e_pml[i]/(dy*(sigma_e_pml[i]*key[i]+alpha_e_y[i]*key[i]*key[i]));
    }
}

void bey_init(float*bey,float*sigma_e_pml,float*key,float*alpha_e_y)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        bey[i]=exp(-1*(dt/eps0)*(sigma_e_pml[i]/key[i]+alpha_e_y[i]));
    }
}

void amy_init(float*amy,float*sigma_m_pml,float*kmy,float*alpha_m_y,float*bmy)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        amy[i]=(bmy[i]-1)*sigma_m_pml[i]/(dx*(sigma_m_pml[i]*kmy[i]+alpha_m_y[i]*kmy[i]*kmy[i]));
    }
}

void bmy_init(float*bmy,float*sigma_m_pml,float*kmy,float*alpha_m_y)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        bmy[i]=exp(-1*(dt/mu0)*(sigma_m_pml[i]/kmy[i]+alpha_m_y[i]));
    }
}

void amx_init(float*amx,float*sigma_m_pml,float*kmx,float*alpha_m_x,float*bmx)
{
    int size=ncells;

    //cout<<" amx = "<<amx[0]<<endl;
    //amx[0]=0.0;
    //cout<<" amx = "<<amx[0]<<endl;
    for(int i=0;i<size;i++)
    {
        amx[i]=(bmx[i]-1)*sigma_m_pml[i]/(dx*(sigma_m_pml[i]*kmx[i]+alpha_m_x[i]*kmx[i]*kmx[i]));
        cout<<" amx = "<<amx[i]<<endl;
    }
}

void bmx_init(float*bmx,float*sigma_m_pml,float*kmx,float*alpha_m_x)
{
    int size=10;
    float argument;
    //float constant;
    for(int i=0;i<size;i++)
    {
        //constant = dt/mu0;
        //cout<< "dt/mu0 = "<<constant<<endl;
        argument = -1*(dt/mu0)*((sigma_m_pml[i]/kmx[i])+alpha_m_x[i]);
        bmx[i]=exp(argument);
        //cout<<"argument of bmx = "<<argument<<endl;
        //cout<<"bmx = "<<bmx[i]<<endl;
    }
}

void alpha_e_init(float*alpha_e)
{
    float rho;
    int size=ncells;
    for(int i=0;i<ncells;i++)
    {
        rho = ((float)i+0.25)/ncells;
        alpha_e[i]=alpha_min+(alpha_max-alpha_min)*rho;
        	//cout<<"alpha_e = "<<alpha_e[i]<<endl;
    }
}

void alpha_m_init(float*alpha_e,float*alpha_m)
{
    int size=ncells;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.75)/ncells;
        alpha_m[i]=(mu0/eps0)*(alpha_min+(alpha_max-alpha_min)*rho);
        //cout<<"alpha_m = "<<alpha_m[i]<<endl;
    }
}

void k_e_init(float*k)
{
    int size=ncells;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.25)/ncells;
        k[i]=pow(rho,npml)*(kmax-1)+1;
        //cout<<"kex ["<<i<<"]= "<<k[i]<<endl;

    }
}

void k_m_init(float*k)
{
    int size=ncells;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.75)/ncells;
        k[i]=pow(rho,npml)*(kmax-1)+1;
        //cout<<"kmx ["<<i<<"]= "<<k[i]<<endl;

    }
}

void sigma_e_pml_init(float* sigma_e_pml)  
{
    float sigma_max = (npml+1)/(150*PI*dx);
    int size = 10;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.25)/ncells;
        sigma_e_pml[i]=sigma_max*sigma_factor*pow(rho,npml);
        cout<<"sigma_e_pml = "<<sigma_e_pml[i]<<endl;
    }
}

void sigma_m_pml_init(float*sigma_m_pml,float*sigma_e_pml)
{
    float rho;
    int size = 10;
    float sigma_max = (npml+1)/(150*PI*dx);
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.75)/ncells;
        sigma_m_pml[i]=(mu0/eps0)*sigma_max*sigma_factor*pow(rho,npml);
        cout<<"sigma_m_pml "<<sigma_m_pml[i]<<endl;
    }
}

void Psi_ezy_init(float*Psi_ezy)
{  
    int size=nx*20;
    for(int i=0;i<size;i++)
    {
        Psi_ezy[i]=0.0;
    }
}

void Psi_ezx_init(float*Psi_ezx)
{
    int size=ny*20;
    for(int i=0;i<size;i++)
    {
        Psi_ezx[i]=0.0;
    }
}

void Psi_hyx_init(float*Psi_hyx)
{
    int size=ny*20;
    for(int i=0;i<size;i++)
    {
        Psi_hyx[i]=0.0;
    }
}

void Psi_hxy_init(float*Psi_hxy)
{
    int size=nx*20;  
    for(int i=0;i<size;i++)
    {
        Psi_hxy[i]=0.0;
    }
}

void CJ_Init(hipComplex * cjzyn,int size)
{
    hipComplex nullComplex(0,0);
    for( int i =0; i<size;i++)
    {
        cjzyn[i] = nullComplex;
    }
}
