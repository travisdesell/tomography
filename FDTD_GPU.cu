//#define GLEW_STATIC
//#pragma comment(lib,"glew32.lib")
//#include <windows.h>
//#include <gl/glew.h>
//#include <glut.h>
#include <complex>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <cstdlib>
#include <fstream>
#include <hip/hip_runtime.h>
//#include "stdafx.h"
#include <iomanip>
#include <time.h>
//#include <cuda_gl_interop.h>

//#include <cuComplex.h>
#include <vector>
#include <math_functions.h>
//#include "EasyBMP.h"
//#include "EasyBMP_DataStructures.h"
//#include "EasyBMP_VariousBMPutilities.h"

#define GL_GLEXT_PROTOTYPES
#define PI 3.141592653589793238
#define alpha_max 0.01
#define alpha_min 0.000
#define eps0 8.85418e-12
#define sigma_factor 1.0
#define ncells 10
#define mu0 (PI*4e-7)
#define center_freq (5e9)
#define eta0 (sqrt(mu0/eps0))
#define c0 (1.0/sqrt(mu0*eps0))
#define dt (dx/c0/2)// dx/c0/2
#define domain_size 0.18
#define dx (0.001)
#define NF2FFdistfromboundary 100
#define source_position 0.5
#define dy (0.001)
#define number_of_time_steps 3000
#define f1x (nx/2 - 150)       
#define f2x (nx/2+150) 
#define f1y (ny/2)
#define f2y (ny/2)
//#define nx ((int)ceil(domain_size/dx))
//#define ny ((int)ceil(domain_size/dy))
#define nx 400
#define ny 400
#define d (10*dx)
#define npml 2
#define kmax 10
#define isPW 1
#define isscattering 1
#define HANDLE_ERROR( err ) err
#define sigma_max_pml (3/(200*PI*dx))
#define size_NF2FF_total (2*nx-8*NF2FFdistfromboundary+2*ny-4)
#define size_cjzy (nx-2*NF2FFdistfromboundary-2)
#define size_cjzx (ny-2*NF2FFdistfromboundary)
#define numberofobservationangles  60
#define t0 (sqrt(20.0)*tau) // t0 = sqrt(20)*tau
#define l0 (nx*dx/2-breast_radius) 
#define pwidth 10
#define nc 20 // 20 cells per wavelength
#define  fmax  (c0/(nc*dx))// change if dy is bigger though now they're the same  fmax is the highest frequency this program can handle
#define tau (3.3445267e-11) // float ta bu = sqrt(2.3)*nc*dx/(PI*c0*1/sqrt(eps_r_MAX));  from a calculation of fmax.
//#define tau (5.288161e-11)
#define target_x (nx/2+105+25)//105 is breast_radius / dx
#define target_y (ny/2)
#define source_x (nx/2)      //(target_x-105-80)
#define source_y (ny/2)
#define breast_radius 0.0315 //87.535 mm  .  Sample size = 1.
#define tumor_size (20)
//#include <unistd.h>
//const cuComplex jcmpx (0.0, 1.0);
/*static void HandleError( cudaError_t err, const char *file,  int line ) {
  if (err != cudaSuccess) {
  printf( "%s in %s at line %d\n", cudaGetErrorString( err ),  file, line );
  exit( EXIT_FAILURE );
  }
  }*/


//__constant__ float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_Cezj,*dev_Jz,*dev_Chyh,*dev_Chxh,*dev_Chyez,*dev_Chxez,*dev_bex,*dev_bey,*dev_aex,*dev_aey,*dev_bmy,*dev_bmx,*dev_amy,*dev_amx,*dev_C_Psi_ezy,
//*dev_C_Psi_ezx,*dev_C_Psi_hxy,*dev_C_Psi_hyx;
struct hipComplex {
    float   r;
    float   i;
    __host__  __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __host__ __device__ hipComplex(float a): r(a), i(0) {}
    float magnitude2( void ) { return r * r + i * i; }
    __host__  __device__  hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __host__ __device__ hipComplex operator*(const float& a){
        return hipComplex(r*a,i*a);
    }

    __host__  __device__  hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
    __host__ __device__ hipComplex operator+(const float& a){
        return hipComplex(r+a,i);
    }
    __host__ __device__ void operator+=(const float& f){
        r += f;
    }
    __host__ __device__ void operator+=(const hipComplex& C);
    hipComplex();
};

__host__ __device__ hipComplex operator*(const float &f, const hipComplex &C)
{

    return hipComplex(C.r*f,C.i*f);
}

__host__ __device__ void hipComplex::operator+=(const hipComplex& C)
{
    r +=C.r;
    i += C.i;
}

__host__ __device__ float cuabs(hipComplex x)
{
    return sqrt(x.i*x.i + x.r*x.r);
}

__host__ __device__ hipComplex cuexp(hipComplex arg)
{
    hipComplex res(0,0);
    float s, c;
    float e = expf(arg.r);
    sincosf(arg.i,&s,&c);
    res.r = c * e;
    res.i = s * e;
    return res;

}

__device__ int isOnNF2FFBound(int x, int y)
{
    if(x==NF2FFdistfromboundary||x==nx-NF2FFdistfromboundary||y==NF2FFdistfromboundary||y==ny-NF2FFdistfromboundary)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ int getxfromthreadIdNF2FF(int index)
{
    int x=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))//yn
    {
        x = index+NF2FFdistfromboundary+1;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))//xp
    {
        x = nx-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))//yp
    {
        x = nx-NF2FFdistfromboundary  - (index-(nx-4*NF2FFdistfromboundary+ny-2))-2;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))//xn notice 2*nx-8*NF2FFdistfromboundary+2*ny-4 is the max index term.
    {
        x = NF2FFdistfromboundary;
    }
    return x;
}

__device__ int getyfromthreadIdNF2FF(int index)
{
    int y=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))
    {
        y = NF2FFdistfromboundary;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))
    {
        y = (index-(nx-2*NF2FFdistfromboundary-2))+NF2FFdistfromboundary;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))
    {
        y = ny-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))
    {
        y = ny-NF2FFdistfromboundary-(index-(2*nx-6*NF2FFdistfromboundary+ny-4))-1;
    }
    return y;
}

int CPUgetxfromthreadIdNF2FF(int index)
{
    int x=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))//yn
    {
        x = index+NF2FFdistfromboundary+1;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))//xp
    {
        x = nx-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))//yp
    {
        x = nx-NF2FFdistfromboundary  - (index-(nx-4*NF2FFdistfromboundary+ny-2))-2;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))//xn notice 2*nx-8*NF2FFdistfromboundary+2*ny-4 is the max index term.
    {
        x = NF2FFdistfromboundary;
    }
    return x;
}

int CPUgetyfromthreadIdNF2FF(int index)
{
    int y=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))
    {
        y = NF2FFdistfromboundary;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))
    {
        y = (index-(nx-2*NF2FFdistfromboundary-2))+NF2FFdistfromboundary;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))
    {
        y = ny-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))
    {
        y = ny-NF2FFdistfromboundary-(index-(2*nx-6*NF2FFdistfromboundary+ny-4))-1;
    }
    return y;
}

__device__ __host__ int isOnxn(int x)
{
    if(x==(NF2FFdistfromboundary))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnxp(int x)
{
    if(x==(nx-NF2FFdistfromboundary-1))
    { 
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnyp(int x,int y)
{
    if(y==(ny-NF2FFdistfromboundary-1)&&!isOnxn(x)&&!isOnxp(x))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnyn(int x, int y)
{
    if((y==(NF2FFdistfromboundary))&&!isOnxn(x)&&!(isOnxp(x)))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ int dgetCell(int x, int y, int size)
{
    return x +y*size;
}

__global__ void calculate_JandM(float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex*cjzxn,hipComplex*cjzyn,hipComplex*cmxyp,hipComplex*cmyxp,hipComplex*cmxyn,hipComplex*cmyxn)
{
    float freq = *f;
    int index = threadIdx.x+blockIdx.x*blockDim.x;// should launch 2*nx-8*NF2FFdistfromboundary+2*ny-4 threads. 
    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(isOnyp(x,y))
        {
            Ez = (dev_Ez[dgetCell(x,y+1,nx+1)]+dev_Ez[dgetCell(x,y,nx+1)])/2;
            float Hx = dev_Hx[dgetCell(x,y,nx)];
            cjzyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Hx*deltatime*cuexp((float)(-1)*j*(float)2*pi*freq*(float)(*timestep)*deltatime);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements

            cmxyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Ez*deltatime*cuexp((float)-1.0*j*(float)2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
        else if(isOnxp(x))//X faces override y faces at their intersections
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            float Hy = dev_Hy[dgetCell(x,y,nx)];

            cjzxp[index-(nx-2*NF2FFdistfromboundary-2)] += Hy*deltatime*cuexp(-1*j*2*pi*freq*(float)(*timestep)*(float)dt);//cjzxp and cmyxp have ny-2*NF2FFBound elements

            cmyxp[index-(nx-2*NF2FFdistfromboundary-2)] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*pi*freq*((float)(*timestep)+0.5)*(float)dt);// this is the discrete fourier transform, by the way.
        }
        else if(isOnyn(x,y))
        {  
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x,y+1,nx+1)])/2;
            float Hx=dev_Hx[dgetCell(x,y,nx)];

            cjzyn[index] += Hx*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt);	//cjzyn and cmxyn need to have nx-2*NF2FFbound-2 elements
            cmxyn[index] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
        else if(isOnxn(x))
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            cjzxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*dev_Hy[dgetCell(x,y,nx)]*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt); // cjzxn and cmyxn must have ny-2*NFdistfromboundary elements
            cmyxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Ez*(float)dt*cuexp(-1.0*j*2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
    }

}

__host__ __device__ float fwf(float timestep,float x, float y,float Phi_inc,float l)
{

    float ar;
    float ky, kx;//k hat
    sincosf(Phi_inc,&ky,&kx);

    ar = (float)timestep*dt-(float)t0-(1/(float)c0)*(ky*y*dx+kx*x*dy-l);
    //ar = timestep*dt-t0;

    //return exp(-1*(ar*ar)/(tau*tau));// gaussian pulse  argument is k dot r, 
    return exp(-1*ar*ar/(tau*tau));
    //return sin(2*PI*1e9*timestep*dt);
}

__global__ void H_field_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex)
{
    float buffer_Hy;
    float buffer_Hx;
    float Chez = (dt/dx)/(mu0);
    int x = threadIdx.x +blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if(x<nx&&y<nx)
    {
        buffer_Hy = dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        buffer_Hx = dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        if(x<ncells)
        {
            buffer_Hy= dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-x];
            dev_Psi_hyx[dgetCell(x,y,20)]=dev_bmx[ncells-1-x]*dev_Psi_hyx[dgetCell(x,y,20)]+dev_amx[ncells-1-x]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x,y,20)] ;
        } 
        if(x>=(nx-ncells))
        {
            buffer_Hy=dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[x-nx+ncells];
            dev_Psi_hyx[dgetCell(x-nx+20,y,2*ncells)]=dev_bmx[x-nx+ncells]*dev_Psi_hyx[dgetCell(x-nx+20,y,20)]+dev_amx[x-nx+ncells]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x-nx+20,y,20)];
        }
        if(y<ncells)
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-y];
            dev_Psi_hxy[dgetCell(x,y,nx)]=dev_bmy[ncells-1-y]*dev_Psi_hxy[dgetCell(x,y,nx)]+dev_amy[ncells-1-y]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y,nx)];  
        }
        if(y>=(ny-ncells))
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[y-ny+ncells];
            dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]=dev_bmy[y-ny+ncells]*dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]+dev_amy[y-ny+ncells]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y-nx+20,nx)];
        }
        //__syncthreads();
        if(isnan(buffer_Hx)) 
        {
            dev_Hx[dgetCell(x,y,nx)] = 0.0;
        }
        else 
        {
            dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        }

        if(isnan(buffer_Hy)) {
            dev_Hy[dgetCell(x,y,nx)] = 0.0;
        }
        else
        {
            dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
        }

        //dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        //dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
    }
}

__global__ void H_inc_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex)
{
    float buffer_Hy;
    float buffer_Hx;
    float Chez = (dt/dx)/(mu0);
    int x = threadIdx.x +blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if(x<nx&&y<nx)
    {
        buffer_Hy = dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        buffer_Hx = dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
        if(x<ncells)
        {
            buffer_Hy= dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-x];
            dev_Psi_hyx[dgetCell(x,y,20)]=dev_bmx[ncells-1-x]*dev_Psi_hyx[dgetCell(x,y,20)]+dev_amx[ncells-1-x]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x,y,20)] ;
        } 
        if(x>=(nx-ncells))
        {
            buffer_Hy=dev_Hy[dgetCell(x,y,nx)]+Chez*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[x-nx+ncells];
            dev_Psi_hyx[dgetCell(x-nx+20,y,2*ncells)]=dev_bmx[x-nx+ncells]*dev_Psi_hyx[dgetCell(x-nx+20,y,20)]+dev_amx[x-nx+ncells]*(dev_Ez[dgetCell(x+1,y,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[dgetCell(x-nx+20,y,20)];
        }
        if(y<ncells)
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[ncells-1-y];
            dev_Psi_hxy[dgetCell(x,y,nx)]=dev_bmy[ncells-1-y]*dev_Psi_hxy[dgetCell(x,y,nx)]+dev_amy[ncells-1-y]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y,nx)];  
        }
        if(y>=(ny-ncells))
        {
            buffer_Hx=dev_Hx[dgetCell(x,y,nx)]-Chez*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)])/kex[y-ny+ncells];
            dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]=dev_bmy[y-ny+ncells]*dev_Psi_hxy[dgetCell(x,y-ny+20,nx)]+dev_amy[y-ny+ncells]*(dev_Ez[dgetCell(x,y+1,nx+1)]-dev_Ez[dgetCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[dgetCell(x,y-nx+20,nx)];
        }
        //__syncthreads();
        if(isnan(buffer_Hx)) 
        {
            dev_Hx[dgetCell(x,y,nx)] = 0.0;
        }
        else 
        {
            dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        }

        if(isnan(buffer_Hy)) {
            dev_Hy[dgetCell(x,y,nx)] = 0.0;
        }
        else
        {
            dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
        }

        //dev_Hx[dgetCell(x,y,nx)] = buffer_Hx;
        //dev_Hy[dgetCell(x,y,nx)] = buffer_Hy;
    }
}

__global__ void E_field_update(int *i,float*dev_Ez,float*dev_Hy,float*dev_Hx,float*dev_Psi_ezx,float*dev_aex,float*dev_aey,float*dev_bex,float*dev_bey,float*dev_Psi_ezy,float*kex,float*Cezhy,float*Cezhx,float*Ceze,float*Cezeip,float*Cezeic,float*Ezip,float*Ezic)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    //	int offset = x+y*blockDim.x*gridDim.x;
    float buffer_Ez;
    //float Ceh = (dt/dx)/(eps0);
    float Cezj = -dt/eps0;

    if(x<=nx&&y<=ny)
    {

        //if(x==0||x==nx||y==0||y==ny)
        if(x==nx||y==ny||x==0||y==0)
        {
            buffer_Ez=0.0;
        }
        else
        {
            if(isscattering)
            {
                if(!isPW)
                {
                    buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])
                        -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])
                        +Cezeic[dgetCell(x,y,nx+1)]*Ezic[dgetCell(x,y,nx+1)]
                        +Cezeip[dgetCell(x,y,nx+1)]*Ezip[dgetCell(x,y,nx+1)];
                }
                else
                {
                    buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])
                        -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])
                        +Cezeic[dgetCell(x,y,nx+1)]*fwf((float)(*i)+0.5,x,y,0,l0)
                        +Cezeip[dgetCell(x,y,nx+1)]*fwf((float)(*i)-0.5,x,y,0,l0);
                }
            }
            else
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)]);
                if(x==(int)(source_x)&&y==(int)(source_y))
                {
                    buffer_Ez=buffer_Ez + 100*Cezj*fwf((float)(*i),0,0,0,0);
                }
            }

            //if(x==((int)nx/2)&&y==((int)nx/2))
            //{
            //	//buffer_Ez=buffer_Ez + Cezj*dev_Jz[*i];
            //	buffer_Ez=buffer_Ez + Cezj*fwf((float)(*i),0,0,0,0);
            //}
            if(x<=ncells&&x!=0)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[ncells-x]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[ncells-x];
                dev_Psi_ezx[dgetCell(x-1,y-1,20)] = dev_bex[ncells-x]*dev_Psi_ezx[dgetCell(x-1,y-1,20)]+dev_aex[ncells-x]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)]);
                buffer_Ez += Cezhy[dgetCell(x,y,nx+1)]*dx*dev_Psi_ezx[dgetCell(x-1,y-1,2*ncells)];
            }
            if(x>=(nx-ncells)&&x!=nx)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[x-nx+ncells]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[x-nx+ncells];
                dev_Psi_ezx[dgetCell(x-nx+20,y-1,20)]=dev_bex[x-nx+ncells]*dev_Psi_ezx[dgetCell(x-nx+20,y-1,20)]+dev_aex[x-nx+ncells]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)]);
                buffer_Ez+=Cezhy[dgetCell(x,y,nx+1)]*dx*dev_Psi_ezx[dgetCell(x-nx+20,y-1,2*ncells)];
            }
            if(y<=ncells&&y!=0)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[ncells-y]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[ncells-y];
                dev_Psi_ezy[dgetCell(x-1,y-1,nx)]=dev_bey[(ncells-y)]*dev_Psi_ezy[dgetCell(x-1,y-1,nx)]+dev_aey[(ncells-y)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhx[dgetCell(x,y,nx+1)]*dy*dev_Psi_ezy[dgetCell(x-1,y-1,nx)];
            }
            if(y>=(ny-ncells)&&y!=ny)
            {
                buffer_Ez = Ceze[dgetCell(x,y,nx+1)]*dev_Ez[dgetCell(x,y,nx+1)]+Cezhy[dgetCell(x,y,nx+1)]*(dev_Hy[dgetCell(x,y,nx)]-dev_Hy[dgetCell(x-1,y,nx)])/kex[y-ny+ncells]
                    -Cezhx[dgetCell(x,y,nx+1)]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)])/kex[y-ny+ncells];
                dev_Psi_ezy[dgetCell(x-1,y-ny+20,nx)]=dev_bey[y-ny+ncells]*dev_Psi_ezy[dgetCell(x-1,y-ny+20,nx)]+dev_aey[y-ny+ncells]*(dev_Hx[dgetCell(x,y,nx)]-dev_Hx[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhx[dgetCell(x,y,nx+1)]*dy*dev_Psi_ezy[dgetCell(x-1,y-ny+20,nx)];
            }
        }
        //		unsigned char green = 128+127*buffer_Ez/0.4;
        /*ptr[offset].x = 0;
          ptr[offset].y = green;
          ptr[offset].z = 0;
          ptr[offset].w = 255;*///OpenGL stuff

        //__syncthreads();
        if(isnan(buffer_Ez)) {
            dev_Ez[dgetCell(x,y,nx+1)] = 0.0;
        }
        else {
            dev_Ez[dgetCell(x,y,nx+1)] = buffer_Ez;
        }
        //dev_Ez[dgetCell(x,y,nx+1)] = buffer_Ez;
    }

}

__global__ void E_inc_update(int *i,float*dev_Hy_inc,float*dev_Hx_inc,float*dev_Psi_ezx_inc,float*dev_aex,float*dev_aey,float*dev_bex,float*dev_bey,float*dev_Psi_ezy_inc,float*kex,float*dev_Ezip,float*dev_Ezic)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    //	int offset = x+y*blockDim.x*gridDim.x;
    float buffer_Ez;
    //float Ceh = (dt/dx)/(eps0);
    float Cezj = -dt/eps0;
    float Ceze = 1;
    float Cezhy = (dt/(dx*eps0));

    if(x<=nx&&y<=ny)
    {

        //if(x==0||x==nx||y==0||y==ny)
        if(x==nx||y==ny||x==0||y==0)
        {
            buffer_Ez=0.0;
        }
        else
        {
            buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])
                -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)]);

            if(x==((int)source_x)&&y==(int)(source_y))
            {
                //buffer_Ez=buffer_Ez + Cezj*dev_Jz[*i];
                buffer_Ez=buffer_Ez + 100*Cezj*fwf((float)(*i),0,0,0,0);
            }
            if(x<=ncells&&x!=0)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[ncells-x]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[ncells-x];
                dev_Psi_ezx_inc[dgetCell(x-1,y-1,20)] = dev_bex[ncells-x]*dev_Psi_ezx_inc[dgetCell(x-1,y-1,20)]+dev_aex[ncells-x]*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)]);
                buffer_Ez += Cezhy*dx*dev_Psi_ezx_inc[dgetCell(x-1,y-1,2*ncells)];
            }
            if(x>=(nx-ncells)&&x!=nx)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[x-nx+ncells]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[x-nx+ncells];
                dev_Psi_ezx_inc[dgetCell(x-nx+20,y-1,20)]=dev_bex[x-nx+ncells]*dev_Psi_ezx_inc[dgetCell(x-nx+20,y-1,20)]+dev_aex[x-nx+ncells]*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)]);
                buffer_Ez+=Cezhy*dx*dev_Psi_ezx_inc[dgetCell(x-nx+20,y-1,2*ncells)];
            }
            if(y<=ncells&&y!=0)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[ncells-y]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[ncells-y];
                dev_Psi_ezy_inc[dgetCell(x-1,y-1,nx)]=dev_bey[(ncells-y)]*dev_Psi_ezy_inc[dgetCell(x-1,y-1,nx)]+dev_aey[(ncells-y)]*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhy*dy*dev_Psi_ezy_inc[dgetCell(x-1,y-1,nx)];
            }
            if(y>=(ny-ncells)&&y!=ny)
            {
                buffer_Ez = Ceze*dev_Ezic[dgetCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[dgetCell(x,y,nx)]-dev_Hy_inc[dgetCell(x-1,y,nx)])/kex[y-ny+ncells]
                    -Cezhy*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)])/kex[y-ny+ncells];
                dev_Psi_ezy_inc[dgetCell(x-1,y-ny+20,nx)]=dev_bey[y-ny+ncells]*dev_Psi_ezy_inc[dgetCell(x-1,y-ny+20,nx)]+dev_aey[y-ny+ncells]*(dev_Hx_inc[dgetCell(x,y,nx)]-dev_Hx_inc[dgetCell(x,y-1,nx)]);
                buffer_Ez-=Cezhy*dy*dev_Psi_ezy_inc[dgetCell(x-1,y-ny+20,nx)];
            }
        }
        dev_Ezip[dgetCell(x,y,nx+1)] = dev_Ezic[dgetCell(x,y,nx+1)];
        dev_Ezic[dgetCell(x,y,nx+1)] = buffer_Ez;
    }

}

float calc_radiated_power(hipComplex *cjzxp,hipComplex *cjzyp,hipComplex *cjzxn,hipComplex *cjzyn,hipComplex *cmxyp,hipComplex *cmyxp,hipComplex *cmxyn,hipComplex *cmyxn)
{
    int indexofleg1 = nx-2*NF2FFdistfromboundary-2;
    int indexofleg2 = nx+ny-4*NF2FFdistfromboundary-2;
    int indexofleg3 = 2*nx+ny-6*NF2FFdistfromboundary-4;
    int maxindex = 2*nx-8*NF2FFdistfromboundary+2*ny-4;
    int index;
    hipComplex cjz(0,0);
    hipComplex power = 0;

    for(index = 0; index<indexofleg1;index++)
    {   cjz = hipComplex(cjzyn[index].r,-1.0*cjzyn[index].i);//conjugation
        //z x x = y dot -y = -1
        power+=-1.0*cjz*cmxyn[index]*dx;// the negative one comes from the dot product between JxM and the n hat vector
    }
    for(index = indexofleg1; index<indexofleg2;index++)
    {
        cjz = hipComplex(cjzxp[index-indexofleg1].r,-1.0*cjzxp[index-indexofleg1].i);//making the conjugate
        // z cross y = -x dot x = -1
        power+= -1.0*cjz*cmyxp[index-indexofleg1]*dy;//positive x unit normal vector
    }
    for(index = indexofleg2;index<indexofleg3;index++)
    {
        // z cross x = y dot y = 1
        cjz = hipComplex(cjzyp[index-indexofleg2].r,-1.0*cjzyp[index-indexofleg2].i);
        power+= cjz*cmxyp[index-indexofleg2]*dx;//postive y unit normal vector
    }
    for(index = indexofleg3;index<maxindex;index++)
    {
        // z cross y = -x dot -x = 1 
        cjz = hipComplex(cjzxn[index-indexofleg3].r,-1.0*cjzxn[index-indexofleg3].i);
        power += cjz*cmyxn[index-indexofleg3]*dy;// negative x hat n vector
    }
    float realpower = power.r;
    realpower *= 0.5;
    return realpower;
}

float calc_incident_power(float freq)
{
    return (0.5/eta0)*pow(tau*sqrt(PI)*exp(-tau*tau*2*PI*freq*2*PI*freq/4),2);// just gonna assume gaussian pulse.  This is the fourier transform of the gaussian pulse.
}

__global__ void calculate_JandM_total(float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex*cjzxn,hipComplex*cjzyn,hipComplex*cmxyp,hipComplex*cmyxp,hipComplex*cmxyn,hipComplex*cmyxn,float*dev_Ezic,float*dev_Ezip,float*dev_Hx_inc,float*dev_Hy_inc)
{
    float freq = *f;
    int index = threadIdx.x+blockIdx.x*blockDim.x;// should launch 2*nx-8*NF2FFdistfromboundary+2*ny-4 threads. 
    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(isOnyp(x,y))
        {
            Ez = (dev_Ez[dgetCell(x,y+1,nx+1)]+dev_Ez[dgetCell(x,y,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x,y+1,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x,y+1,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hx = dev_Hx[dgetCell(x,y,nx)] + dev_Hx_inc[dgetCell(x,y,nx)];
            cjzyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Hx*deltatime*cuexp((float)(-1)*j*(float)2*pi*freq*(float)(*timestep)*deltatime);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements
            cmxyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Ez*deltatime*cuexp((float)-1.0*j*(float)2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
        else if(isOnxp(x))//X faces override y faces at their intersections
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x+1,y,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x+1,y,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hy = dev_Hy[dgetCell(x,y,nx)] + dev_Hy_inc[dgetCell(x,y,nx)];

            cjzxp[index-(nx-2*NF2FFdistfromboundary-2)] += Hy*deltatime*cuexp(-1*j*2*pi*freq*(float)(*timestep)*(float)dt);//cjzxp and cmyxp have ny-2*NF2FFBound elements

            cmyxp[index-(nx-2*NF2FFdistfromboundary-2)] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*pi*freq*((float)(*timestep)-0.5)*(float)dt);// this is the discrete fourier transform, by the way.
        }
        else if(isOnyn(x,y))
        {  
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x,y+1,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x,y+1,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x,y+1,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hx=dev_Hx[dgetCell(x,y,nx)]+dev_Hx_inc[dgetCell(x,y,nx)];

            cjzyn[index] += Hx*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt);	//cjzyn and cmxyn need to have nx-2*NF2FFbound-2 elements
            cmxyn[index] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
        else if(isOnxn(x))
        {
            Ez = (dev_Ez[dgetCell(x,y,nx+1)]+dev_Ez[dgetCell(x+1,y,nx+1)])/2;
            Ez += (dev_Ezic[dgetCell(x+1,y,nx+1)] + dev_Ezic[dgetCell(x,y,nx+1)] + dev_Ezip[dgetCell(x+1,y,nx+1)] + dev_Ezip[dgetCell(x,y,nx+1)])/4;
            float Hy = dev_Hy[dgetCell(x,y,nx)] + dev_Hy_inc[dgetCell(x,y,nx)];
            cjzxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Hy*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt); // cjzxn and cmyxn must have ny-2*NFdistfromboundary elements
            cmyxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Ez*(float)dt*cuexp(-1.0*j*2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
    }

}

__host__ __device__ int getOptimizationCell(int x, int y)
{
    int x_coord,y_coord;
    x_coord = (x-(nx/2-(int)(breast_radius/dx)))/(2*breast_radius/(9*dx));
    y_coord = (y-(ny/2-breast_radius/dy))/(2*breast_radius/(9*dy));//the optimization space is 216 FDTD cells wide and high. //The optimization space is split into 25 by 25 optimization cells. 
    //each optimization cell has 24 by 24 FDTD cells within it. That's what the 108, 24 and 25 are about.  
    return x_coord+9*y_coord;//The max return should be, 9*9-1, hopefully.
}

void N2FPostProcess (float* D,float f, hipComplex *N,hipComplex *L,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex *cjzxn,hipComplex *cjzyn,hipComplex *cmxyp,hipComplex *cmyxp,hipComplex *cmxyn,hipComplex *cmyxn)
{
    int indexofleg1 = nx-2*NF2FFdistfromboundary-2;
    int indexofleg2 = nx+ny-4*NF2FFdistfromboundary-2;
    int indexofleg3 = 2*nx+ny-6*NF2FFdistfromboundary-4;
    int maxindex = 2*nx-8*NF2FFdistfromboundary+2*ny-4;
    int x,y;

    float rhoprime;
    float Psi;
    int Phi_index;
    hipComplex  Mphi(0,0);
    float Phi;


    float k = 2*PI*f/c0;
    hipComplex  negativeone(-1.0,0.0);
    int index = 0;
    hipComplex jcmpx(0,1);
    //float Prad = calc_radiated_power(cjzxp,cjzyp,cjzxn,cjzyn,cmxyp,cmyxp,cmxyn,cmyxn);
    float Prad = calc_incident_power(f);
    //std::cout<<"Prad = "<<Prad<<std::endl;
    float flx, fly;
    for(Phi_index = 0; Phi_index<numberofobservationangles;Phi_index++)
    {
        Phi = 2*PI/numberofobservationangles*(float)Phi_index;
        for(index = 0;index<indexofleg1;index++)
        {

            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x;//float x
            fly = (float)y + 0.5;
            rhoprime = sqrt(pow((dx*((-1.0*(float)nx/2)+1+flx)),2)+pow((dy*(-1.0*(float)ny/2+1+fly)),2));
            Psi = atan2(-1*((float)ny/2)+1+fly,-1*((float)nx/2)+1+flx)-Phi;
            N[Phi_index]+=-1.0*cjzyn[index]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;
            L[Phi_index]+=-1.0*sin(Phi)*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*cmxyn[index]*dx;//Lphi = 

        }
        for(index = indexofleg1;index<indexofleg2;index++)
        {

            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x+0.5;
            fly = (float)y;
            rhoprime = sqrt(pow((dx*(((float)nx/2)-1-flx)),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2(-1*((float)ny/2)+1+fly,(-1*((float)nx/2)+1+flx))-Phi;
            N[Phi_index]+=-1.0*cjzxp[index-indexofleg1]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
            L[Phi_index]+=cos(Phi)*cmyxp[index-indexofleg1]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;//L_phi = -Lxsin(phi)+Lycos(Phi) here we only have Ly
        }
        for(index=indexofleg2;index<indexofleg3;index++)
        {
            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x;
            fly = (float)y + 0.5;
            rhoprime = sqrt(pow((dx*(((float)nx/2)-1-flx)),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2((-1*(float)ny/2+1+fly),(-1*((float)nx/2)+1+flx))-Phi;
            N[Phi_index]+=-1.0*cjzyp[index-indexofleg2]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;
            L[Phi_index]+=-1.0*sin(Phi)*cmxyp[index-indexofleg2]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;//
        }
        for(index = indexofleg3;index<maxindex;index++)
        {
            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x+0.5;
            fly = (float)y;
            rhoprime = sqrt(pow(dx*(((float)nx/2)-1-flx),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2(-1*((float)ny/2)+1+fly,-1*(float)nx/2+1+flx)-Phi;
            N[Phi_index]+=-1.0*cjzxn[index-indexofleg3]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
            L[Phi_index]+= cos(Phi)*cmyxn[index-indexofleg3]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
        }
        D[Phi_index] = (k*k*cuabs(L[Phi_index]+(float)eta0*N[Phi_index])*cuabs(L[Phi_index]+(float)eta0*N[Phi_index])/((float)8*(float)PI*(float)eta0*Prad*33.329));//why 33.329?  I dunno, something is probably wrong with Prad.
    }

}

float fitness(float* D,int numberobservationangles, float* measurement)
{
    float fit = 0;
    for(int i =0;i<numberobservationangles;i++)
    {
        fit -= pow((measurement[i]-D[i]),2)/pow(measurement[i],2);
    }

    return fit;
}

//static void draw_func(void){
//	glDrawPixels(nx,ny,GL_RGBA,GL_UNSIGNED_BYTE,0);
//	glutSwapBuffers;
//}

using namespace std;

void Ceze_init(float * eps_r_z, float* sig_e_z, float* Ceze);
void Cezhy_init(float* eps_r_z, float* sigma_e_z,float*Cezhy,float*kex);
void Cezhx_init(float* eps_r_z,float*sigma_e_z,float*Cezhx,float*kex);
void eps_r_z_init(float * eps_r_z,const vector<float> &argument);
void sigma_e_z_init(float *sigma_e_z,float*sigma_e_pml,const vector<float> &argument);
void Cezj_init(float*eps_r_z,float*sigma_e_z,float*Cezj);
void Ez_init(float*Ez);
void Ey_init(float*Ey);
//void Jz_init(float*Jz);
void Chxh_init(float*mu_r_x,float*sigma_m_x,float*Chxh);
void Chxez_init(float*mu_r_x,float*sigma_m_x,float*Chxez);
//void Chxm_init(float*mu_r_x,float*sigma_m_x,float*Chxm);
void Chyh_init(float*mu_r_y,float*sigma_m_y,float*Chyh);
void Chyez_init(float*mu_r_y,float*sigma_m_y,float*Chyez);
//void Chym_init(float*mu_r_y,float*sigma_m_y,float*Chym);
void Hy_init(float*Hy);
void Hx_init(float*Hx);
void My_init(float*My);
void Mx_init(float*Mx);
void mu_r_y_init(float*mu_r_y);
void mu_r_x_init(float*mu_r_x);
void sigma_m_y_init(float*sigma_m_y_init);
void sigma_m_x_init(float*sigma_m_x_init);
int getCell(int x,int y,int size);
void Jz_waveform(float * time,float*Jz_impressed);
void waveform_time_init(float*time1);
float* Make2DfloatArray(int arraySizeX, int arraySizeY);
void C_Psi_ezy_init(float *C_Psi_ezy,float*Cezhx);
void C_Psi_ezx_init(float* C_Psi_ezx,float*Cezhy);
void C_Psi_hyx_init(float*C_Psi_hyx,float*Chyez);
void C_psi_hxy_init(float *C_Psi_hxy,float*Chxez);

void aex_init(float*aex,float*sigma_e_pml,float*kex,float*alpha_e_x,float*bex);
void bex_init(float*bex ,float*sigma_e_pml,float*kex,float*alpha_e_x);   
void bey_init(float*bey,float*sigma_e_pml,float*key,float*alpha_e_y);
void amy_init(float*amy,float*sigma_m_pml,float*kmy,float*alpha_m_y,float*bmy);
void bmy_init(float*bmy,float*sigma_m_pml,float*kmy,float*alpha_m_y);
void amx_init(float*amx,float*sigma_m_pml,float*kmx,float*alpha_m_x,float*bmx);
void bmx_init(float*bmx,float*sigma_m_pml,float*kmx,float*alpha_m_x);
void alpha_e_init(float*alpha_e);
void alpha_m_init(float*alpha_e,float*alpha_m);
void k_e_init(float*k);
void k_m_init(float*k);
void sigma_e_pml_init(float* sigma_e_pml);
void sigma_m_pml_init(float*sigma_m_pml,float*sigma_e_pml);
void Psi_ezy_init(float*Psi_ezy);
void Psi_ezx_init(float*Psi_ezx);
void Psi_hyx_init(float*Psi_hyx);
void Psi_hxy_init(float*Psi_hxy);
void CJ_Init(hipComplex * cjzyn,int size);
__global__ void scattered_parameter_init(float*eps_r_z,float*sigma_e_z,float*Cezeic,float*Cezeip);

double FDTD_GPU(const vector<double> &arguments)
{
    //BMP Output_Image;
    //BMP Scattered_Field_snapshot;
    //Output_Image.SetSize((nx+1),(ny+1));
    //Output_Image.SetBitDepth(16);
    //Scattered_Field_snapshot.SetSize((nx+1),(ny+1));
    //Scattered_Field_snapshot.SetBitDepth(16);
    //RGBApixel Temp;
    //string outputfilename;
    //ebmpBYTE StepSize;

    cout << "calculating FDTD GPU" << endl;

    hipSetDevice(0);

    vector<float> image;
    for(int lerp = 0; lerp<81;lerp++)//This is setting the material parameters of the optimization cells.
    {
        image.push_back((float)arguments.at(lerp));
        //image.push_back(20);
    }
    for(int lerp = 81; lerp<81*2;lerp++)
    {
        image.push_back((float)arguments.at(lerp));
        //image.push_back(0);
    }


    //GLuint bufferObj;
    //cudaGraphicsResource *resource;
    hipError_t error;
    //int dev;
    //cudaDeviceProp prop;
    //memset(&prop,sizeof(cudaDeviceProp),sizeof(cudaDeviceProp));
    //prop.major = 1;
    //prop.minor = 1;
    //cudaChooseDevice(&dev,&prop);
    //	cudaGLSetGLDevice(dev);
    /*glutInit(&argc,argv);
      glewInit();
      glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);  
      glutInitWindowSize(nx,ny);
      glutCreateWindow("bitmap");
      glGenBuffers(1,&bufferObj);
      glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
      glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, nx*ny*4,NULL,GL_DYNAMIC_DRAW_ARB);
      cudaGraphicsGLRegisterBuffer(&resource,bufferObj,cudaGraphicsMapFlagsNone);*/
    //uchar4* devPtr;
    //size_t size;
    //cudaGraphicsMapResources(1,&resource,NULL);
    //cudaGraphicsResourceGetMappedPointer((void**)&devPtr,&size,resource);

    float*Ceze,*Cezhy,*Cezhx,*dev_Cezeic,*dev_Cezeip,*Ez,*eps_r_z,*sigma_e_z,*Hy,*Hx,
        *kex,*aex,*bex,*amx,*bmx,*alpha_e,*alpha_m,*sigma_e_pml,*sigma_m_pml
            ,*Psi_ezy,*Psi_ezx,*Psi_hyx,*Psi_hxy,*kmx;//*Cezj later if using loop current source
    float* dev_sigma_e_z,*dev_eps_r_z;
    float freq = center_freq;
    float *dev_freq,*D,*D_tot;
    float* Ezip,*Ezic,*dev_Ezip,*dev_Ezic,*Hy_inc,*Hx_inc,*dev_Hy_inc,*dev_Hx_inc,*dev_Psi_ezy_inc,*dev_Psi_ezx_inc,*dev_Psi_hyx_inc,*dev_Psi_hxy_inc,
        *Psi_ezy_inc,*Psi_ezx_inc,*Psi_hyx_inc,*Psi_hxy_inc;

    hipComplex *cjzxp,*cjzyp,*cjzxn,*cjzyn,*cmxyp,*cmyxp,*cmxyn,*cmyxn,*cjzxp_tot,*cjzyp_tot,*cjzxn_tot,*cjzyn_tot,*cmxyp_tot,*cmyxp_tot,*cmxyn_tot,*cmyxn_tot;
    hipComplex *hcjzxp,*hcjzyp,*hcjzxn,*hcjzyn,*hcmxyp,*hcmyxp,*hcmxyn,*hcmyxn,*hcjzxp_tot,*hcjzyp_tot,*hcjzxn_tot,*hcjzyn_tot,*hcmxyp_tot,*hcmyxp_tot,*hcmxyn_tot
        ,*hcmyxn_tot;
    int grid_x = int(ceil((float)nx/22));
    int grid_y = int(ceil((float)ny/22));
    dim3 grid(grid_x,grid_y);
    dim3 block(22,22);
    Hy_inc = (float*)malloc(sizeof(float)*(nx*ny));
    Hx_inc = (float*)malloc(sizeof(float)*(nx*ny));
    Ezip = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    Ezic = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    for(int index = 0;index<(1+nx)*(1+ny);index++)
    {
        Ezip[index] = 0;
        Ezic[index] = 0;
        if(index<(nx*ny))
        {
            Hy_inc[index] = 0;
            Hx_inc[index] = 0;
        }
    }
    hipMalloc(&dev_Ezip,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Ezic,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Hy_inc,sizeof(float)*(nx)*(ny));
    hipMalloc(&dev_Hx_inc,sizeof(float)*(nx)*(ny));
    hipMemcpy(dev_Ezip,Ezip,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_Ezic,Ezic,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_Hy_inc,Hy_inc,sizeof(float)*ny*nx,hipMemcpyHostToDevice);
    hipMemcpy(dev_Hx_inc,Hx_inc,sizeof(float)*ny*nx,hipMemcpyHostToDevice);

    Ceze = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    Cezhy = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    Cezhx = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    //Cezj = (float*)malloc(sizeof(float)*(1+nx)*(1+ny)); // if using loop current source
    Ez = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    eps_r_z =  (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    sigma_e_z = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    D = (float*)malloc(sizeof(float)*numberofobservationangles);//D = (float*)malloc(numberofobservationangles*sizeof(float));
    Hy=(float*)malloc(sizeof(float)*nx*ny);
    Hx=(float*)malloc(sizeof(float)*nx*ny);
    kex = (float*)malloc(sizeof(float)*10);
    kmx = (float*)malloc(sizeof(float)*10);
    aex=(float*)malloc(sizeof(float)*10);
    bex=(float*)malloc(sizeof(float)*10);
    amx=(float*)malloc(sizeof(float)*10);
    bmx=(float*)malloc(sizeof(float)*10);
    alpha_e=(float*)malloc(sizeof(float)*10);
    alpha_m=(float*)malloc(sizeof(float)*10);
    sigma_e_pml=(float*)malloc(sizeof(float)*10);
    sigma_m_pml=(float*)malloc(sizeof(float)*10);
    Psi_ezy=(float*)malloc(sizeof(float)*ny*20);
    Psi_ezx=(float*)malloc(sizeof(float)*nx*20);
    Psi_hyx=(float*)malloc(sizeof(float)*ny*20);
    Psi_hxy=(float*)malloc(sizeof(float)*nx*20);
    Psi_ezy_inc=(float*)malloc(sizeof(float)*ny*20);
    Psi_ezx_inc=(float*)malloc(sizeof(float)*nx*20);
    Psi_hyx_inc=(float*)malloc(sizeof(float)*ny*20);
    Psi_hxy_inc=(float*)malloc(sizeof(float)*nx*20);
    hcjzyp = (hipComplex*)malloc(sizeof(hipComplex )*size_cjzy);
    hcjzyn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcjzxp = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcjzxn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcmxyn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcmxyp = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcmyxp = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcmyxn = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);

    hcjzyp_tot = (hipComplex*)malloc(sizeof(hipComplex )*size_cjzy);
    hcjzyn_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcjzxp_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcjzxn_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcmxyn_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcmxyp_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzy);
    hcmyxp_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    hcmyxn_tot = (hipComplex *)malloc(sizeof(hipComplex )*size_cjzx);
    CJ_Init(hcjzyp,size_cjzy);//C**** coefficients are for surface current/ field duality for NF2FF processing.
    CJ_Init(hcjzyn,size_cjzy);
    CJ_Init(hcjzxp,size_cjzx);
    CJ_Init(hcjzxn,size_cjzx);
    CJ_Init(hcmxyn,size_cjzy);
    CJ_Init(hcmxyp,size_cjzy);
    CJ_Init(hcmyxp,size_cjzx);
    CJ_Init(hcmyxn,size_cjzx);
    CJ_Init(hcjzyp_tot,size_cjzy);//C**** coefficients are for surface current/ field duality for NF2FF processing.
    CJ_Init(hcjzyn_tot,size_cjzy);
    CJ_Init(hcjzxp_tot,size_cjzx);
    CJ_Init(hcjzxn_tot,size_cjzx);
    CJ_Init(hcmxyn_tot,size_cjzy);
    CJ_Init(hcmxyp_tot,size_cjzy);
    CJ_Init(hcmyxp_tot,size_cjzx);
    CJ_Init(hcmyxn_tot,size_cjzx);
    Psi_ezy_init(Psi_ezy);
    Psi_ezx_init(Psi_ezx);
    Psi_hyx_init(Psi_hyx);
    Psi_hxy_init(Psi_hxy);
    Psi_ezy_init(Psi_ezy_inc);
    Psi_ezx_init(Psi_ezx_inc);
    Psi_hyx_init(Psi_hyx_inc);
    Psi_hxy_init(Psi_hxy_inc);
    eps_r_z_init(eps_r_z,image);
    sigma_e_z_init(sigma_e_z,sigma_e_pml,image);
    Hy_init(Hy);
    Hx_init(Hx);
    //float*time1;
    //time1 = (float*)malloc(sizeof(float)*number_of_time_steps);
    Ceze_init(eps_r_z,sigma_e_z,Ceze);	
    k_e_init(kex);
    k_m_init(kmx);
    Cezhy_init(eps_r_z,sigma_e_z,Cezhy,kex);
    Cezhx_init(eps_r_z,sigma_e_z,Cezhx,kex);
    sigma_e_pml_init(sigma_e_pml);
    sigma_m_pml_init(sigma_m_pml,sigma_e_pml);
    alpha_e_init(alpha_e);
    alpha_m_init(alpha_e,alpha_m);
    bex_init(bex ,sigma_e_pml,kex,alpha_e);
    aex_init(aex,sigma_e_pml,kex,alpha_e,bex);
    bmx_init(bmx,sigma_m_pml,kmx,alpha_m);
    amx_init(amx,sigma_m_pml,kmx,alpha_m,bmx);
    Ez_init(Ez);
    //Jz_init(Jz);
    //system("pause");   
    //FILE* file = fopen("results.txt", "w");

    //float*Jz_impressed = (float*)malloc(sizeof(float)*number_of_time_steps);
    //waveform_time_init(time1);
    //Jz_waveform(time1,Jz_impressed);

    //int source_position_index_x = int(nx*source_position/domain_size)+1;

    //	int source_position_index_y = int(ny*source_position/domain_size)+1;
    float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_bex,*dev_aex,*dev_bmx,*dev_amx,*dev_kex,*dev_kmx;//dev_Cezj if using loop current source
    float *dev_Ez,*dev_Hy,*dev_Hx;

    float*dev_Psi_ezy,*dev_Psi_ezx,*dev_Psi_hyx,*dev_Psi_hxy;

    hipMalloc(&dev_eps_r_z,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_sigma_e_z,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezeic,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezeip,sizeof(float)*(nx+1)*(ny+1));
    hipMemcpy(dev_eps_r_z,eps_r_z,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_sigma_e_z,sigma_e_z,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    scattered_parameter_init<<<grid,block>>>(dev_eps_r_z,dev_sigma_e_z,dev_Cezeic,dev_Cezeip);
    float *Cezeic = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    float *Cezeip = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    hipMemcpy(Cezeic,dev_Cezeic,sizeof(float)*(nx+1)*(ny+1),hipMemcpyDeviceToHost);
    hipMemcpy(Cezeip,dev_Cezeip,sizeof(float)*(nx+1)*(ny+1),hipMemcpyDeviceToHost);
    float radius;
    //for(int i = 0; i<(nx+1);i++)
    //{
    //	for(int j =0; j<(ny+1);j++)
    //	{
    //		radius = sqrt((i-nx/2)*(i-nx/2)*dx*dx+(j-ny/2)*(j-ny/2)*dy*dy);
    //		if(radius<breast_radius)
    //		{
    //		cout<<"Cezeip = "<<Cezeip[getCell(i,j,nx+1)]<<"(i,j) = ("<<i<<","<<j<<")"<<endl;
    //		//cin.ignore();
    //		}
    //	}
    //}

    hipMalloc(&dev_kex,sizeof(float)*10);
    hipMalloc(&dev_kmx,sizeof(float)*10);
    hipMalloc(&dev_Ez,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Hy,sizeof(float)*nx*ny);
    hipMalloc(&dev_freq ,sizeof(float));
    hipMalloc(&dev_Hx,sizeof(float)*nx*ny);
    hipMalloc(&dev_Psi_ezy,sizeof(float)*20*(nx+1));
    hipMalloc(&dev_Psi_ezx,sizeof(float)*20*(ny+1));
    hipMalloc(&dev_Psi_hyx,sizeof(float)*20*(ny));
    hipMalloc(&dev_Psi_hxy,sizeof(float)*20*(nx));
    hipMalloc(&dev_Psi_ezy_inc,sizeof(float)*20*(nx+1));
    hipMalloc(&dev_Psi_ezx_inc,sizeof(float)*20*(ny+1));
    hipMalloc(&dev_Psi_hyx_inc,sizeof(float)*20*(ny));
    hipMalloc(&dev_Psi_hxy_inc,sizeof(float)*20*(nx));
    hipMalloc(&cjzxp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzxn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzxp_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyp_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzxn_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyn_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyp_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyn_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxp_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxn_tot,sizeof(hipComplex)*size_NF2FF_total);
    hipMemcpy(dev_freq,&freq,sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_Psi_ezy,Psi_ezy,sizeof(float)*20*(nx),hipMemcpyHostToDevice);
    hipMemcpy(dev_Psi_ezx,Psi_ezx,sizeof(float)*20*(ny),hipMemcpyHostToDevice);
    hipMemcpy(dev_Psi_hyx,Psi_hyx,sizeof(float)*20*(ny),hipMemcpyHostToDevice );
    hipMemcpy(dev_Psi_hxy,Psi_hxy,sizeof(float)*20*nx,hipMemcpyHostToDevice);
    hipMemcpy(dev_Psi_ezy_inc,Psi_ezy_inc,sizeof(float)*20*nx,hipMemcpyHostToDevice);
    hipMemcpy(dev_Psi_ezx_inc,Psi_ezx_inc,sizeof(float)*20*ny,hipMemcpyHostToDevice);
    hipMemcpy(dev_Psi_hyx_inc,Psi_hyx_inc,sizeof(float)*20*ny,hipMemcpyHostToDevice );
    hipMemcpy(dev_Psi_hxy_inc,Psi_hxy_inc,sizeof(float)*20*nx,hipMemcpyHostToDevice);
    hipMemcpy(dev_Ez,Ez,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_Hy,Hy,sizeof(float)*nx*ny,hipMemcpyHostToDevice);
    hipMemcpy(dev_Hx,Hx,sizeof(float)*nx*ny,hipMemcpyHostToDevice);
    hipMalloc(&dev_bex,sizeof(float)*10);
    hipMalloc(&dev_bmx,sizeof(float)*10);
    hipMalloc(&dev_amx,sizeof(float)*10);
    hipMalloc(&dev_aex,sizeof(float)*10);
    hipMalloc(&dev_Ceze,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezhy,sizeof(float)*(nx+1)*(ny+1));


    //cudaMalloc(&dev_Cezj,sizeof(float)*(nx+1)*(ny+1)); if using current source

    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }
    hipMemcpy(cjzyn,hcjzyn,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cjzxp,hcjzxp,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);
    hipMemcpy(cjzyp,hcjzyp,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cjzxn,hcjzxn,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);
    hipMemcpy(cmxyn,hcmxyn,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cmxyp,hcmxyp,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cmyxn,hcmyxn,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);
    hipMemcpy(cmyxp,hcmyxp,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);

    hipMemcpy(cjzyn_tot,hcjzyn_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cjzxp_tot,hcjzxp_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);
    hipMemcpy(cjzyp_tot,hcjzyp_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cjzxn_tot,hcjzxn_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);
    hipMemcpy(cmxyn_tot,hcmxyn_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cmxyp_tot,hcmxyp_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyHostToDevice);
    hipMemcpy(cmyxn_tot,hcmyxn_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);
    hipMemcpy(cmyxp_tot,hcmyxp_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyHostToDevice);

    hipMemcpy(dev_kex,kex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_kmx,kmx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_aex,aex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_bex,bex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_bmx,bmx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_amx,amx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_Ceze,Ceze,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_Cezhy,Cezhy,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }

    int*dev_i;
    hipMalloc(&dev_i,sizeof(int));
    float test_Ez;

    dim3 gridNF2FF((int)ceil(size_NF2FF_total/512.0));
    dim3 blockNF2FF(512);
    //system("PAUSE");
    //time_t start,end;
    float test_Ez_2;
    //time(&start);
    //error = cudaGetLastError();
    //printf("%s\n",cudaGetErrorString(error));
    //ofstream myfile;
    //ifstream fields;
    //fields.open("Field_snapshot.txt");
    //if(!fields)
    //{
    //	cout<<"Couldn't open file"<<endl;
    //	cin>>test_Ez_2;
    //}
    //else
    //{
    //	cout<<"File opening success!"<<endl;
    //}
    //myfile.open("Scattered_Ez_at_f2.txt");
    //myfile<<"foci1      foci2"<<endl;
    //ostringstream convert;
    //int x,y;
    /* The calculation part! */
    for(int i=0;i<number_of_time_steps;i++)
    {
        hipMemcpy(dev_i,&i,sizeof(int),hipMemcpyHostToDevice);

        //H_inc_update<<<grid,block>>>(dev_Hy_inc,dev_Hx_inc,dev_Ezic,dev_bmx,dev_Psi_hyx_inc,dev_amx,dev_bmx,dev_amx,dev_Psi_hxy_inc,dev_kmx);
        //E_inc_update<<<grid,block>>>(dev_i,dev_Hy_inc,dev_Hx_inc,dev_Psi_ezx_inc,dev_aex,dev_aex,dev_bex,dev_bex,dev_Psi_ezy_inc,dev_kex,dev_Ezip,dev_Ezic);

        H_field_update<<<grid,block>>>(dev_Hy,dev_Hx,dev_Ez,dev_bmx,dev_Psi_hyx,dev_amx,dev_bmx,dev_amx,dev_Psi_hxy,dev_kmx);

        // H_field_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex,float*Chxez,float*Chyez,float*Chyh,float*Chxh)
        //cudaMemcpy(dev_i,&i,sizeof(int),cudaMemcpyHostToDevice);
        E_field_update<<<grid,block>>>(dev_i,dev_Ez,dev_Hy,dev_Hx,dev_Psi_ezx,dev_aex,dev_aex,dev_bex,dev_bex,dev_Psi_ezy,dev_kex,dev_Cezhy,dev_Cezhy,dev_Ceze,dev_Cezeip,dev_Cezeic,dev_Ezic,dev_Ezip);

        calculate_JandM<<<gridNF2FF,blockNF2FF>>>(dev_freq, dev_i,dev_Ez,dev_Hy,dev_Hx,cjzxp,cjzyp,cjzxn,cjzyn,cmxyp,cmyxp,cmxyn,cmyxn);
        //float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,cuComplex *cjzxp,cuComplex *cjzyp,cuComplex*cjzxn,cuComplex*cjzyn,cuComplex*cmxyp,cuComplex*cmyxp,cuComplex*cmxyn,cuComplex*cmyxn,float*dev_Ezic,float*dev_Ezip,float*dev_Hx_inc,float*dev_Hy_inc
        //if(isscattering)
        //{
        //calculate_JandM_total<<<gridNF2FF,blockNF2FF>>>(dev_freq,dev_i,dev_Ez,dev_Hy,dev_Hx,cjzxp_tot,cjzyp_tot,cjzxn_tot,cjzyn_tot,cmxyp_tot,cmyxp_tot,cmxyn_tot,cmyxn_tot,dev_Ezic,dev_Ezip,dev_Hx_inc,dev_Hy_inc);
        //}
        //		unsigned char green = 128+127*buffer_Ez/0.4;
        /*ptr[offset].x = 0;
          ptr[offset].y = green;
          ptr[offset].z = 0;
          ptr[offset].w = 255;*///OpenGL stuff
        //	cudaMemcpy(&test_Ez,&dev_Ez[getCell(nx/2,ny/2,nx+1)],sizeof(float),cudaMemcpyDeviceToHost);
        //cudaMemcpy(&test_Ez,(dev_Ez+5000),sizeof(float),cudaMemcpyDeviceToHost);
        //cudaMemcpy(&test_Ez_2,&dev_Ez[getCell(nx/2-100,ny/2,nx+1)],sizeof(float),cudaMemcpyDeviceToHost);
        //cudaMemcpy(&test_Ez,(dev_Ez+getCell(nx/2,ny/2,nx+1)),sizeof(float),cudaMemcpyDeviceToHost);
        //cout<<"Ez (V/m) "<<test_Ez<<" "<<i<<endl;
        //myfile<<test_Ez<<"          "<<test_Ez_2<<endl;
        //if(i==0)
        //{
        //	//cudaMemcpy(Ez,dev_Ez,sizeof(float)*(nx+1)*(ny+1),cudaMemcpyDeviceToHost);
        //	for(int ind = 0;ind<(nx+1)*(ny+1);ind++)
        //	{
        //	fields>>test_Ez;
        //	Ez[ind] = Ez[ind]-test_Ez;// before this operation, Ez[ind] is the total field.  After this step, Ez[ind] is just the scattered field
        //	}
        //	for(int x = 0;x<=nx;x++)
        //	{
        //		for(int y =0;y<=ny;y++)
        //		{
        //		//Temp.Red = 255*((eps_r_z[getCell(x,y,nx+1)]-1))/25;
        //		Temp.Red =0;
        //		Temp.Green = (ebmpBYTE)(127+128*(Ez[getCell(x,y,nx+1)])/(1));
        //	//	Temp.Green = 0;
        //	//	if((x == f2x)||(y==f2y))
        //		//{
        //		//	Temp.Blue = (ebmpBYTE)254;
        //		//}
        //		//else
        //		//{
        //			Temp.Blue = 255*sqrt(sigma_e_z[getCell(x,y,nx+1)]-1)/sqrt(10000.0);
        //		//}
        //		//Temp.Blue = 0; 
        //		Temp.Alpha = 0;
        //		/*if((sqrt(((float)x-f1x)*((float)x-f1x)+((float)y-f1y)*((float)y-f1y))+sqrt(((float)x-f2x)*((float)x-f2x)+((float)y-f2y)*((float)y-f2y)))>500)
        //		{
        //			Temp.Green = (ebmpBYTE)0;
        //		}*/
        //		Scattered_Field_snapshot.SetPixel(x,y,Temp);
        //	

        //		}
        //	}
        //	for(y  = 0;y<=nx;y++)
        //	{
        //		myfile<<Ez[getCell(f2x,y,nx+1)]<<" ";
        //	}
        //	myfile.close();

        //	
        //	Scattered_Field_snapshot.WriteToFile("Target_image.bmp");
        //}

        //if(!(i%20 - 4))
        //{
        //	cudaMemcpy(Ez,dev_Ez,sizeof(float)*(nx+1)*(ny+1),cudaMemcpyDeviceToHost);
        //	for(int x = 0;x<=nx;x++)
        //	{
        //		for(int y = 0;y<=ny;y++)
        //		{
        //			

        //			radius = sqrt((x-nx/2)*dx*(x-nx/2)*dx+(y-ny/2)*dy*dy*(y-ny/2));
        //			Temp.Red = (ebmpBYTE)255*((eps_r_z[getCell(x,y,nx+1)]-1)/60);
        //			Temp.Green = (ebmpBYTE)(127+128*(Ez[getCell(x,y,nx+1)])/(0.2));
        //			if(isOnxp(x)||isOnyp(x,y)||isOnyn(x,y)||isOnxn(x))
        //			{
        //				Temp.Green = 0;
        //			}
        //			Temp.Blue = 0;
        //			
        //			//if(radius>breast_radius)
        //			//{
        //			//	Temp.Blue = 50;
        //			//}
        //			Temp.Alpha =0;
        //			Output_Image.SetPixel(x,y,Temp);
        //			
        //		
        //		}
        //	}
        //			ostringstream convert;
        //			convert << i/20;
        //			outputfilename = "Current_source_times_r_"+convert.str()+".bmp";
        //			
        //			Output_Image.WriteToFile(outputfilename.c_str());
        //}

        //cout<<"Ez (V/m) "<<test_Ez<<" "<<i<<endl;
        //cout<<fwf( i,nx, ny, 0)<<endl;
    }
    hipMemcpy(hcjzyn,cjzyn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcjzxp,cjzxp,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
    hipMemcpy(hcjzyp,cjzyp,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcjzxn,cjzxn,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
    hipMemcpy(hcmxyn,cmxyn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcmyxp,cmyxp,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
    hipMemcpy(hcmxyp,cmxyp,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcmyxn,cmyxn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);

    hipMemcpy(hcjzyn_tot,cjzyn_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcjzxp_tot,cjzxp_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
    hipMemcpy(hcjzyp_tot,cjzyp_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcjzxn_tot,cjzxn_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
    hipMemcpy(hcmxyn_tot,cmxyn_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcmyxp_tot,cmyxp_tot,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
    hipMemcpy(hcmxyp_tot,cmxyp_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
    hipMemcpy(hcmyxn_tot,cmyxn_tot,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);

    hipComplex *L,*N;
    for(int i=0;i<numberofobservationangles;i++)
    {
        //cout<<"hcmyxp = "<<hcmyxp[i].r<<" + i"<<hcmyxp[i].i<<endl;
    }

    L  = (hipComplex*)malloc(sizeof(hipComplex)*size_NF2FF_total);
    N = (hipComplex*)malloc(sizeof(hipComplex)*size_NF2FF_total);

    CJ_Init(L,size_NF2FF_total);
    CJ_Init(N,size_NF2FF_total);

    N2FPostProcess(D, freq,N,L,hcjzxp,hcjzyp,hcjzxn,hcjzyn,hcmxyp,hcmyxp,hcmxyn,hcmyxn);
    CJ_Init(L,size_NF2FF_total);
    CJ_Init(N,size_NF2FF_total);
    D_tot = (float*)malloc(sizeof(float)*numberofobservationangles);
    N2FPostProcess(D_tot,freq,N,L,hcjzxp_tot,hcjzyp_tot,hcjzxn_tot,hcjzyn_tot,hcmxyp_tot,hcmyxp_tot,hcmxyn_tot,hcmyxn_tot);
    float measurement[numberofobservationangles] = {0.38446 , 0.362389 , 0.309065 , 0.237687 , 0.162638 , 0.101565 , 0.0642376 , 0.0457471 , 0.0406768 , 0.0462104 , 0.0534992 , 0.0586805 , 0.0681197 , 0.0845823 , 0.105639 , 0.130494 , 0.15567 , 0.169704 , 0.162106 , 0.135797 , 0.102823 , 0.0717831 , 0.0478674 , 0.0364377 , 0.0385978 , 0.0501895 , 0.067232 , 0.0870665 , 0.10573 , 0.118834 , 0.123803 , 0.11963 , 0.106446 , 0.087042 , 0.0667677 , 0.0498503 , 0.0385427 , 0.0365824 , 0.0478755 , 0.0714402 , 0.102585 , 0.135752 , 0.161719 , 0.169557 , 0.156789 , 0.13172 , 0.104872 , 0.0826252 , 0.0674893 , 0.0588946 , 0.0528211 , 0.0459529 , 0.0410625 , 0.0449233 , 0.0647455 , 0.105435 , 0.165446 , 0.237711 , 0.310644 , 0.365682 };
    //measurement = (float*)malloc(sizeof(float)*numberofobservationangles);

    //float Phi;

    //	for(int i = 0;i<numberofobservationangles;i++)
    //	{
    //	Phi = 360*(float)i/numberofobservationangles;
    //	cout<<"D "<<D[i]<<" Phi = "<<Phi<<endl;
    //}
    //	for(int i=0;i<numberofobservationangles;i++)
    //	{
    //	cout<<"L = "<<L[i].r<<" + i"<<L[i].i<<" "<<i<<endl;
    //	}
    //for(int i=0;i<numberofobservationangles;i++)
    //	{
    //	cout<<"N = "<<N[i].r<<" + i"<<N[i].i<<" "<<i<<endl;
    //	}

    float fit;
    fit=fitness(D,numberofobservationangles, measurement);
    //ofstream D_file;
    //ofstream D_tot_file;
    //if(isscattering)
    //{
    //if(isPW)
    //	{
    //		D_tot_file.open("Total_fieldPW.txt");
    //		D_file.open("RCSPW.txt");
    //	}
    //	else
    //	{
    //		D_tot_file.open("Total_field.txt");
    //		D_file.open("RCS.txt");
    //	}

    //	for(int i = 0;i<numberofobservationangles;i++)
    //		{
    //			D_file<<D[i]<<" , ";
    //			D_tot_file<<D_tot[i]<<" ";

    //		}
    //	}
    //else
    //	{
    //	D_file.open("Directivity.txt");
    //	for(int i = 0;i<numberofobservationangles;i++)
    //	{
    //D_file<<D[i]<<" , ";
    //	}
    //}

    /*cudaMemcpy(Hy,dev_Hy,sizeof(float)*(nx)*(ny),cudaMemcpyDeviceToHost);
      for(i = 0; i < (nx+1);i++) {
      for(j = 0; j <(ny+1);j++) {
      cout << Ez[getCell(i,j,nx+1)] << " ";
      }
      cout << endl;
      }
      cout << endl;*/
    //cout<<" fitness = "<<fit<<endl;
    error = hipGetLastError();
    //if(error != cudaSuccess) {
    //	printf("%s\n",cudaGetErrorString(error));
    //}
    //time(&end);
    //double dif = difftime(end,start);
    //cout<<"It took "<<dif<<" seconds"<<endl;
    //system("PAUSE");

    free(Ceze);
    free(Cezhy);
    free(Cezhx);
    free(Ez);
    free(eps_r_z);
    free(sigma_e_z);
    free(Hy);
    free(Hx);
    free(kex);
    free(aex);
    free(bex);
    free(amx);
    free(bmx);
    free(alpha_e);
    free(alpha_m);
    free(sigma_e_pml);
    free(sigma_m_pml);
    free(Psi_ezy);
    free(Psi_ezx);
    free(Psi_hyx);
    free(Psi_hxy);
    free(kmx);
    free(D);
    free(D_tot);
    free(Ezip);
    free(Ezic);
    free(Hy_inc);
    free(Hx_inc);
    free(Psi_ezy_inc);
    free(Psi_ezx_inc);
    free(Psi_hyx_inc);
    free(Psi_hxy_inc);
    free(hcjzxp);
    free(hcjzyp);
    free(hcjzxn);
    free(hcjzyn);
    free(hcmxyp);
    free(hcmyxp);
    free(hcmxyn);
    free(hcmyxn);
    free(hcjzxp_tot);
    free(hcjzyp_tot);
    free(hcjzxn_tot);
    free(hcjzyn_tot);
    free(hcmxyp_tot);
    free(hcmyxp_tot);
    free(hcmxyn_tot);
    free(hcmyxn_tot);
    free(Cezeic);
    free(Cezeip);
    free(L);
    free(N);
    //free(measurement);
    //float *Cezeic = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    //float *Cezeip = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));

    //float*Ceze,*Cezhy,*Cezhx,*dev_Cezeic,*dev_Cezeip,*Ez,*eps_r_z,*sigma_e_z,*Hy,*Hx,
    //	*kex,*aex,*bex,*amx,*bmx,*alpha_e,*alpha_m,*sigma_e_pml,*sigma_m_pml
    //	,*Psi_ezy,*Psi_ezx,*Psi_hyx,*Psi_hxy,*kmx;//*Cezj later if using loop current source
    //float* dev_sigma_e_z,*dev_eps_r_z;
    //float freq = center_freq;
    //float *dev_freq,*D,*D_tot;
    //float* Ezip,*Ezic,*dev_Ezip,*dev_Ezic,*Hy_inc,*Hx_inc,*dev_Hy_inc,*dev_Hx_inc,*dev_Psi_ezy_inc,*dev_Psi_ezx_inc,*dev_Psi_hyx_inc,*dev_Psi_hxy_inc,
    //	*Psi_ezy_inc,*Psi_ezx_inc,*Psi_hyx_inc,*Psi_hxy_inc;
    //
    //cuComplex *cjzxp,*cjzyp,*cjzxn,*cjzyn,*cmxyp,*cmyxp,*cmxyn,*cmyxn,*cjzxp_tot,*cjzyp_tot,*cjzxn_tot,*cjzyn_tot,*cmxyp_tot,*cmyxp_tot,*cmxyn_tot,*cmyxn_tot;
    //cuComplex *hcjzxp,*hcjzyp,*hcjzxn,*hcjzyn,*hcmxyp,*hcmyxp,*hcmxyn,*hcmyxn,*hcjzxp_tot,*hcjzyp_tot,*hcjzxn_tot,*hcjzyn_tot,*hcmxyp_tot,*hcmyxp_tot,*hcmxyn_tot
    //	,*hcmyxn_tot;


    hipFree(dev_Cezeic);
    hipFree(dev_Cezeip);
    hipFree(dev_sigma_e_z);
    hipFree(dev_eps_r_z);
    hipFree(dev_freq);
    hipFree(dev_Ezip);
    hipFree(dev_Ezic);
    hipFree(dev_Hy_inc);
    hipFree(dev_Hx_inc);
    hipFree(dev_Psi_ezy_inc);
    hipFree(dev_Psi_ezx_inc);
    hipFree(dev_Psi_hyx_inc);
    hipFree(dev_Psi_hxy_inc);
    hipFree(cjzxp);
    hipFree(cjzyp);
    hipFree(cjzxn);
    hipFree(cjzyn);
    hipFree(cmxyp);
    hipFree(cmyxp);
    hipFree(cmxyn);
    hipFree(cmyxn);
    hipFree(cjzxp_tot);
    hipFree(cjzyp_tot);
    hipFree(cjzxn_tot);
    hipFree(cjzyn_tot);
    hipFree(cmxyp_tot);
    hipFree(cmyxp_tot);
    hipFree(cmxyn_tot);
    hipFree(cmyxn_tot);
    hipFree(dev_Ceze);
    hipFree(dev_Cezhy);
    hipFree(dev_Cezhx);

    hipFree(dev_bex);
    hipFree(dev_aex);
    hipFree(dev_bmx);
    hipFree(dev_amx);
    hipFree(dev_kex);
    hipFree(dev_kmx);
    hipFree(dev_Ez);
    hipFree(dev_Hy);
    hipFree(dev_Hx);
    hipFree(dev_Psi_ezy);
    hipFree(dev_Psi_ezx);
    hipFree(dev_Psi_hyx);
    hipFree(dev_Psi_hxy);
    //float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_Jz,*dev_bex,*dev_aex,*dev_bmx,*dev_amx,*dev_kex,*dev_kmx;//dev_Cezj if using loop current source
    //float *dev_Ez,*dev_Hy,*dev_Hx;

    //float*dev_Psi_ezy,*dev_Psi_ezx,*dev_Psi_hyx,*dev_Psi_hxy;

    cout << "fitness is: " << fit << endl;
    return (double)fit;
}

__global__ void scattered_parameter_init(float*eps_r_z,float*sigma_e_z,float*Cezeic,float*Cezeip)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    if(x<(nx+1)&&y<(ny+1))
    {
        Cezeic[dgetCell(x,y,nx+1)] = (2*(eps0-eps0*eps_r_z[dgetCell(x,y,nx+1)])-sigma_e_z[dgetCell(x,y,nx+1)]*dt)/(2*eps0*eps_r_z[dgetCell(x,y,nx+1)]+sigma_e_z[dgetCell(x,y,nx+1)]*dt);
        Cezeip[dgetCell(x,y,nx+1)] = -1*(2*(eps0-eps0*eps_r_z[dgetCell(x,y,nx+1)])+sigma_e_z[dgetCell(x,y,nx+1)]*dt)/(2*eps0*eps_r_z[dgetCell(x,y,nx+1)]+sigma_e_z[dgetCell(x,y,nx+1)]*dt);

    }
}

int getCell(int x, int y,int size)//size will just be the width in the x dimension of the array.
{
    return x+y*size;
}

float* Make2DfloatArray(int arraySizeX, int arraySizeY)
{
    float* theArray;
    theArray = (float*) malloc(arraySizeX*arraySizeY*sizeof(float*));

    return theArray;
} 

void waveform_time_init(float*time1)
{

    int size = number_of_time_steps;
    for(int i = 0;i<size;i++) 
    {
        time1[i]=(float)i*dt;
    }
}

void Jz_waveform(float * time,float*Jz_impressed)
{
    float w = 2*PI*center_freq;//center_freq is the frequency
    for(int i = 0;i<number_of_time_steps;i++)
    {
        Jz_impressed[i]= 10*sin(w*time[i]);
        //Jz_impressed[i]=exp(-1*((time[i]-2e-10)/5e-11)*(time[i]-2e-10)/(5e-11));

    }
}

void Ceze_init(float * eps_r_z, float* sig_e_z, float* Ceze)
{
    int size = nx+1;
    for(int j=0;j<ny+1;j++)
    {
        for(int i=0;i<size;i++)
        {
            Ceze[getCell(i,j,nx+1)] = (2*eps_r_z[getCell(i,j,nx+1)]*eps0-dt*sig_e_z[getCell(i,j,nx+1)])/(2*eps_r_z[getCell(i,j,nx+1)]*eps0+dt*sig_e_z[getCell(i,j,nx+1)]);
        }
    }
}

void Cezhy_init(float*eps_r_z, float* sigma_e_z,float* Cezhy,float*kex)
{
    int size = nx+1;
    for(int j =0;j<ny+1;j++)
    {
        for(int i=0;i<size;i++)
        {
            Cezhy[getCell(i,j,size)] = (2*dt/dx)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);

        }
    }
}

void Cezhx_init(float* eps_r_z,float*sigma_e_z,float*Cezhx,float*kex)
{
    int size=nx+1;
    for(int j=0;j<ny+1;j++)
    {
        for(int i =0;i<nx+1;i++)
        {
            Cezhx[getCell(i,j,size)]=(2*dt/dy)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);

        }
    }
}

void Cezj_init(float*eps_r_z,float*sigma_e_z,float*Cezj)
{
    int size =nx+1;
    for(int j=0;j<ny+1;j++)
    {
        for(int i=0;i<nx+1;i++)
        {
            Cezj[getCell(i,j,size)] = (-2*dt)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);

        }
    }
}

void Ez_init(float*Ez)
{
    int size=nx+1;
    for(int j = 0;j<ny+1;j++)
    {
        for(int i = 0;i<nx+1;i++)
        {
            Ez[getCell(i,j,size)] = (float)0;
        }
    }
}

/*void Jz_init(float*Jz)
  {
  for(int j =0;j<ny+1;j++)
  {
  for(int i = 0;i<nx+1;i++)
  {
  Jz[getCell(i,j,nx+1)] = 0;
  }
  }
  }*/

void Chyh_init(float*mu_r_y,float*sigma_m_y,float*Chyh)
{
    int size=nx;
    for(int i = 0;i<nx;i++)
        for(int j =0;j<ny;j++)
        {
            {
                Chyh[getCell(i,j,size)] = (2*mu_r_y[getCell(i,j,size)]*mu0-dt*sigma_m_y[getCell(i,j,size)])/(2*mu_r_y[getCell(i,j,size)]*mu0+dt*sigma_m_y[getCell(i,j,size)]);
            }
        }
}

void Chxh_init(float*mu_r_x,float*sigma_m_x,float*Chxh)
{
    int size=nx;
    for(int i = 0;i<nx;i++)
        for(int j =0;j<ny;j++)
        {
            {
                Chxh[getCell(i,j,size)] = (2*mu_r_x[getCell(i,j,size)]*mu0-dt*sigma_m_x[getCell(i,j,size)])/(2*mu_r_x[getCell(i,j,size)]*mu0+dt*sigma_m_x[getCell(i,j,size)]);
            }
        }
}

void Chyez_init(float*mu_r_y,float*sigma_m_y,float*Chyez)
{
    int size = nx;
    for(int j =0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Chyez[getCell(i,j,size)] = (2*dt/dx)/(2*mu_r_y[getCell(i,j,size)]*mu0+dt*sigma_m_y[getCell(i,j,size)]);
        }
    }
}

void Chxez_init(float*mu_r_x,float*sigma_m_x,float*Chxez)
{
    int size = nx;
    for(int j =0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Chxez[getCell(i,j,size)] = (2*dt/dy)/(2*mu_r_x[getCell(i,j,size)]*mu0+dt*sigma_m_x[getCell(i,j,size)]);
        }
    }
}

/*void Chym_init(float*mu_r_y,float*sigma_m_y,float*Chym)
  {
  int size = nx;
  for(int j =0;j<ny;j++)
  {
  for(int i = 0;i<size;i++)
  {
  Chym[getCell(i,j,size)] = (-2*dt)/(2*mu_r_y[getCell(i,j,size)]*mu0+dt*sigma_m_y[getCell(i,j,size)]);

  }
  }
  }
  void Chxm_init(float*mu_r_x,float*sigma_m_x,float*Chxm)
  {
  int size = nx;
  for(int j =0;j<ny;j++)
  {
  for(int i = 0;i<size;i++)
  {
  Chxm[getCell(i,j,size)] = (-2*dt)/(2*mu_r_x[getCell(i,j,size)]*mu0+dt*sigma_m_x[getCell(i,j,size)]);

  }
  }
  }*/

void eps_r_z_init(float * eps_r_z,const vector<float> &argument)
{
    int size = nx+1;
    float radius,tumor_radius,tumor_radius_2,tumor_radius_3;
    for(int j =0;j<ny+1;j++)
    {
        for(int i = 0;i<nx+1;i++)
        {


            eps_r_z[getCell(i,j,size)] = 1;
            radius = sqrt(pow( ((float)i-nx/2)*dx,2) + pow( ((float)j-ny/2)*dy,2));
            if(radius<=breast_radius)
            {
                eps_r_z[getCell(i,j,size)] = (float)argument.at(getOptimizationCell(i,j));
                //eps_r_z[getCell(i,j,size)] = 10;

            }


            //radius = sqrt(((float)i-(target_x))*dx*((float)i-(target_x))*dx+((float)j-(target_y))*dy*((float)j-target_y)*dy);
            //tumor_radius = sqrt(((float)i-(target_x-25))*dx*((float)i-(target_x-25))*dx+((float)j-(target_y+50))*dy*((float)j-(target_y+50))*dy);
            //tumor_radius_2 = sqrt(((float)i-(target_x+25))*dx*((float)i-(target_x+25))*dx+((float)j-(target_y-50))*dy*((float)j-(target_y-50))*dy);
            //tumor_radius_3 = sqrt(((float)i-(target_x-25))*dx*((float)i-(target_x-25))*dx+((float)j-(target_y-25))*dy*((float)j-(target_y-25))*dy);
            //if(radius>breast_radius)
            //{
            //	eps_r_z[getCell(i,j,size)] = 1;

            //}
            //
            //if(radius>breast_radius)
            //{
            //	eps_r_z[getCell(i,j,size)] = 1;
            //	//cout<<"eps_r_z = "<<eps_r_z[getCell(i,j,size)]<<" (i,j) = ("<<i<<","<<j<<")"<<endl;
            //}
            //else if(i>=(nx/2-108)&&i<(nx/2+108)&&j>=(ny/2-108)&&j<(ny/2+108))
            //{
            //	eps_r_z[getCell(i,j,size)] = (float)argument.at(getOptimizationCell(i,j));
            ////	cout<<"eps_r_z = "<<eps_r_z[getCell(i,j,size)]<<" (i,j) = ("<<i<<","<<j<<")"<<endl;
            //}

        }
    }
}

void sigma_e_z_init(float * sigma_e_z,float*sigma_e_pml, const vector<float> &argument)
{
    int size = nx+1;
    float radius;


    for(int j =0;j<ny+1;j++)
    {
        for(int i = 0;i<nx+1;i++)
        {
            sigma_e_z[getCell(i,j,size)] = 0;
            radius = sqrt(pow( ((float)i-nx/2)*dx,2) + pow( ((float)j-ny/2)*dy,2));
            if(radius<=breast_radius)
            {
                sigma_e_z[getCell(i,j,size)] = (float)argument.at(getOptimizationCell(i,j)+9*9);
                //eps_r_z[getCell(i,j,size)] = 10;

            }
            //if(sqrt(pow((float)i-f2x,2)*dx+pow((float)j-f2y,2)*dy)<(100*dx))
            //{
            //	sigma_e_z[getCell(i,j,size)] = 0.00;
            //}
            //else
            //{
            //	sigma_e_z[getCell(i,j,size)] = 0;
            //}
            //	radius = sqrt(((float)i-nx/2)*dx*((float)i-nx/2)*dx+((float)j-ny/2)*dy*((float)j-ny/2)*dy);
            //if(radius>breast_radius)
            //{
            //	sigma_e_z[getCell(i,j,size)] = 0;
            //	//cout<<"sigma_e_z = "<<sigma_e_z[getCell(i,j,size)]<<" (i,j) = ("<<i<<","<<j<<")"<<endl;
            //}
            //else if(i>=(nx/2-108)&&i<(nx/2+108)&&j>=(ny/2-108)&&j<(ny/2+108))
            //{
            //	sigma_e_z[getCell(i,j,size)] = (float)argument.at(getOptimizationCell(i,j)+9*9);//total of 81 optimization cells
            ////	cout<<"sigma_e_z = "<<sigma_e_z[getCell(i,j,size)]<<" (i,j) = ("<<i<<","<<j<<")"<<endl;
            //}

        }
    }
}

void Hy_init(float*Hy)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Hy[getCell(i,j,size)] = 0;
        }
    }
}

void Hx_init(float*Hx)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Hx[getCell(i,j,size)] = 0;
        }
    }
}

void My_init(float*My)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            My[getCell(i,j,size)] = 0;
        }
    }
}

void Mx_init(float*Mx)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            Mx[getCell(i,j,size)] = 0;
        }
    }
}

void mu_r_y_init(float*mu_r_y)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            mu_r_y[getCell(i,j,size)] =1.000;
        }
    }

}

void mu_r_x_init(float*mu_r_x)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            mu_r_x[getCell(i,j,size)]=1.000;
        }
    }

}

void sigma_m_y_init(float*sigma_m_y)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            sigma_m_y[getCell(i,j,size)] = 0;
        }
    }
}

void sigma_m_x_init(float*sigma_m_x)
{
    int size = nx;
    for(int j=0;j<ny;j++)
    {
        for(int i = 0;i<size;i++)
        {
            sigma_m_x[getCell(i,j,size)] = 0;
        }
    }
}

void C_Psi_ezy_init(float *C_Psi_ezy,float*Cezhx)
{
    int size = 20;
    for(int j = 0;j<ny;j++)
        for( int i =0;i<size;i++)
        {
            if(i<10)
            {
                C_Psi_ezy[getCell(i,j,size)]=dy*Cezhx[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_ezy[getCell(i,j,size)]=dy*Cezhx[getCell(nx-20+i,j,nx)];
            }
        }
}

void C_Psi_ezx_init(float* C_Psi_ezx,float*Cezhy)
{
    int size_y=20;
    for(int j=0;j<size_y;j++)
    {
        for(int i=0;i<nx;i++)
        {
            if(j<10)
            {
                C_Psi_ezx[getCell(i,j,nx)] = dx*Cezhy[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_ezx[getCell(i,j,nx)] = dx*Cezhy[getCell(i,ny-20+j,nx)];
            }
        }
    }
}

void C_Psi_hyx_init(float*C_Psi_hyx,float*Chyez)
{
    int size_x=20;
    for(int j=0;j<ny;j++)
    {
        for(int i=0;i<size_x;i++)
        {
            if(i<10)
            {
                C_Psi_hyx[getCell(i,j,size_x)]=dx*Chyez[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_hyx[getCell(i,j,size_x)]=dx*Chyez[getCell(nx-20+i,j,nx)];
            }
        }
    }
}

void C_psi_hxy_init(float *C_Psi_hxy,float*Chxez)
{
    int size_y=20;
    for(int j=0;j<size_y;j++)
    {
        for(int i=0;i<nx;i++)
        {
            if(j<11)
            {
                C_Psi_hxy[getCell(i,j,nx)]=dy*Chxez[getCell(i,j,nx)];
            }
            else
            {
                C_Psi_hxy[getCell(i,j,nx)]=dy*Chxez[getCell(i,ny-20+j,nx)];
            }
        }
    }
}

void aex_init(float*aex,float*sigma_e_pml,float*kex,float*alpha_e_x,float*bex)
{
    int size=ncells;
    //aex[0]=0.0;
    //cout<<"aex[0] = "<<aex[0]<<endl;
    for(int i=0;i<size;i++)
    {
        aex[i]=((bex[i]-1)*sigma_e_pml[i])/(dx*(sigma_e_pml[i]*kex[i]+alpha_e_x[i]*kex[i]*kex[i]));
        //cout<<"aex["<<i<<"] = "<<aex[i]<<endl;
    }
}

void bex_init(float*bex ,float*sigma_e_pml,float*kex,float*alpha_e_x)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        bex[i]=exp(-1*(dt/eps0)*(sigma_e_pml[i]/kex[i]+alpha_e_x[i]));
    }
}

void aey_init(float*aey,float*sigma_e_pml,float*key,float*alpha_e_y,float*bey)
{
    for(int i=0;i>ncells;i++)
    {
        aey[i]=(bey[i]-1)*sigma_e_pml[i]/(dy*(sigma_e_pml[i]*key[i]+alpha_e_y[i]*key[i]*key[i]));
    }
}

void bey_init(float*bey,float*sigma_e_pml,float*key,float*alpha_e_y)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        bey[i]=exp(-1*(dt/eps0)*(sigma_e_pml[i]/key[i]+alpha_e_y[i]));
    }
}

void amy_init(float*amy,float*sigma_m_pml,float*kmy,float*alpha_m_y,float*bmy)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        amy[i]=(bmy[i]-1)*sigma_m_pml[i]/(dx*(sigma_m_pml[i]*kmy[i]+alpha_m_y[i]*kmy[i]*kmy[i]));
    }
}

void bmy_init(float*bmy,float*sigma_m_pml,float*kmy,float*alpha_m_y)
{
    int size=ncells;
    for(int i=0;i<size;i++)
    {
        bmy[i]=exp(-1*(dt/mu0)*(sigma_m_pml[i]/kmy[i]+alpha_m_y[i]));
    }
}

void amx_init(float*amx,float*sigma_m_pml,float*kmx,float*alpha_m_x,float*bmx)
{
    int size=ncells;

    //cout<<" amx = "<<amx[0]<<endl;
    //amx[0]=0.0;
    //cout<<" amx = "<<amx[0]<<endl;
    for(int i=0;i<size;i++)
    {
        amx[i]=(bmx[i]-1)*sigma_m_pml[i]/(dx*(sigma_m_pml[i]*kmx[i]+alpha_m_x[i]*kmx[i]*kmx[i]));
        //	cout<<" amx = "<<amx[i]<<endl;
    }
}

void bmx_init(float*bmx,float*sigma_m_pml,float*kmx,float*alpha_m_x)
{
    int size=10;
    float argument;
    //float constant;
    for(int i=0;i<size;i++)
    {
        //constant = dt/mu0;
        //cout<< "dt/mu0 = "<<constant<<endl;
        argument = -1*(dt/mu0)*((sigma_m_pml[i]/kmx[i])+alpha_m_x[i]);
        bmx[i]=exp(argument);
        //cout<<"argument of bmx = "<<argument<<endl;
        //cout<<"bmx = "<<bmx[i]<<endl;
    }
}

void alpha_e_init(float*alpha_e)
{
    float rho;
    int size=ncells;
    for(int i=0;i<ncells;i++)
    {
        rho = ((float)i+0.25)/ncells;
        alpha_e[i]=alpha_min+(alpha_max-alpha_min)*rho;
        //	cout<<"alpha_e = "<<alpha_e[i]<<endl;
    }
}

void alpha_m_init(float*alpha_e,float*alpha_m)
{
    int size=ncells;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.75)/ncells;
        alpha_m[i]=(mu0/eps0)*(alpha_min+(alpha_max-alpha_min)*rho);
        //cout<<"alpha_m = "<<alpha_m[i]<<endl;
    }
}

void k_e_init(float*k)
{
    int size=ncells;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.25)/ncells;
        k[i]=pow(rho,npml)*(kmax-1)+1;
        //cout<<"k ["<<i<<"]= "<<k[i]<<endl;

    }
}

void k_m_init(float*k)
{
    int size=ncells;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.75)/ncells;
        k[i]=pow(rho,npml)*(kmax-1)+1;
        //cout<<"k ["<<i<<"]= "<<k[i]<<endl;

    }
}

void sigma_e_pml_init(float* sigma_e_pml)  
{
    float sigma_max = (npml+1)/(150*PI*dx);
    int size = 10;
    float rho;
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.25)/ncells;
        sigma_e_pml[i]=sigma_max*sigma_factor*pow(rho,npml);
        //cout<<"sigma_e_pml = "<<sigma_e_pml[i]<<endl;
    }
}

void sigma_m_pml_init(float*sigma_m_pml,float*sigma_e_pml)
{
    float rho;
    int size = 10;
    float sigma_max = (npml+1)/(150*PI*dx);
    for(int i=0;i<size;i++)
    {
        rho = ((float)i+0.75)/ncells;
        sigma_m_pml[i]=(mu0/eps0)*sigma_max*sigma_factor*pow(rho,npml);
        //cout<<"sigma_m_pml "<<sigma_m_pml[i]<<endl;
    }
}

void Psi_ezy_init(float*Psi_ezy)
{  
    int size=nx*20;
    for(int i=0;i<size;i++)
    {
        Psi_ezy[i]=0.0;
    }
}

void Psi_ezx_init(float*Psi_ezx)
{
    int size=ny*20;
    for(int i=0;i<size;i++)
    {
        Psi_ezx[i]=0.0;
    }
}

void Psi_hyx_init(float*Psi_hyx)
{
    int size=ny*20;
    for(int i=0;i<size;i++)
    {
        Psi_hyx[i]=0.0;
    }
}

void Psi_hxy_init(float*Psi_hxy)
{
    int size=nx*20;  
    for(int i=0;i<size;i++)
    {
        Psi_hxy[i]=0.0;
    }
}

void CJ_Init(hipComplex * cjzyn,int size)
{
    hipComplex nullComplex(0,0);
    for( int i =0; i<size;i++)
    {
        cjzyn[i] = nullComplex;
    }
}
