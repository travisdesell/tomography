#include "hip/hip_runtime.h"
//#define GLEW_STATIC
//#pragma comment(lib,"glew32.lib")
//#include <windows.h>
//#include <gl/glew.h>
//#include <glut.h>
#include <complex>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <stdlib.h>
#include <fstream>
#include <cstdlib>
#include <fstream>
#include <hip/hip_runtime.h>
//#include "stdafx.h"
#include <iomanip>
#include <time.h>
//#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_complex.h>
#include <vector>
#include <math_functions.h>
//#include "EasyBMP.h"
//#include "EasyBMP_DataStructures.h"
//#include "EasyBMP_VariousBMPutilities.h"

#include "FDTD_common.hxx"

//#include <unistd.h>
//const hipComplex jcmpx (0.0, 1.0);
/*static void HandleError( hipError_t err, const char *file,  int line ) {
  if (err != hipSuccess) {
  printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
  exit( EXIT_FAILURE );
  }
  }*/


//__constant__ float*dev_Ceze,*dev_Cezhy,*dev_Cezhx,*dev_Cezj,*dev_Jz,*dev_Chyh,*dev_Chxh,*dev_Chyez,*dev_Chxez,*dev_bex,*dev_bey,*dev_aex,*dev_aey,*dev_bmy,*dev_bmx,*dev_amy,*dev_amx,*dev_C_Psi_ezy,
//*dev_C_Psi_ezx,*dev_C_Psi_hxy,*dev_C_Psi_hyx;
struct hipComplex {
    float   r;
    float   i;
    __host__  __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __host__ __device__ hipComplex(float a): r(a), i(0) {}
    float magnitude2( void ) { return r * r + i * i; }
    __host__  __device__  hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __host__ __device__ hipComplex operator*(const float& a){
        return hipComplex(r*a,i*a);
    }

    __host__  __device__  hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
    __host__ __device__ hipComplex operator+(const float& a){
        return hipComplex(r+a,i);
    }
    __host__ __device__ void operator+=(const float& f){
        r += f;
    }
    __host__ __device__ void operator+=(const hipComplex& C);
    hipComplex();
};

__host__ __device__ hipComplex operator*(const float &f, const hipComplex &C)
{

    return hipComplex(C.r*f,C.i*f);
}

__host__ __device__ void hipComplex::operator+=(const hipComplex& C)
{
    r +=C.r;
    i += C.i;
}

__host__ __device__ float cuabs(hipComplex x)
{
    return sqrt(x.i*x.i + x.r*x.r);
}

__host__ __device__ hipComplex cuexp(hipComplex arg)
{
    hipComplex res(0,0);
    float s, c;
    float e = expf(arg.r);
    sincosf(arg.i,&s,&c);
    res.r = c * e;
    res.i = s * e;
    return res;

}

__device__ int isOnNF2FFBound(int x, int y)
{
    if(x==NF2FFdistfromboundary||x==nx-NF2FFdistfromboundary||y==NF2FFdistfromboundary||y==ny-NF2FFdistfromboundary)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ int getxfromthreadIdNF2FF(int index)
{
    int x=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))//yn
    {
        x = index+NF2FFdistfromboundary+1;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))//xp
    {
        x = nx-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))//yp
    {
        x = nx-NF2FFdistfromboundary  - (index-(nx-4*NF2FFdistfromboundary+ny-2))-2;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))//xn notice 2*nx-8*NF2FFdistfromboundary+2*ny-4 is the max index term.
    {
        x = NF2FFdistfromboundary;
    }
    return x;
}

__device__ int getyfromthreadIdNF2FF(int index)
{
    int y=0;
    if(index<(nx-2*NF2FFdistfromboundary-2))
    {
        y = NF2FFdistfromboundary;
    }
    else if(index<(nx-4*NF2FFdistfromboundary+ny-2))
    {
        y = (index-(nx-2*NF2FFdistfromboundary-2))+NF2FFdistfromboundary;
    }
    else if(index<(2*nx-6*NF2FFdistfromboundary+ny-4))
    {
        y = ny-NF2FFdistfromboundary-1;
    }
    else if(index<(2*nx-8*NF2FFdistfromboundary+2*ny-4))
    {
        y = ny-NF2FFdistfromboundary-(index-(2*nx-6*NF2FFdistfromboundary+ny-4))-1;
    }
    return y;
}
__device__ __host__ int isOnxn(int x)
{
    if(x==(NF2FFdistfromboundary))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnxp(int x)
{
    if(x==(nx-NF2FFdistfromboundary-1))
    { 
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnyp(int x,int y)
{
    if(y==(ny-NF2FFdistfromboundary-1)&&!isOnxn(x)&&!isOnxp(x))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

__device__ __host__ int isOnyn(int x, int y)
{
    if((y==(NF2FFdistfromboundary))&&!isOnxn(x)&&!(isOnxp(x)))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}


__global__ void calculate_JandM(float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex*cjzxn,hipComplex*cjzyn,hipComplex*cmxyp,hipComplex*cmyxp,hipComplex*cmxyn,hipComplex*cmyxn)
{
    float freq = *f;
    int index = threadIdx.x+blockIdx.x*blockDim.x;// should launch 2*nx-8*NF2FFdistfromboundary+2*ny-4 threads. 
    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(isOnyp(x,y))
        {
            Ez = (dev_Ez[getCell(x,y+1,nx+1)]+dev_Ez[getCell(x,y,nx+1)])/2;
            float Hx = dev_Hx[getCell(x,y,nx)];
            cjzyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Hx*deltatime*cuexp((float)(-1)*j*(float)2*pi*freq*(float)(*timestep)*deltatime);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements

            cmxyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Ez*deltatime*cuexp((float)-1.0*j*(float)2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
        else if(isOnxp(x))//X faces override y faces at their intersections
        {
            Ez = (dev_Ez[getCell(x,y,nx+1)]+dev_Ez[getCell(x+1,y,nx+1)])/2;
            float Hy = dev_Hy[getCell(x,y,nx)];

            cjzxp[index-(nx-2*NF2FFdistfromboundary-2)] += Hy*deltatime*cuexp(-1*j*2*pi*freq*(float)(*timestep)*(float)dt);//cjzxp and cmyxp have ny-2*NF2FFBound elements

            cmyxp[index-(nx-2*NF2FFdistfromboundary-2)] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*pi*freq*((float)(*timestep)+0.5)*(float)dt);// this is the discrete fourier transform, by the way.
        }
        else if(isOnyn(x,y))
        {  
            Ez = (dev_Ez[getCell(x,y,nx+1)]+dev_Ez[getCell(x,y+1,nx+1)])/2;
            float Hx=dev_Hx[getCell(x,y,nx)];

            cjzyn[index] += Hx*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt);  //cjzyn and cmxyn need to have nx-2*NF2FFbound-2 elements
            cmxyn[index] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
        else if(isOnxn(x))
        {
            Ez = (dev_Ez[getCell(x,y,nx+1)]+dev_Ez[getCell(x+1,y,nx+1)])/2;
            cjzxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*dev_Hy[getCell(x,y,nx)]*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt); // cjzxn and cmyxn must have ny-2*NFdistfromboundary elements
            cmyxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Ez*(float)dt*cuexp(-1.0*j*2.0*(float)PI*freq*((float)(*timestep)+0.5)*(float)dt);
        }
    }

}


__host__ __device__ float fwf(float timestep,float x, float y,float Phi_inc,float l)
{

    float ar;
    float ky, kx;//k hat
    sincosf(Phi_inc,&ky,&kx);

    ar = (float)timestep*dt-(float)t0-(1/(float)c0)*(ky*y*dx+kx*x*dy-l);
    //ar = timestep*dt-t0;

    //return exp(-1*(ar*ar)/(tau*tau));// gaussian pulse  argument is k dot r, 
    return exp(-1*ar*ar/(tau*tau));
    //return sin(2*PI*1e9*timestep*dt);
}

__global__ void H_field_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex)
{
    float buffer_Hy;
    float buffer_Hx;
    float Chez = (dt/dx)/(mu0);
    int x = threadIdx.x +blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if(x<nx&&y<nx)
    {
        buffer_Hy = dev_Hy[getCell(x,y,nx)]+Chez*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
        buffer_Hx = dev_Hx[getCell(x,y,nx)]-Chez*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
        if(x<ncells)
        {
            buffer_Hy= dev_Hy[getCell(x,y,nx)]+Chez*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[ncells-1-x];
            dev_Psi_hyx[getCell(x,y,20)]=dev_bmx[ncells-1-x]*dev_Psi_hyx[getCell(x,y,20)]+dev_amx[ncells-1-x]*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[getCell(x,y,20)] ;
        } 
        if(x>=(nx-ncells))
        {
            buffer_Hy=dev_Hy[getCell(x,y,nx)]+Chez*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[x-nx+ncells];
            dev_Psi_hyx[getCell(x-nx+20,y,2*ncells)]=dev_bmx[x-nx+ncells]*dev_Psi_hyx[getCell(x-nx+20,y,20)]+dev_amx[x-nx+ncells]*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[getCell(x-nx+20,y,20)];
        }
        if(y<ncells)
        {
            buffer_Hx=dev_Hx[getCell(x,y,nx)]-Chez*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[ncells-1-y];
            dev_Psi_hxy[getCell(x,y,nx)]=dev_bmy[ncells-1-y]*dev_Psi_hxy[getCell(x,y,nx)]+dev_amy[ncells-1-y]*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[getCell(x,y,nx)];  
        }
        if(y>=(ny-ncells))
        {
            buffer_Hx=dev_Hx[getCell(x,y,nx)]-Chez*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[y-ny+ncells];
            dev_Psi_hxy[getCell(x,y-ny+20,nx)]=dev_bmy[y-ny+ncells]*dev_Psi_hxy[getCell(x,y-ny+20,nx)]+dev_amy[y-ny+ncells]*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[getCell(x,y-nx+20,nx)];
        }
        //__syncthreads();
        if(isnan(buffer_Hx)) 
        {
            dev_Hx[getCell(x,y,nx)] = 0.0;
        }
        else 
        {
            dev_Hx[getCell(x,y,nx)] = buffer_Hx;
        }

        if(isnan(buffer_Hy)) {
            dev_Hy[getCell(x,y,nx)] = 0.0;
        }
        else
        {
            dev_Hy[getCell(x,y,nx)] = buffer_Hy;
        }

        //dev_Hx[getCell(x,y,nx)] = buffer_Hx;
        //dev_Hy[getCell(x,y,nx)] = buffer_Hy;
    }
}

__global__ void H_inc_update(float*dev_Hy,float*dev_Hx,float*dev_Ez,float*dev_bmx,float*dev_Psi_hyx,float*dev_amx,float*dev_bmy,float*dev_amy,float*dev_Psi_hxy,float*kex)
{
    float buffer_Hy;
    float buffer_Hx;
    float Chez = (dt/dx)/(mu0);
    int x = threadIdx.x +blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if(x<nx&&y<nx)
    {
        buffer_Hy = dev_Hy[getCell(x,y,nx)]+Chez*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
        buffer_Hx = dev_Hx[getCell(x,y,nx)]-Chez*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
        if(x<ncells)
        {
            buffer_Hy= dev_Hy[getCell(x,y,nx)]+Chez*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[ncells-1-x];
            dev_Psi_hyx[getCell(x,y,20)]=dev_bmx[ncells-1-x]*dev_Psi_hyx[getCell(x,y,20)]+dev_amx[ncells-1-x]*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[getCell(x,y,20)] ;
        } 
        if(x>=(nx-ncells))
        {
            buffer_Hy=dev_Hy[getCell(x,y,nx)]+Chez*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[x-nx+ncells];
            dev_Psi_hyx[getCell(x-nx+20,y,2*ncells)]=dev_bmx[x-nx+ncells]*dev_Psi_hyx[getCell(x-nx+20,y,20)]+dev_amx[x-nx+ncells]*(dev_Ez[getCell(x+1,y,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hy+=Chez*dx*dev_Psi_hyx[getCell(x-nx+20,y,20)];
        }
        if(y<ncells)
        {
            buffer_Hx=dev_Hx[getCell(x,y,nx)]-Chez*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[ncells-1-y];
            dev_Psi_hxy[getCell(x,y,nx)]=dev_bmy[ncells-1-y]*dev_Psi_hxy[getCell(x,y,nx)]+dev_amy[ncells-1-y]*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[getCell(x,y,nx)];  
        }
        if(y>=(ny-ncells))
        {
            buffer_Hx=dev_Hx[getCell(x,y,nx)]-Chez*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)])/kex[y-ny+ncells];
            dev_Psi_hxy[getCell(x,y-ny+20,nx)]=dev_bmy[y-ny+ncells]*dev_Psi_hxy[getCell(x,y-ny+20,nx)]+dev_amy[y-ny+ncells]*(dev_Ez[getCell(x,y+1,nx+1)]-dev_Ez[getCell(x,y,nx+1)]);
            buffer_Hx-=Chez*dy*dev_Psi_hxy[getCell(x,y-nx+20,nx)];
        }
        //__syncthreads();
        if(isnan(buffer_Hx)) 
        {
            dev_Hx[getCell(x,y,nx)] = 0.0;
        }
        else 
        {
            dev_Hx[getCell(x,y,nx)] = buffer_Hx;
        }

        if(isnan(buffer_Hy)) {
            dev_Hy[getCell(x,y,nx)] = 0.0;
        }
        else
        {
            dev_Hy[getCell(x,y,nx)] = buffer_Hy;
        }

        //dev_Hx[getCell(x,y,nx)] = buffer_Hx;
        //dev_Hy[getCell(x,y,nx)] = buffer_Hy;
    }
}

__global__ void E_field_update(int *i,float*dev_Ez,float*dev_Hy,float*dev_Hx,float*dev_Psi_ezx,float*dev_aex,float*dev_aey,float*dev_bex,float*dev_bey,float*dev_Psi_ezy,float*kex,float*Cezhy,float*Cezhx,float*Ceze,float*Cezeip,float*Cezeic,float*Phi)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    //  int offset = x+y*blockDim.x*gridDim.x;
    float buffer_Ez;
    //float Ceh = (dt/dx)/(eps0);
    float Cezj = -dt/eps0;
    float length_offset;

    if(x<=nx&&y<=ny)
    {

        //if(x==0||x==nx||y==0||y==ny)
        if(x==nx||y==ny||x==0||y==0)
        {
            buffer_Ez=0.0;
        }
        else
        {
            if(isscattering)
            {

                buffer_Ez = Ceze[getCell(x,y,nx+1)]*dev_Ez[getCell(x,y,nx+1)]+Cezhy[getCell(x,y,nx+1)]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)])
                    -Cezhx[getCell(x,y,nx+1)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)])
                    +Cezeic[getCell(x,y,nx+1)]*fwf((float)(*i)+0.5,x-nx/2,y-ny/2,*Phi,-breast_radius)
                    +Cezeip[getCell(x,y,nx+1)]*fwf((float)(*i)-0.5,x-nx/2,y-ny/2,*Phi,-breast_radius);

            }
            else
            {
                buffer_Ez = Ceze[getCell(x,y,nx+1)]*dev_Ez[getCell(x,y,nx+1)]+Cezhy[getCell(x,y,nx+1)]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)])
                    -Cezhx[getCell(x,y,nx+1)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)]);
                if(x==(int)(source_x)&&y==(int)(source_y))
                {
                    buffer_Ez=buffer_Ez + 100*Cezj*fwf((float)(*i),0,0,0,0);
                }
            }

            //if(x==((int)nx/2)&&y==((int)nx/2))
            //{
            //	//buffer_Ez=buffer_Ez + Cezj*dev_Jz[*i];
            //	buffer_Ez=buffer_Ez + Cezj*fwf((float)(*i),0,0,0,0);
            //}
            if(x<=ncells&&x!=0)
            {
                buffer_Ez = Ceze[getCell(x,y,nx+1)]*dev_Ez[getCell(x,y,nx+1)]+Cezhy[getCell(x,y,nx+1)]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)])/kex[ncells-x]
                    -Cezhx[getCell(x,y,nx+1)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)])/kex[ncells-x];
                dev_Psi_ezx[getCell(x-1,y-1,20)] = dev_bex[ncells-x]*dev_Psi_ezx[getCell(x-1,y-1,20)]+dev_aex[ncells-x]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)]);
                buffer_Ez += Cezhy[getCell(x,y,nx+1)]*dx*dev_Psi_ezx[getCell(x-1,y-1,2*ncells)];
            }
            if(x>=(nx-ncells)&&x!=nx)
            {
                buffer_Ez = Ceze[getCell(x,y,nx+1)]*dev_Ez[getCell(x,y,nx+1)]+Cezhy[getCell(x,y,nx+1)]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)])/kex[x-nx+ncells]
                    -Cezhx[getCell(x,y,nx+1)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)])/kex[x-nx+ncells];
                dev_Psi_ezx[getCell(x-nx+20,y-1,20)]=dev_bex[x-nx+ncells]*dev_Psi_ezx[getCell(x-nx+20,y-1,20)]+dev_aex[x-nx+ncells]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)]);
                buffer_Ez+=Cezhy[getCell(x,y,nx+1)]*dx*dev_Psi_ezx[getCell(x-nx+20,y-1,2*ncells)];
            }
            if(y<=ncells&&y!=0)
            {
                buffer_Ez = Ceze[getCell(x,y,nx+1)]*dev_Ez[getCell(x,y,nx+1)]+Cezhy[getCell(x,y,nx+1)]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)])/kex[ncells-y]
                    -Cezhx[getCell(x,y,nx+1)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)])/kex[ncells-y];
                dev_Psi_ezy[getCell(x-1,y-1,nx)]=dev_bey[(ncells-y)]*dev_Psi_ezy[getCell(x-1,y-1,nx)]+dev_aey[(ncells-y)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)]);
                buffer_Ez-=Cezhx[getCell(x,y,nx+1)]*dy*dev_Psi_ezy[getCell(x-1,y-1,nx)];
            }
            if(y>=(ny-ncells)&&y!=ny)
            {
                buffer_Ez = Ceze[getCell(x,y,nx+1)]*dev_Ez[getCell(x,y,nx+1)]+Cezhy[getCell(x,y,nx+1)]*(dev_Hy[getCell(x,y,nx)]-dev_Hy[getCell(x-1,y,nx)])/kex[y-ny+ncells]
                    -Cezhx[getCell(x,y,nx+1)]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)])/kex[y-ny+ncells];
                dev_Psi_ezy[getCell(x-1,y-ny+20,nx)]=dev_bey[y-ny+ncells]*dev_Psi_ezy[getCell(x-1,y-ny+20,nx)]+dev_aey[y-ny+ncells]*(dev_Hx[getCell(x,y,nx)]-dev_Hx[getCell(x,y-1,nx)]);
                buffer_Ez-=Cezhx[getCell(x,y,nx+1)]*dy*dev_Psi_ezy[getCell(x-1,y-ny+20,nx)];
            }
        }
        //		unsigned char green = 128+127*buffer_Ez/0.4;
        /*ptr[offset].x = 0;
          ptr[offset].y = green;
          ptr[offset].z = 0;
          ptr[offset].w = 255;*///OpenGL stuff

        //__syncthreads();
        if(isnan(buffer_Ez)) {
            dev_Ez[getCell(x,y,nx+1)] = 0.0;
        }
        else {
            dev_Ez[getCell(x,y,nx+1)] = buffer_Ez;
        }
        //dev_Ez[getCell(x,y,nx+1)] = buffer_Ez;
    }

}

__global__ void Field_reset(float* Ez, float* Hy, float* Hx, float* Psi_ezy,float* Psi_ezx,float* Psi_hyx,float* Psi_hxy,hipComplex*cjzyn,hipComplex*cjzxp,hipComplex*cjzyp,hipComplex*cjzxn,hipComplex*cmxyn,hipComplex*cmyxp,hipComplex*cmxyp,hipComplex*cmyxn)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y+blockDim.y*blockIdx.y;
    int index = x + y*blockDim.x*gridDim.x;
    if(x<=ncells&&x!=0)
    {
        Psi_ezx[getCell(x-1,y-1,20)] =0;
    }
    if(x>=(nx-ncells)&&x!=nx)
    {
        Psi_ezx[getCell(x-nx+20,y-1,20)]=0;
    }
    if(y<=ncells&&y!=0)
    {
        Psi_ezy[getCell(x-1,y-1,nx)]=0;
    }
    if(y>=(ny-ncells)&&y!=ny)
    {
        Psi_ezy[getCell(x-1,y-ny+20,nx)]=0;
    }
    if(x<ncells)
    {

        Psi_hyx[getCell(x,y,20)]=0;
    } 
    if(x>=(nx-ncells))
    {
        Psi_hyx[getCell(x-nx+20,y,2*ncells)]=0.0;
    }
    if(y<ncells)
    {
        Psi_hxy[getCell(x,y,nx)]=0.0;
    }
    if(y>=(ny-ncells))
    {
        Psi_hxy[getCell(x,y-ny+20,nx)]=0.0;
    }
    if(x<=nx&&y<=ny)
    {
        Ez[getCell(x,y,nx+1)] = 0.0;
    }
    if(x<nx&&y<ny)
    {
        Hy[getCell(x,y,nx)] = 0.0;
        Hx[getCell(x,y,nx)] = 0.0;
    }

    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(index<size_cjzy)
        {
            cjzyp[index] = hipComplex(0,0);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements
            cjzyn[index] = hipComplex(0,0);
            cmxyp[index] = hipComplex(0,0);
            cmxyn[index] = hipComplex(0,0);
        }
        if(index<size_cjzx)
        {
            cjzxp[index] = hipComplex(0,0);
            cjzxn[index] = hipComplex(0,0);
            cmyxp[index] = hipComplex(0,0);
            cmyxn[index] = hipComplex(0,0);
        }

    }


}

__global__ void E_inc_update(int *i,float*dev_Hy_inc,float*dev_Hx_inc,float*dev_Psi_ezx_inc,float*dev_aex,float*dev_aey,float*dev_bex,float*dev_bey,float*dev_Psi_ezy_inc,float*kex,float*dev_Ezip,float*dev_Ezic,float*Phi)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    //	int offset = x+y*blockDim.x*gridDim.x;
    float buffer_Ez;
    //float Ceh = (dt/dx)/(eps0);
    float Cezj = -dt/eps0;
    float Ceze = 1;
    float Cezhy = (dt/(dx*eps0));

    if(x<=nx&&y<=ny)
    {

        //if(x==0||x==nx||y==0||y==ny)
        if(x==nx||y==ny||x==0||y==0)
        {
            buffer_Ez=0.0;
        }
        else
        {
            buffer_Ez = Ceze*dev_Ezic[getCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)])
                -Cezhy*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)]);

            if(x==((int)source_x)&&y==(int)(source_y))
            {
                //buffer_Ez=buffer_Ez + Cezj*dev_Jz[*i];
                buffer_Ez=buffer_Ez + 100*Cezj*fwf((float)(*i),0,0,0,0);
            }
            if(x<=ncells&&x!=0)
            {
                buffer_Ez = Ceze*dev_Ezic[getCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)])/kex[ncells-x]
                    -Cezhy*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)])/kex[ncells-x];
                dev_Psi_ezx_inc[getCell(x-1,y-1,20)] = dev_bex[ncells-x]*dev_Psi_ezx_inc[getCell(x-1,y-1,20)]+dev_aex[ncells-x]*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)]);
                buffer_Ez += Cezhy*dx*dev_Psi_ezx_inc[getCell(x-1,y-1,2*ncells)];
            }
            if(x>=(nx-ncells)&&x!=nx)
            {
                buffer_Ez = Ceze*dev_Ezic[getCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)])/kex[x-nx+ncells]
                    -Cezhy*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)])/kex[x-nx+ncells];
                dev_Psi_ezx_inc[getCell(x-nx+20,y-1,20)]=dev_bex[x-nx+ncells]*dev_Psi_ezx_inc[getCell(x-nx+20,y-1,20)]+dev_aex[x-nx+ncells]*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)]);
                buffer_Ez+=Cezhy*dx*dev_Psi_ezx_inc[getCell(x-nx+20,y-1,2*ncells)];
            }
            if(y<=ncells&&y!=0)
            {
                buffer_Ez = Ceze*dev_Ezic[getCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)])/kex[ncells-y]
                    -Cezhy*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)])/kex[ncells-y];
                dev_Psi_ezy_inc[getCell(x-1,y-1,nx)]=dev_bey[(ncells-y)]*dev_Psi_ezy_inc[getCell(x-1,y-1,nx)]+dev_aey[(ncells-y)]*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)]);
                buffer_Ez-=Cezhy*dy*dev_Psi_ezy_inc[getCell(x-1,y-1,nx)];
            }
            if(y>=(ny-ncells)&&y!=ny)
            {
                buffer_Ez = Ceze*dev_Ezic[getCell(x,y,nx+1)]+Cezhy*(dev_Hy_inc[getCell(x,y,nx)]-dev_Hy_inc[getCell(x-1,y,nx)])/kex[y-ny+ncells]
                    -Cezhy*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)])/kex[y-ny+ncells];
                dev_Psi_ezy_inc[getCell(x-1,y-ny+20,nx)]=dev_bey[y-ny+ncells]*dev_Psi_ezy_inc[getCell(x-1,y-ny+20,nx)]+dev_aey[y-ny+ncells]*(dev_Hx_inc[getCell(x,y,nx)]-dev_Hx_inc[getCell(x,y-1,nx)]);
                buffer_Ez-=Cezhy*dy*dev_Psi_ezy_inc[getCell(x-1,y-ny+20,nx)];
            }
        }
        dev_Ezip[getCell(x,y,nx+1)] = dev_Ezic[getCell(x,y,nx+1)];
        dev_Ezic[getCell(x,y,nx+1)] = buffer_Ez;
    }

}

float calc_radiated_power(hipComplex *cjzxp,hipComplex *cjzyp,hipComplex *cjzxn,hipComplex *cjzyn,hipComplex *cmxyp,hipComplex *cmyxp,hipComplex *cmxyn,hipComplex *cmyxn)
{
    int indexofleg1 = nx-2*NF2FFdistfromboundary-2;
    int indexofleg2 = nx+ny-4*NF2FFdistfromboundary-2;
    int indexofleg3 = 2*nx+ny-6*NF2FFdistfromboundary-4;
    int maxindex = 2*nx-8*NF2FFdistfromboundary+2*ny-4;
    int index;
    hipComplex cjz(0,0);
    hipComplex power = 0;

    for(index = 0; index<indexofleg1;index++)
    {   cjz = hipComplex(cjzyn[index].r,-1.0*cjzyn[index].i);//conjugation
        //z x x = y dot -y = -1
        power+=-1.0*cjz*cmxyn[index]*dx;// the negative one comes from the dot product between JxM and the n hat vector
    }
    for(index = indexofleg1; index<indexofleg2;index++)
    {
        cjz = hipComplex(cjzxp[index-indexofleg1].r,-1.0*cjzxp[index-indexofleg1].i);//making the conjugate
        // z cross y = -x dot x = -1
        power+= -1.0*cjz*cmyxp[index-indexofleg1]*dy;//positive x unit normal vector
    }
    for(index = indexofleg2;index<indexofleg3;index++)
    {
        // z cross x = y dot y = 1
        cjz = hipComplex(cjzyp[index-indexofleg2].r,-1.0*cjzyp[index-indexofleg2].i);
        power+= cjz*cmxyp[index-indexofleg2]*dx;//postive y unit normal vector
    }
    for(index = indexofleg3;index<maxindex;index++)
    {
        // z cross y = -x dot -x = 1 
        cjz = hipComplex(cjzxn[index-indexofleg3].r,-1.0*cjzxn[index-indexofleg3].i);
        power += cjz*cmyxn[index-indexofleg3]*dy;// negative x hat n vector
    }
    float realpower = power.r;
    realpower *= 0.5;
    return realpower;
}

__global__ void calculate_JandM_total(float* f,int* timestep,float*dev_Ez,float*dev_Hy,float*dev_Hx,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex*cjzxn,hipComplex*cjzyn,hipComplex*cmxyp,hipComplex*cmyxp,hipComplex*cmxyn,hipComplex*cmyxn,float*dev_Ezic,float*dev_Ezip,float*dev_Hx_inc,float*dev_Hy_inc)
{
    float freq = *f;
    int index = threadIdx.x+blockIdx.x*blockDim.x;// should launch 2*nx-8*NF2FFdistfromboundary+2*ny-4 threads. 
    if(index<=size_NF2FF_total)
    {
        const hipComplex j(0.0,1.0);
        int x = getxfromthreadIdNF2FF(index);
        int y = getyfromthreadIdNF2FF(index);

        float Ez;
        hipComplex pi(PI , 0);
        hipComplex two(2.0,0.0);
        hipComplex negativeone(-1.0,0);
        hipComplex deltatime(dt,0);

        if(isOnyp(x,y))
        {
            Ez = (dev_Ez[getCell(x,y+1,nx+1)]+dev_Ez[getCell(x,y,nx+1)])/2;
            Ez += (dev_Ezic[getCell(x,y+1,nx+1)] + dev_Ezic[getCell(x,y,nx+1)] + dev_Ezip[getCell(x,y+1,nx+1)] + dev_Ezip[getCell(x,y,nx+1)])/4;
            float Hx = dev_Hx[getCell(x,y,nx)] + dev_Hx_inc[getCell(x,y,nx)];
            cjzyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Hx*deltatime*cuexp((float)(-1)*j*(float)2*pi*freq*(float)(*timestep)*deltatime);//cjzyp and cmxyp have nx - 2*NF2FFBoundary -2 elements
            cmxyp[index-(nx+ny-4*NF2FFdistfromboundary-2)] += -1*Ez*deltatime*cuexp((float)-1.0*j*(float)2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
        else if(isOnxp(x))//X faces override y faces at their intersections
        {
            Ez = (dev_Ez[getCell(x,y,nx+1)]+dev_Ez[getCell(x+1,y,nx+1)])/2;
            Ez += (dev_Ezic[getCell(x+1,y,nx+1)] + dev_Ezic[getCell(x,y,nx+1)] + dev_Ezip[getCell(x+1,y,nx+1)] + dev_Ezip[getCell(x,y,nx+1)])/4;
            float Hy = dev_Hy[getCell(x,y,nx)] + dev_Hy_inc[getCell(x,y,nx)];

            cjzxp[index-(nx-2*NF2FFdistfromboundary-2)] += Hy*deltatime*cuexp(-1*j*2*pi*freq*(float)(*timestep)*(float)dt);//cjzxp and cmyxp have ny-2*NF2FFBound elements

            cmyxp[index-(nx-2*NF2FFdistfromboundary-2)] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*pi*freq*((float)(*timestep)-0.5)*(float)dt);// this is the discrete fourier transform, by the way.
        }
        else if(isOnyn(x,y))
        {  
            Ez = (dev_Ez[getCell(x,y,nx+1)]+dev_Ez[getCell(x,y+1,nx+1)])/2;
            Ez += (dev_Ezic[getCell(x,y+1,nx+1)] + dev_Ezic[getCell(x,y,nx+1)] + dev_Ezip[getCell(x,y+1,nx+1)] + dev_Ezip[getCell(x,y,nx+1)])/4;
            float Hx=dev_Hx[getCell(x,y,nx)]+dev_Hx_inc[getCell(x,y,nx)];

            cjzyn[index] += Hx*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt);	//cjzyn and cmxyn need to have nx-2*NF2FFbound-2 elements
            cmxyn[index] += Ez*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
        else if(isOnxn(x))
        {
            Ez = (dev_Ez[getCell(x,y,nx+1)]+dev_Ez[getCell(x+1,y,nx+1)])/2;
            Ez += (dev_Ezic[getCell(x+1,y,nx+1)] + dev_Ezic[getCell(x,y,nx+1)] + dev_Ezip[getCell(x+1,y,nx+1)] + dev_Ezip[getCell(x,y,nx+1)])/4;
            float Hy = dev_Hy[getCell(x,y,nx)] + dev_Hy_inc[getCell(x,y,nx)];
            cjzxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Hy*(float)dt*cuexp((float)(-1)*j*(float)2.0*(float)PI*freq*(float)(*timestep)*(float)dt); // cjzxn and cmyxn must have ny-2*NFdistfromboundary elements
            cmyxn[index-(2*nx+ny-6*NF2FFdistfromboundary-4)] += -1*Ez*(float)dt*cuexp(-1.0*j*2.0*(float)PI*freq*((float)(*timestep)-0.5)*(float)dt);
        }
    }

}

__host__ __device__ int getOptimizationCell(int x, int y)
{
    int x_coord,y_coord;
    x_coord = (x-(nx/2-(int)(breast_radius/dx)))/(2*breast_radius/(9*dx));
    y_coord = (y-(ny/2-breast_radius/dy))/(2*breast_radius/(9*dy));//the optimization space is 216 FDTD cells wide and high. //The optimization space is split into 25 by 25 optimization cells. 
    //each optimization cell has 24 by 24 FDTD cells within it. That's what the 108, 24 and 25 are about.  
    return x_coord+9*y_coord;//The max return should be, 9*9-1, hopefully.
}

void N2FPostProcess (float* D,float f, hipComplex *N,hipComplex *L,hipComplex *cjzxp,hipComplex *cjzyp,hipComplex *cjzxn,hipComplex *cjzyn,hipComplex *cmxyp,hipComplex *cmyxp,hipComplex *cmxyn,hipComplex *cmyxn)
{
    int indexofleg1 = nx-2*NF2FFdistfromboundary-2;
    int indexofleg2 = nx+ny-4*NF2FFdistfromboundary-2;
    int indexofleg3 = 2*nx+ny-6*NF2FFdistfromboundary-4;
    int maxindex = 2*nx-8*NF2FFdistfromboundary+2*ny-4;
    int x,y;

    float rhoprime;
    float Psi;
    int Phi_index;
    hipComplex  Mphi(0,0);
    float Phi;


    float k = 2*PI*f/c0;
    hipComplex  negativeone(-1.0,0.0);
    int index = 0;
    hipComplex jcmpx(0,1);
    //float Prad = calc_radiated_power(cjzxp,cjzyp,cjzxn,cjzyn,cmxyp,cmyxp,cmxyn,cmyxn);
    float Prad = calc_incident_power(f);
    //std::cout<<"Prad = "<<Prad<<std::endl;
    float flx, fly;
    for(Phi_index = 0; Phi_index<numberofobservationangles;Phi_index++)
    {
        Phi = 2*PI/numberofobservationangles*(float)Phi_index;
        for(index = 0;index<indexofleg1;index++)
        {

            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x;//float x
            fly = (float)y + 0.5;
            rhoprime = sqrt(pow((dx*((-1.0*(float)nx/2)+1+flx)),2)+pow((dy*(-1.0*(float)ny/2+1+fly)),2));
            Psi = atan2(-1*((float)ny/2)+1+fly,-1*((float)nx/2)+1+flx)-Phi;
            N[Phi_index]+=-1.0*cjzyn[index]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;
            L[Phi_index]+=-1.0*sin(Phi)*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*cmxyn[index]*dx;//Lphi = 

        }
        for(index = indexofleg1;index<indexofleg2;index++)
        {

            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x+0.5;
            fly = (float)y;
            rhoprime = sqrt(pow((dx*(((float)nx/2)-1-flx)),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2(-1*((float)ny/2)+1+fly,(-1*((float)nx/2)+1+flx))-Phi;
            N[Phi_index]+=-1.0*cjzxp[index-indexofleg1]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
            L[Phi_index]+=cos(Phi)*cmyxp[index-indexofleg1]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;//L_phi = -Lxsin(phi)+Lycos(Phi) here we only have Ly
        }
        for(index=indexofleg2;index<indexofleg3;index++)
        {
            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x;
            fly = (float)y + 0.5;
            rhoprime = sqrt(pow((dx*(((float)nx/2)-1-flx)),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2((-1*(float)ny/2+1+fly),(-1*((float)nx/2)+1+flx))-Phi;
            N[Phi_index]+=-1.0*cjzyp[index-indexofleg2]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;
            L[Phi_index]+=-1.0*sin(Phi)*cmxyp[index-indexofleg2]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dx;//
        }
        for(index = indexofleg3;index<maxindex;index++)
        {
            x = CPUgetxfromthreadIdNF2FF(index);
            y = CPUgetyfromthreadIdNF2FF(index);
            flx = (float)x+0.5;
            fly = (float)y;
            rhoprime = sqrt(pow(dx*(((float)nx/2)-1-flx),2)+pow((dy*(((float)ny/2)-1-fly)),2)); 
            Psi = atan2(-1*((float)ny/2)+1+fly,-1*(float)nx/2+1+flx)-Phi;
            N[Phi_index]+=-1.0*cjzxn[index-indexofleg3]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
            L[Phi_index]+= cos(Phi)*cmyxn[index-indexofleg3]*cuexp(1.0*jcmpx*k*rhoprime*cos(Psi))*dy;
        }
        D[Phi_index] = (k*k*cuabs(L[Phi_index]+(float)eta0*N[Phi_index])*cuabs(L[Phi_index]+(float)eta0*N[Phi_index])/((float)8*(float)PI*(float)eta0*Prad*33.329));//why 33.329?  I dunno, something is probably wrong with Prad.
    }

}

//static void draw_func(void){
//	glDrawPixels(nx,ny,GL_RGBA,GL_UNSIGNED_BYTE,0);
//	glutSwapBuffers;
//}

using namespace std;

__global__ void scattered_parameter_init(float*eps_r_z,float*sigma_e_z,float*Cezeic,float*Cezeip);

double FDTD_GPU(const vector<double> &arguments) {
    cout << "calculating FDTD GPU" << endl;

    hipSetDevice(0);

    vector<float> image;
    //This is setting the material parameters of the optimization cells.
    for (int lerp = 0; lerp < 81; lerp++) {
        image.push_back((float)arguments.at(lerp));
        //image.push_back(10);
    }

    for (int lerp = 81; lerp < 81 * 2; lerp++) {
        image.push_back((float)arguments.at(lerp));
        // image.push_back(0);
    }
    hipError_t error;

    float freq = center_freq;

    int grid_x = int(ceil((float)nx / 22));
    int grid_y = int(ceil((float)ny / 22));

    dim3 grid(grid_x, grid_y);
    dim3 block(22, 22);

    float *Ez = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    float *eps_r_z = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    float *sigma_e_z = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    float *Ceze = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    float *Cezhy = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    float *Cezhx = (float*)malloc(sizeof(float)*(1+nx)*(1+ny));
    //Cezj later if using loop current source
    //float *Cezj = (float*)malloc(sizeof(float)*(1+nx)*(1+ny)); // if using loop current source

    int size = nx + 1;
    float radius;//tumor_radius,tumor_radius_2,tumor_radius_3;

    for (int j = 0; j < ny + 1; j++) {
        for (int i = 0; i < nx + 1; i++) {
            Ez[getCell(i,j,size)] = (float)0;
            sigma_e_z[getCell(i,j,size)] = 0;
            eps_r_z[getCell(i,j,size)] = 1;

            radius = sqrt(pow( ((float)i-nx/2)*dx,2) + pow( ((float)j-ny/2)*dy,2));

            //tumor_radius = sqrt(pow( ((float)i - target_x)*dx,2) + pow( ((float)j-target_y)*dy,2));
            if (radius <= breast_radius) {
                eps_r_z[getCell(i,j,size)] = (float)image.at(getOptimizationCell(i,j)); //This is the line that should be uncommented if using as forward solver
                sigma_e_z[getCell(i,j,size)] = (float)image.at(getOptimizationCell(i,j)+9*9);

                //eps_r_z[getCell(i,j,size)] = 10;
                //sigma_e_z[getCell(i,j,size)] = 0.15;
                //if(tumor_radius <= tumor_size)//delete this if using as forward solver
                //{
                //	eps_r_z[getCell(i,j,size)] = 60;
                //	sigma_e_z[getCell(i,j,size)] = 0.7;
                //}
            }
            Ceze[getCell(i,j,nx+1)] = (2*eps_r_z[getCell(i,j,nx+1)]*eps0-dt*sigma_e_z[getCell(i,j,nx+1)])/(2*eps_r_z[getCell(i,j,nx+1)]*eps0+dt*sigma_e_z[getCell(i,j,nx+1)]);
            Cezhy[getCell(i,j,size)] = (2*dt/dx)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);
            Cezhx[getCell(i,j,size)] = (2*dt/dy)/(2*eps_r_z[getCell(i,j,size)]*eps0+dt*sigma_e_z[getCell(i,j,size)]);
        }
    }

    float *sigma_e_pml = (float*)malloc(sizeof(float)*ncells);
    float *sigma_m_pml = (float*)malloc(sizeof(float)*ncells);

    //initialize
    float sigma_max = (npml+1)/(150*PI*dx);
    float rho;
    for (int i = 0; i < ncells; i++) {
        rho = ((float)i+0.25)/ncells;
        sigma_e_pml[i] = sigma_max*sigma_factor*pow(rho,npml);
        sigma_m_pml[i] = (mu0/eps0)*sigma_max*sigma_factor*pow(rho,npml);
        //cout<<"sigma_e_pml = "<<sigma_e_pml[i]<<endl;
        //cout<<"sigma_m_pml "<<sigma_m_pml[i]<<endl;
    }

    float *kex = (float*)malloc(sizeof(float)*ncells);
    float *kmx = (float*)malloc(sizeof(float)*ncells);
    float *aex = (float*)malloc(sizeof(float)*ncells);
    float *bex = (float*)malloc(sizeof(float)*ncells);
    float *amx = (float*)malloc(sizeof(float)*ncells);
    float *bmx = (float*)malloc(sizeof(float)*ncells);
    float *alpha_e = (float*)malloc(sizeof(float)*ncells);
    float *alpha_m = (float*)malloc(sizeof(float)*ncells);


    //Initialize kex and kmx (formerly k_e_init and k_m_init)
    //And alpha_e and alpha_m, and aex, bex, kex, amx, bmx, kmx
    for (int i = 0; i < ncells; i++) {
        rho = ((float)i+0.25)/ncells;
        kex[i]=pow(rho,npml)*(kmax-1)+1;
        kmx[i]=pow(rho,npml)*(kmax-1)+1;
        alpha_e[i]=alpha_min+(alpha_max-alpha_min)*rho;

        rho = ((float)i+0.75)/ncells;
        alpha_m[i]=(mu0/eps0)*(alpha_min+(alpha_max-alpha_min)*rho);

        aex[i]=((bex[i]-1)*sigma_e_pml[i])/(dx*(sigma_e_pml[i]*kex[i]+alpha_e[i]*kex[i]*kex[i]));
        bex[i]=exp(-1*(dt/eps0)*(sigma_e_pml[i]/kex[i]+alpha_e[i]));
        amx[i]=(bmx[i]-1)*sigma_m_pml[i]/(dx*(sigma_m_pml[i]*kmx[i]+alpha_m[i]*kmx[i]*kmx[i]));

        float argument = -1*(dt/mu0)*((sigma_m_pml[i]/kmx[i])+alpha_m[i]);
        bmx[i]=exp(argument);
        //cout<<"kex["<<i<<"]= "<<kex[i]<<endl;
        //cout<<"kmx["<<i<<"]= "<<kmx[i]<<endl;
        //cout<<"aex["<<i<<"]= "<<aex[i]<<endl;
        //cout<<"amx["<<i<<"]= "<<amx[i]<<endl;
        //cout<<"bex["<<i<<"]= "<<bex[i]<<endl;
        //cout<<"bmx["<<i<<"]= "<<bmx[i]<<endl;
        //cout<<"alpha_e = "<<alpha_e[i]<<endl;
        //cout<<"alpha_m = "<<alpha_m[i]<<endl;
    }

    float *Psi_ezy = (float*)malloc(sizeof(float)*ny*20);
    float *Psi_ezx = (float*)malloc(sizeof(float)*nx*20);
    float *Psi_hyx = (float*)malloc(sizeof(float)*ny*20);
    float *Psi_hxy = (float*)malloc(sizeof(float)*nx*20);

    for (int i = 0; i < nx * 20; i++) {
        Psi_ezy[i] = 0.0;
        Psi_hxy[i] = 0.0;
    }

    for (int i = 0; i< ny * 20; i++) {
        Psi_ezx[i] = 0.0;
        Psi_hyx[i] = 0.0;
    }

    float *D = (float*)malloc(sizeof(float)*numberofexcitationangles*numberofobservationangles);//D = (float*)malloc(numberofobservationangles*sizeof(float));

    float *Hy = (float*)malloc(sizeof(float)*nx*ny);
    float *Hx = (float*)malloc(sizeof(float)*nx*ny);

    //This are output values from the device
    hipComplex *hcjzyp = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzy);
    hipComplex *hcjzyn = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzy);
    hipComplex *hcjzxp = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzx);
    hipComplex *hcjzxn = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzx);
    hipComplex *hcmxyn = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzy);
    hipComplex *hcmxyp = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzy);
    hipComplex *hcmyxp = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzx);
    hipComplex *hcmyxn = (hipComplex*)malloc(sizeof(hipComplex)*size_cjzx);

    hipComplex *L  = (hipComplex*)malloc(sizeof(hipComplex)*size_NF2FF_total);
    hipComplex *N  = (hipComplex*)malloc(sizeof(hipComplex)*size_NF2FF_total);

    hipComplex *cjzxp, *cjzyp, *cjzxn, *cjzyn, *cmxyp, *cmyxp, *cmxyn, *cmyxn;

    float *dev_Cezeic, *dev_Cezeip;
    float *dev_sigma_e_z, *dev_eps_r_z;
    float *dev_freq, *dev_Phi;
    float *dev_Ceze, *dev_Cezhy, *dev_Cezhx, *dev_bex, *dev_aex, *dev_bmx, *dev_amx, *dev_kex, *dev_kmx;//dev_Cezj if using loop current source
    float *dev_Ez, *dev_Hy, *dev_Hx;

    float *dev_Psi_ezy, *dev_Psi_ezx, *dev_Psi_hyx, *dev_Psi_hxy;

    hipMalloc(&dev_eps_r_z,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_sigma_e_z,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezeic,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezeip,sizeof(float)*(nx+1)*(ny+1));
    hipMemcpy(dev_eps_r_z,eps_r_z,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_sigma_e_z,sigma_e_z,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    scattered_parameter_init<<<grid,block>>>(dev_eps_r_z,dev_sigma_e_z,dev_Cezeic,dev_Cezeip);
    //float *Cezeic = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    // float *Cezeip = (float*)malloc((sizeof(float))*(nx+1)*(ny+1));
    //hipMemcpy(Cezeic,dev_Cezeic,sizeof(float)*(nx+1)*(ny+1),hipMemcpyDeviceToHost);
    //hipMemcpy(Cezeip,dev_Cezeip,sizeof(float)*(nx+1)*(ny+1),hipMemcpyDeviceToHost);


    hipMalloc(&dev_Phi,sizeof(float));
    hipMalloc(&dev_kex,sizeof(float)*10);
    hipMalloc(&dev_kmx,sizeof(float)*10);
    hipMalloc(&dev_Ez,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Hy,sizeof(float)*nx*ny);
    hipMalloc(&dev_freq ,sizeof(float));
    hipMalloc(&dev_Hx,sizeof(float)*nx*ny);
    hipMalloc(&dev_Psi_ezy,sizeof(float)*20*(nx+1));
    hipMalloc(&dev_Psi_ezx,sizeof(float)*20*(ny+1));
    hipMalloc(&dev_Psi_hyx,sizeof(float)*20*(ny));
    hipMalloc(&dev_Psi_hxy,sizeof(float)*20*(nx));

    hipMalloc(&cjzxp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzxn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cjzyn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmxyn,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxp,sizeof(hipComplex)*size_NF2FF_total);
    hipMalloc(&cmyxn,sizeof(hipComplex)*size_NF2FF_total);

    hipMemcpy(dev_freq,&freq,sizeof(float),hipMemcpyHostToDevice);

    hipMalloc(&dev_bex,sizeof(float)*10);
    hipMalloc(&dev_bmx,sizeof(float)*10);
    hipMalloc(&dev_amx,sizeof(float)*10);
    hipMalloc(&dev_aex,sizeof(float)*10);
    hipMalloc(&dev_Ceze,sizeof(float)*(nx+1)*(ny+1));
    hipMalloc(&dev_Cezhy,sizeof(float)*(nx+1)*(ny+1));


    //hipMalloc(&dev_Cezj,sizeof(float)*(nx+1)*(ny+1)); if using current source

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }
    Field_reset<<<grid,block>>>(dev_Ez, dev_Hy, dev_Hx, dev_Psi_ezy, dev_Psi_ezx, dev_Psi_hyx, dev_Psi_hxy,cjzyn,cjzxp,cjzyp,cjzxn,cmxyn,cmyxp,cmxyp,cmyxn);
    //Field_reset is also good for making all these values zero.


    hipMemcpy(dev_kex,kex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_kmx,kmx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_aex,aex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_bex,bex,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_bmx,bmx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_amx,amx,sizeof(float)*10,hipMemcpyHostToDevice);
    hipMemcpy(dev_Ceze,Ceze,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    hipMemcpy(dev_Cezhy,Cezhy,sizeof(float)*(nx+1)*(ny+1),hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess) {
        printf("%s\n",hipGetErrorString(error));
    }

    int *dev_i;
    hipMalloc(&dev_i,sizeof(int));
    float test_Ez;

    dim3 gridNF2FF((int)ceil(size_NF2FF_total/512.0));
    dim3 blockNF2FF(512);

    float test_Ez_2;
    float Phi;

    for(int Phi_index = 0; Phi_index < numberofexcitationangles; Phi_index++) {

        Phi = Phi_index*2*PI/numberofexcitationangles;
        hipMemcpy(dev_Phi,&Phi,sizeof(float),hipMemcpyHostToDevice);

        for (int i = 0; i < number_of_time_steps; i++) {
            hipMemcpy(dev_i,&i,sizeof(int),hipMemcpyHostToDevice);
            H_field_update<<<grid,block>>>(dev_Hy,dev_Hx,dev_Ez,dev_bmx,dev_Psi_hyx,dev_amx,dev_bmx,dev_amx,dev_Psi_hxy,dev_kmx);
            E_field_update<<<grid,block>>>(dev_i,dev_Ez,dev_Hy,dev_Hx,dev_Psi_ezx,dev_aex,dev_aex,dev_bex,dev_bex,dev_Psi_ezy,dev_kex,dev_Cezhy,dev_Cezhy,dev_Ceze,dev_Cezeip,dev_Cezeic,dev_Phi);
            calculate_JandM<<<gridNF2FF,blockNF2FF>>>(dev_freq, dev_i,dev_Ez,dev_Hy,dev_Hx,cjzxp,cjzyp,cjzxn,cjzyn,cmxyp,cmyxp,cmxyn,cmyxn);

        }

        hipMemcpy(hcjzyn,cjzyn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcjzxp,cjzxp,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
        hipMemcpy(hcjzyp,cjzyp,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcjzxn,cjzxn,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
        hipMemcpy(hcmxyn,cmxyn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcmyxp,cmyxp,sizeof(hipComplex)*size_cjzx,hipMemcpyDeviceToHost);
        hipMemcpy(hcmxyp,cmxyp,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);
        hipMemcpy(hcmyxn,cmyxn,sizeof(hipComplex)*size_cjzy,hipMemcpyDeviceToHost);

        hipComplex nullComplex(0,0);
        for (int i = 0; i < size_NF2FF_total; i++) {
            L[i] = nullComplex;
            N[i] = nullComplex;
        }

        N2FPostProcess(D + Phi_index*numberofobservationangles, freq,N,L,hcjzxp,hcjzyp,hcjzxn,hcjzyn,hcmxyp,hcmyxp,hcmxyn,hcmyxn);
        //notice the D + Phi_index*numberofobservationangles. D is in total 4*numberofobservaion angles, so that's how we fill them in sequentially.

        //for(int i = 0;i<numberofobservationangles;i++)  // This is for recording simulated measured data
        //{
        //measurement_data<<*(D+Phi_index*numberofobservationangles+i)<<" , ";
        //cout<<*(D+Phi_index*numberofobservationangles+i)<<endl;
        //}

        //measurement_data<<endl;
        Field_reset<<<grid,block>>>(dev_Ez, dev_Hy, dev_Hx, dev_Psi_ezy, dev_Psi_ezx, dev_Psi_hyx, dev_Psi_hxy,cjzyn,cjzxp,cjzyp,cjzxn,cmxyn,cmyxp,cmxyp,cmyxn);

    }



    float measurement[numberofobservationangles*numberofexcitationangles] = {0.544912 , 0.518606 , 0.439233 , 0.330533 , 0.219116 , 0.135115 , 0.0923969 , 0.0774134 , 0.0740459 , 0.0739238 , 0.0660047 , 0.0465372 , 0.0248307 , 0.00913681 , 0.00186162 , 0.0038402 , 0.0130785 , 0.0238094 , 0.0312918 , 0.035705 , 0.0388307 , 0.039513 , 0.0368443 , 0.0338221 , 0.0324815 , 0.0305907 , 0.0270149 , 0.0239178 , 0.0224438 , 0.021849 , 0.0217346 , 0.0222152 , 0.023146 , 0.0245181 , 0.0267161 , 0.0286964 , 0.0276803 , 0.0235098 , 0.0197177 , 0.0183168 , 0.0196998 , 0.0261493 , 0.0375584 , 0.0479223 , 0.0511598 , 0.0461443 , 0.035713 , 0.0249863 , 0.0203708 , 0.0260456 , 0.0395441 , 0.054163 , 0.0660136 , 0.0763823 , 0.0935922 , 0.132053 , 0.201299 , 0.299247 , 0.410792 , 0.504467 , 
        0.0490085 , 0.0278468 , 0.0123693 , 0.00899709 , 0.0196632 , 0.0401112 , 0.0623734 , 0.0809561 , 0.096057 , 0.113814 , 0.145125 , 0.200388 , 0.283438 , 0.386362 , 0.486139 , 0.549594 , 0.547993 , 0.475775 , 0.358033 , 0.230962 , 0.118935 , 0.039843 , 0.00700227 , 0.0112335 , 0.0300356 , 0.0494414 , 0.0605159 , 0.0585777 , 0.0503323 , 0.045704 , 0.0474064 , 0.0523123 , 0.0558987 , 0.0545722 , 0.0475098 , 0.0366045 , 0.0248037 , 0.0155752 , 0.0115322 , 0.0127167 , 0.0176523 , 0.0243556 , 0.0310764 , 0.037444 , 0.0432292 , 0.0469609 , 0.0471761 , 0.0435653 , 0.0369347 , 0.0293987 , 0.0235478 , 0.0206039 , 0.020754 , 0.0247748 , 0.0336772 , 0.047007 , 0.0618746 , 0.0734482 , 0.0763332 , 0.0674785 , 
        0.0463129 , 0.0448933 , 0.0398454 , 0.0319834 , 0.0239428 , 0.0174267 , 0.0129155 , 0.0116624 , 0.0154122 , 0.0247183 , 0.0376821 , 0.0494142 , 0.0552493 , 0.0544909 , 0.0501016 , 0.0466044 , 0.047395 , 0.0522298 , 0.0576919 , 0.0588555 , 0.0504011 , 0.0311956 , 0.0107719 , 0.00755493 , 0.0394798 , 0.116099 , 0.232324 , 0.36478 , 0.478314 , 0.541685 , 0.541186 , 0.484009 , 0.391878 , 0.291105 , 0.204554 , 0.145352 , 0.113254 , 0.0973423 , 0.0835717 , 0.0637299 , 0.0397899 , 0.0189781 , 0.00814281 , 0.0118845 , 0.0291142 , 0.0513172 , 0.0680543 , 0.0744519 , 0.0718442 , 0.0622228 , 0.0473734 , 0.0329352 , 0.0245156 , 0.0212818 , 0.0204027 , 0.0228792 , 0.0298908 , 0.0380399 , 0.0432513 , 0.0455291 , 
        0.0469428 , 0.049667 , 0.0453111 , 0.0370016 , 0.0278006 , 0.0201062 , 0.0173687 , 0.020228 , 0.0242543 , 0.0264199 , 0.0275476 , 0.027771 , 0.0262174 , 0.0237332 , 0.0219206 , 0.0212424 , 0.0214967 , 0.0226845 , 0.0248514 , 0.0275874 , 0.0300439 , 0.0318892 , 0.0340621 , 0.0369823 , 0.0388068 , 0.0379494 , 0.0350817 , 0.030462 , 0.0230471 , 0.0133404 , 0.00457234 , 0.00152755 , 0.00874873 , 0.0260448 , 0.0463293 , 0.0633742 , 0.0751071 , 0.0775575 , 0.0756597 , 0.0916989 , 0.141021 , 0.22185 , 0.328433 , 0.44207 , 0.524772 , 0.544711 , 0.498668 , 0.407614 , 0.29953 , 0.199594 , 0.128704 , 0.0929922 , 0.0772499 , 0.0654169 , 0.0536587 , 0.0399619 , 0.0255793 , 0.0193488 , 0.0253531 , 0.0373143 , 
    };//I've just hardcoded the measurement values.  Maybe later we'll read them from a text file.


    float fit;
    fit = fitness(D, numberofobservationangles * numberofexcitationangles, measurement);

    error = hipGetLastError();

    free(Ceze);
    free(Cezhy);
    free(Cezhx);
    free(Ez);
    free(eps_r_z);
    free(sigma_e_z);
    free(Hy);
    free(Hx);
    free(kex);
    free(aex);
    free(bex);
    free(amx);
    free(bmx);
    free(alpha_e);
    free(alpha_m);
    free(sigma_e_pml);
    free(sigma_m_pml);
    free(Psi_ezy);
    free(Psi_ezx);
    free(Psi_hyx);
    free(Psi_hxy);
    free(kmx);
    free(D);

    free(hcjzxp);
    free(hcjzyp);
    free(hcjzxn);
    free(hcjzyn);
    free(hcmxyp);
    free(hcmyxp);
    free(hcmxyn);
    free(hcmyxn);

    free(L);
    free(N);

    hipFree(dev_Cezeic);
    hipFree(dev_Cezeip);
    hipFree(dev_sigma_e_z);
    hipFree(dev_eps_r_z);
    hipFree(dev_freq);

    hipFree(cjzxp);
    hipFree(cjzyp);
    hipFree(cjzxn);
    hipFree(cjzyn);
    hipFree(cmxyp);
    hipFree(cmyxp);
    hipFree(cmxyn);
    hipFree(cmyxn);

    hipFree(dev_Ceze);
    hipFree(dev_Cezhy);
    hipFree(dev_Cezhx);

    hipFree(dev_bex);
    hipFree(dev_aex);
    hipFree(dev_bmx);
    hipFree(dev_amx);
    hipFree(dev_kex);
    hipFree(dev_kmx);
    hipFree(dev_Ez);
    hipFree(dev_Hy);
    hipFree(dev_Hx);
    hipFree(dev_Psi_ezy);
    hipFree(dev_Psi_ezx);
    hipFree(dev_Psi_hyx);
    hipFree(dev_Psi_hxy);

    cout << "fitness is: " << fit << endl;
    return (double)fit;
}

__global__ void scattered_parameter_init(float*eps_r_z,float*sigma_e_z,float*Cezeic,float*Cezeip)
{
    int x=threadIdx.x+blockDim.x*blockIdx.x;
    int y=threadIdx.y+blockDim.y*blockIdx.y;
    if(x<(nx+1)&&y<(ny+1))
    {
        Cezeic[getCell(x,y,nx+1)] = (2*(eps0-eps0*eps_r_z[getCell(x,y,nx+1)])-sigma_e_z[getCell(x,y,nx+1)]*dt)/(2*eps0*eps_r_z[getCell(x,y,nx+1)]+sigma_e_z[getCell(x,y,nx+1)]*dt);
        Cezeip[getCell(x,y,nx+1)] = -1*(2*(eps0-eps0*eps_r_z[getCell(x,y,nx+1)])+sigma_e_z[getCell(x,y,nx+1)]*dt)/(2*eps0*eps_r_z[getCell(x,y,nx+1)]+sigma_e_z[getCell(x,y,nx+1)]*dt);

    }
}
